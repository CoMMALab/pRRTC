#include "hip/hip_runtime.h"
#include "RRT_interleaved.hh"
#include "Robots.hh"
#include "collision_backends.cuh"
#include "collision/environment.hh"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <vector>
#include <iostream>
#include <cassert>
#include <algorithm>

/*
Parallelized RRT with parallelized collision checking.
Interleaved strategy: sample states in parallel, then check edges in parallel, then repeat.
*/

__device__ int atomic_free_index;
__device__ bool reached_goal = false;
__device__ int found_goal_idx;

const int MAX_SAMPLES = 1000000;
const int MAX_ITERS = 1000000;
const int COORD_BOUND = 3.0;
const int NUM_NEW_CONFIGS = 1024;
const int GRANULARITY = 4096;
const float RRT_RADIUS = 2.0;

using namespace ppln;

__device__ inline void print_config(float *config, int dim) {
    for (int i = 0; i < dim; i++) {
        printf("%f ", config[i]);
    }
    printf("\n");
}

// granularity = number of interpolated points to check along each edge
// total number of threads we need is edges * granularity
// Each block is of size granularity and it checks one edge. Each thread in the block checks a consecutive interpolated point along the edge.
template <typename Robot>
__global__ void validate_edges(float *new_configs, int *new_config_parents, float *new_config_dist, bool *cc_result, int *num_colliding_edges, ppln::collision::Environment<float> *env, float *nodes, int granularity, int num_samples) {
    static constexpr auto dim = Robot::dimension;
    int tid_in_block = threadIdx.x;
    int bid = blockIdx.x;
    // total_threads = num_samples * granularity;
    if (bid >= num_samples) return;
    if (tid_in_block >= granularity) return;
    
    __shared__ float len;
    __shared__ float shared_edge_start[dim];
    if (tid_in_block == 0) {
        float *edge_start = &nodes[new_config_parents[bid] * dim];
        len = new_config_dist[bid];
        for (int i = 0; i < dim; i++) {
            shared_edge_start[i] = edge_start[i];
        }
    }
    __syncthreads();



    float delta = len / (float) granularity;

    // calculate the configuration this thread will be checking
    float config[dim];
    for (int i = 0; i < dim; i++) {
        config[i] = shared_edge_start[i] + (tid_in_block * delta);
    }

    // check for collision
    bool config_in_collision = not ppln::collision::fkcc<Robot>(config, env);
    cc_result[bid] |= config_in_collision;
}


// initialize cuda random
__global__ void init_rng(hiprandState* states, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}

// each thread is responsible for finding a new edge to check
// sample a new state -> connect it to nearest neighbor in our tree
template <typename Robot>
__global__ void sample_edges(float *new_configs, int *new_config_parents, float *new_config_dist, float *nodes, float *goal_configs, int num_goals, hiprandState *rng_states, int num_samples) {
    static constexpr auto dim = Robot::dimension;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_samples) return;
    hiprandState local_rng_state = rng_states[tid];

    float *config = &new_configs[tid * dim];

    // if this is the first thread always sample a random goal
    if (tid == 0) {
        int goal_idx = (int)(num_goals * hiprand_uniform(&local_rng_state));
        float *goal_config = goal_configs + (dim * sizeof(float) * goal_idx);
        for (int i = 0; i < dim; i++) {
            config[i] = goal_config[i];
        }
    }
    // otherwise sample a random config
    else {
        for (int i = 0; i < dim; i++) {
            config[i] = (2 * COORD_BOUND * hiprand_uniform(&local_rng_state)) - COORD_BOUND;
        }
        rng_states[tid] = local_rng_state;
    }

    // find nearest neighbor
    float min_dist = 1000000000.0;
    int nearest_idx = -1;
    float dist;
    for (int i = 0; i < atomic_free_index; i++) {
        dist = device_utils::l2_dist(&nodes[i * dim], config, dim);
        if (dist < min_dist) {
            nearest_idx = i;
            min_dist = dist;
        }
    }

    // if (tid == 0) {
    //     printf("%d\n", atomic_free_index);
    //     printf("%f\n", min_dist);
    // }

    // keep it within the rrt range
    float scale = min(1.0f, RRT_RADIUS / min_dist);
    for (int i = 0; i < dim; i++) {
        config[i] *= scale;
    }
    min_dist *= scale;
    
    new_config_dist[tid] = min_dist;

    // set the parent of the new config
    new_config_parents[tid] = nearest_idx;
}

// grow the RRT tree after we figure out what edges have no collisions
// each thread is responsible for adding one edge to the tree
template <typename Robot>
__global__ void grow_tree(float *new_configs, int *new_config_parents, float *new_config_dist, bool *cc_result, float *nodes, int *parents, int *num_colliding_edges, float *goal_configs, int num_goals, int num_samples) {
    static constexpr auto dim = Robot::dimension;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_samples) return;
    if (cc_result[tid]) return;  // this edge had a collision, don't add it

    // The first edge is always to the goal so if we get here, we need to check if we reached the goal.
    if (tid == 0) {
        int goal_size = dim * sizeof(float);
        float dist_to_goal;
        for (int i = 0; i < num_goals; i++) {
            dist_to_goal = device_utils::l2_dist(new_configs, &goal_configs[i * goal_size], dim);
            if (dist_to_goal < 0.0001) {
                reached_goal = true;
                found_goal_idx = i;
                return;
            }
        }
    }

    // Atomically get the next free index
    int my_index = atomicAdd(&atomic_free_index, 1);
    
    // Copy the configuration to the nodes array
    for (int i = 0; i < dim; i++) {
        nodes[my_index * dim + i] = new_configs[tid * dim + i];
    }
    
    // Set the parent
    parents[my_index] = new_config_parents[tid];
}

inline void reset_device_variables() {
    int zero = 0;
    bool false_val = false;
    
    hipMemcpyToSymbol(HIP_SYMBOL(atomic_free_index), &zero, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(reached_goal), &false_val, sizeof(bool));
    hipMemcpyToSymbol(HIP_SYMBOL(found_goal_idx), &zero, sizeof(int));
}


template <typename Robot>
PlannerResult<Robot> solve(typename Robot::Configuration &start, std::vector<typename Robot::Configuration> &goals, ppln::collision::Environment<float> &h_environment) {
    static constexpr auto dim = Robot::dimension;
    std::size_t iter = 0;
    std::size_t start_index = 0;
    std::size_t free_index = start_index + 1;

    auto start_time = std::chrono::steady_clock::now();
    PlannerResult<Robot> res;

    // copy stuff to GPU
    // GPU needs: start, goal, tree, parents, nodes

    float *start_config;
    float *goal_configs;
    int num_goals = goals.size();
    float *nodes;
    int *parents;
    const std::size_t config_size = dim * sizeof(float);
    hipMalloc(&start_config, config_size);
    hipMalloc(&goal_configs, config_size * num_goals);
    hipMalloc(&nodes, MAX_SAMPLES * config_size);
    hipMalloc(&parents, MAX_SAMPLES * sizeof(int));
    hipMemcpy(start_config, start.data(), config_size, hipMemcpyHostToDevice);
    hipMemcpy(goal_configs, goals.data(), config_size, hipMemcpyHostToDevice);
    // add the start config to the tree, and set the start to be it's own parent.
    hipMemcpy(nodes, start.data(), config_size, hipMemcpyHostToDevice);
    hipMemcpy(parents, &start_index, sizeof(int), hipMemcpyHostToDevice);

    // create a hiprandState for each thread -> holds state of RNG for each thread seperately
    // For growing the tree we will create NUM_NEW_CONFIGS threads
    hiprandState *rng_states;
    hipMalloc(&rng_states, NUM_NEW_CONFIGS * sizeof(hiprandState));
    int blockSize = 256;
    int numBlocks = (NUM_NEW_CONFIGS + blockSize - 1) / blockSize;
    init_rng<<<numBlocks, blockSize>>>(rng_states, 0);

    // create arrays on the gpu to hold the newly sampled configs, and their parents, and dist to parent
    float *new_configs;
    hipMalloc(&new_configs, NUM_NEW_CONFIGS * config_size);
    int *new_config_parents;
    hipMalloc(&new_config_parents, NUM_NEW_CONFIGS * sizeof(int));
    float *new_config_dist;
    hipMalloc(&new_config_dist, NUM_NEW_CONFIGS * sizeof(float));
    hipMemset(new_config_dist, 0, NUM_NEW_CONFIGS * sizeof(float));

    // create an array to hold the result of collision check for each new edge
    bool *cc_result;
    hipMalloc(&cc_result, NUM_NEW_CONFIGS * sizeof(bool));
    hipMemset(cc_result, 0, NUM_NEW_CONFIGS);
    int *num_colliding_edges;
    hipMalloc(&num_colliding_edges, sizeof(int));

    // free index for next available position in the nodes array
    hipMemcpyToSymbol(HIP_SYMBOL(atomic_free_index), &free_index, sizeof(int));

    // allocate for obstacles
    ppln::collision::Environment<float> *env;
    hipMalloc(&env, sizeof(env));
    hipMemcpy(env, &h_environment, sizeof(env), hipMemcpyHostToDevice);


    bool done = false;
    // main RRT loop
    while (iter++ < MAX_ITERS && free_index < MAX_SAMPLES) {
        // std::cout << iter << std::endl;
        // sample configurations and get edges to be checked
        sample_edges<Robot><<<numBlocks, blockSize>>>(new_configs, new_config_parents, new_config_dist, nodes, goal_configs, num_goals, rng_states, NUM_NEW_CONFIGS);
        hipDeviceSynchronize();
        // collision check all the edges
        hipMemset(cc_result, 0, NUM_NEW_CONFIGS);
        validate_edges<Robot><<<NUM_NEW_CONFIGS, GRANULARITY>>>(new_configs, new_config_parents, new_config_dist, cc_result, num_colliding_edges, env, nodes, GRANULARITY, NUM_NEW_CONFIGS);
        hipDeviceSynchronize();
        // add all the new edges to the tree
        grow_tree<Robot><<<numBlocks, blockSize>>>(new_configs, new_config_parents, new_config_dist, cc_result, nodes, parents, num_colliding_edges, goal_configs, num_goals, NUM_NEW_CONFIGS);
        hipDeviceSynchronize();
        
        // update free index
        hipMemcpyFromSymbol(&free_index, HIP_SYMBOL(atomic_free_index), sizeof(int), 0, hipMemcpyDeviceToHost);
        hipMemcpyFromSymbol(&done, HIP_SYMBOL(reached_goal), sizeof(bool), 0, hipMemcpyDeviceToHost);
        if (done) break;
    }
    res.tree_size = free_index;
    res.iters = iter;
    res.attempted_tree_size = NUM_NEW_CONFIGS * iter;
    // retrieve data from gpu
    std::vector<int> h_parents(MAX_SAMPLES);
    std::vector<float> h_nodes(MAX_SAMPLES * dim);
    hipMemcpy(h_parents.data(), parents, MAX_SAMPLES * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_nodes.data(), nodes, MAX_SAMPLES * config_size, hipMemcpyDeviceToHost);

    // change representation of RRT nodes from flattened vector to vector of Configurations
    typename Robot::Configuration cfg;
    for (int i = 0; i < MAX_SAMPLES; i++) {
        for (int j = 0; j < dim; j++) {
            cfg[j] = h_nodes[i * dim + j];
        }
        res.nodes.emplace_back(cfg);
    }

    // compute the path
    // std::vector<int> path;
    
    if (done) {
        // get the index of the goal we found in the goals array
        int h_goal_idx;
        hipMemcpyFromSymbol(&h_goal_idx, HIP_SYMBOL(found_goal_idx), sizeof(int), 0, hipMemcpyDeviceToHost);
        // std::cout << "Found Goal: " << h_goal_idx << std::endl;
        res.solved = true;
        // get parent at position 0 in new_config_parents, because that will be the parent of the goal
        int parent_idx = -1;
        hipMemcpy(&parent_idx, new_config_parents, sizeof(int), hipMemcpyDeviceToHost);
        assert(parent_idx != -1);
        typename Robot::Configuration cfg;
        typename Robot::Configuration cfg_parent;
        std::copy_n(h_nodes.begin() + parent_idx, dim, cfg.begin());
        res.cost += l2dist<Robot>(goals[h_goal_idx], cfg);
        // std::cout << "\n----\n";
        // std::cout << "goal: ";
        // for (int i = 0; i < goals[h_goal_idx].size(); i++) std::cout << goals[h_goal_idx][i] << " ";
        // std::cout << '\n';
        while (parent_idx != h_parents[parent_idx]) {
            // std::cout << parent_idx << std::endl;
            std::copy_n(h_nodes.begin() + parent_idx, dim, cfg.begin());
            std::copy_n(h_nodes.begin() + h_parents[parent_idx], dim, cfg_parent.begin());

            // for (int i = 0; i < cfg.size(); i++) std::cout << cfg[i] << " ";
            // std::cout << '\n';
            // for (int i = 0; i < cfg.size(); i++) std::cout << cfg_parent[i] << " ";
            // std::cout << '\n';

            // std::cout << l2dist<Robot>(cfg, cfg_parent) << "\n";
            res.cost += l2dist<Robot>(cfg, cfg_parent);
            res.path.emplace_back(parent_idx);
            parent_idx = h_parents[parent_idx];
        }

        // std::copy_n(h_nodes.begin() + parent_idx, dim, cfg.begin());
        // std::copy_n(h_nodes.begin() + h_parents[parent_idx], dim, cfg_parent.begin());
        // for (int i = 0; i < cfg.size(); i++) std::cout << cfg[i] << " ";
        // std::cout << '\n';
        // for (int i = 0; i < cfg.size(); i++) std::cout << cfg_parent[i] << " ";
        // std::cout << "\nstart: ";
        // for (int i = 0; i < cfg.size(); i++) std::cout << start[i] << " ";
        // std::cout << '\n';
        // std::cout << "\n----\n";
        res.path.emplace_back(parent_idx);
        std::reverse(res.path.begin(), res.path.end());
    }
    res.nanoseconds = get_elapsed_nanoseconds(start_time);
    reset_device_variables();
    hipFree(start_config);
    hipFree(goal_configs);
    hipFree(nodes);
    hipFree(parents);
    hipFree(rng_states);
    hipFree(new_configs);
    hipFree(new_config_parents);
    hipFree(cc_result);
    hipFree(num_colliding_edges);
    hipFree(env);
    return res;
}

template PlannerResult<typename ppln::robots::Sphere> solve<ppln::robots::Sphere>(std::array<float, 3>&, std::vector<std::array<float, 3>>&, ppln::collision::Environment<float>&);
template PlannerResult<typename ppln::robots::Panda> solve<ppln::robots::Panda>(std::array<float, 7>&, std::vector<std::array<float, 7>>&, ppln::collision::Environment<float>&);