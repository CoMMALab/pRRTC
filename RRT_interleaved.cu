#include "hip/hip_runtime.h"
#include "RRT_interleaved.hh"
#include "Robots.hh"
#include "collision_backends.cu"
#include "collision/environment.hh"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <vector>
#include <iostream>
#include <cassert>
#include <algorithm>

/*
Parallelized RRT with parallelized collision checking.
Interleaved strategy: sample states in parallel, then check edges in parallel, then repeat.
*/

__device__ int atomic_free_index;
__device__ bool reached_goal = false;
__device__ int found_goal_idx;

const int MAX_SAMPLES = 1000000;
const int MAX_ITERS = 1000000;
const int COORD_BOUND = 2.0;
const int NUM_NEW_CONFIGS = 1024;
const int GRANULARITY = 1024;
const float RRT_RADIUS = 1.0;


__device__ inline void print_config(float *config, int dim) {
    for (int i = 0; i < dim; i++) {
        printf("%f ", config[i]);
    }
    printf("\n");
}

// granularity = number of interpolated points to check along each edge
// total number of threads we need is edges * granularity
// Each block is of size granularity and it checks one edge. Each thread in the block checks a consecutive interpolated point along the edge.
template <typename Robot>
__global__ void validate_edges(float *new_configs, int *new_config_parents, bool *cc_result, int *num_colliding_edges, ppln::collision::Environment<float> *env, float *nodes, int granularity, int num_samples) {
    static constexpr auto dim = Robot::dimension;
    int tid_in_block = threadIdx.x;
    int bid = blockIdx.x;
    // total_threads = num_samples * granularity;
    if (bid >= num_samples) return;
    if (tid_in_block >= granularity) return;

    float *edge_start = &nodes[new_config_parents[bid] * dim];
    float *edge_end = &new_configs[bid * dim];
    float len = device_utils::l2_dist(edge_start, edge_end, dim);


    float delta = len / (float) granularity;

    // calculate the configuration this thread will be checking
    float config[dim];
    for (int i = 0; i < dim; i++) {
        config[i] = edge_start[i] + (tid_in_block * delta);
    }

    // check for collision
    bool config_in_collision = not ppln::collision::fkcc<Robot>(config, env);
    // if (cc_result[bid] == false && config_in_collision) {
    //     atomicAdd(num_colliding_edges, 1);
    //     // printf("collision: %d\n", num_colliding_edges);
    // }
    cc_result[bid] |= config_in_collision;
}


// initialize cuda random
__global__ void init_rng(hiprandState* states, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}

// each thread is responsible for finding a new edge to check
// sample a new state -> connect it to nearest neighbor in our tree
template <typename Robot>
__global__ void sample_edges(float *new_configs, int *new_config_parents, float *nodes, float *goal_configs, int num_goals, hiprandState *rng_states, int num_samples) {
    static constexpr auto dim = Robot::dimension;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_samples) return;
    hiprandState local_rng_state = rng_states[tid];

    float *config = &new_configs[tid * dim];

    // if this is the first thread always sample a random goal
    if (tid == 0) {
        int goal_idx = (int)(num_goals * hiprand_uniform(&local_rng_state));
        float *goal_config = goal_configs + (dim * sizeof(float) * goal_idx);
        for (int i = 0; i < dim; i++) {
            config[i] = goal_config[i];
        }
    }
    // otherwise sample a random config
    else {
        for (int i = 0; i < dim; i++) {
            config[i] = (2 * COORD_BOUND * hiprand_uniform(&local_rng_state)) - COORD_BOUND;
            // printf("%f, ", config[i]);
        }
        // printf("\n");
        rng_states[tid] = local_rng_state;
    }

    // find nearest neighbor
    float min_dist = 1000000000.0;
    int nearest_idx = -1;
    for (int i = 0; i < atomic_free_index; i++) {
        float dist = device_utils::l2_dist(&nodes[i * dim], config, dim);
        if (dist < min_dist) {
            nearest_idx = i;
            min_dist = dist;
        }
    }

    if (tid == 0) {
        printf("%d\n", atomic_free_index);
        printf("%f\n", min_dist);
    }

    // keep it within the rrt range
    if (min_dist > RRT_RADIUS) {
        float scale = RRT_RADIUS / min_dist;
        // printf("min dist: %f\n", min_dist);
        for (int i = 0; i < dim; i++) {
            config[i] *= scale;
        }
    }
    // printf("dist to new cfg: %f\n", l2_dist(config, &nodes[nearest_idx * dim], dim));

    // set the parent of the new config
    new_config_parents[tid] = nearest_idx;
}

// grow the RRT tree after we figure out what edges have no collisions
// each thread is responsible for adding one edge to the tree
template <typename Robot>
__global__ void grow_tree(float *new_configs, int *new_config_parents, bool *cc_result, float *nodes, int *parents, int *num_colliding_edges, float *goal_configs, int num_goals, int num_samples) {
    static constexpr auto dim = Robot::dimension;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_samples) return;
    if (cc_result[tid]) return;  // this edge had a collision, don't add it
    // printf("growing tree!\n");
    // The first edge is always to the goal so if we get here, we need to check if we reached the goal.
    if (tid == 0) {
        int goal_size = dim * sizeof(float);
        for (int i = 0; i < num_goals; i++) {
            float dist_to_goal = device_utils::l2_dist(new_configs, &goal_configs[i * goal_size], dim);
            if (dist_to_goal < 0.001) {
                reached_goal = true;
                found_goal_idx = i;
                return;
            }
        }
    }

    // Atomically get the next free index
    int my_index = atomicAdd(&atomic_free_index, 1);
    
    // Copy the configuration to the nodes array
    for (int i = 0; i < dim; i++) {
        nodes[my_index * dim + i] = new_configs[tid * dim + i];
    }
    
    // Set the parent
    parents[my_index] = new_config_parents[tid];
}

template <typename Robot>
void solve(typename Robot::Configuration &start, std::vector<typename Robot::Configuration> &goals, ppln::collision::Environment<float> &h_environment) {
    static constexpr auto dim = Robot::dimension;
    std::size_t iter = 0;
    std::size_t start_index = 0;
    std::size_t free_index = start_index + 1;

    

    // copy stuff to GPU
    // GPU needs: start, goal, tree, parents, nodes

    float *start_config;
    float *goal_configs;
    int num_goals = goals.size();
    float *nodes;
    int *parents;
    const std::size_t config_size = dim * sizeof(float);
    hipMalloc(&start_config, config_size);
    hipMalloc(&goal_configs, config_size * num_goals);
    hipMalloc(&nodes, MAX_SAMPLES * config_size);
    hipMalloc(&parents, MAX_SAMPLES * sizeof(int));
    hipMemcpy(start_config, start.data(), config_size, hipMemcpyHostToDevice);
    hipMemcpy(goal_configs, goals.data(), config_size, hipMemcpyHostToDevice);
    // add the start config to the tree, and set the start to be it's own parent.
    hipMemcpy(nodes, start.data(), config_size, hipMemcpyHostToDevice);
    hipMemcpy(parents, &start_index, sizeof(int), hipMemcpyHostToDevice);

    // create a hiprandState for each thread -> holds state of RNG for each thread seperately
    // For growing the tree we will create NUM_NEW_CONFIGS threads
    hiprandState *rng_states;
    hipMalloc(&rng_states, NUM_NEW_CONFIGS * sizeof(hiprandState));
    int blockSize = 256;
    int numBlocks = (NUM_NEW_CONFIGS + blockSize - 1) / blockSize;
    init_rng<<<numBlocks, blockSize>>>(rng_states, 0);

    // create arrays on the gpu to hold the newly sampled configs, and their parents
    float *new_configs;
    hipMalloc(&new_configs, NUM_NEW_CONFIGS * config_size);
    int *new_config_parents;
    hipMalloc(&new_config_parents, NUM_NEW_CONFIGS * sizeof(int));

    // create an array to hold the result of collision check for each new edge
    bool *cc_result;
    hipMalloc(&cc_result, NUM_NEW_CONFIGS * sizeof(bool));
    hipMemset(cc_result, 0, NUM_NEW_CONFIGS);
    int *num_colliding_edges;
    hipMalloc(&num_colliding_edges, sizeof(int));

    // free index for next available position in the nodes array
    hipMemcpyToSymbol(HIP_SYMBOL(atomic_free_index), &free_index, sizeof(int));

    // allocate for obstacles
    ppln::collision::Environment<float> *env;
    hipMalloc(&env, sizeof(env));
    hipMemcpy(env, &h_environment, sizeof(env), hipMemcpyHostToDevice);
    // float *obstacles;
    // assert(h_environment.num_spheres.size() % 4 == 0);
    // std::size_t obstacles_size = h_environment.num_spheres * sizeof(float);
    // hipMalloc(&obstacles, obstacles_size);
    // hipMemcpy(obstacles, h_environment.spheres, obstacles_size, hipMemcpyHostToDevice);


    bool done = false;
    // main RRT loop
    while (iter++ < MAX_ITERS && free_index < MAX_SAMPLES) {
        std::cout << iter << std::endl;
        // sample configurations and get edges to be checked
        sample_edges<Robot><<<numBlocks, blockSize>>>(new_configs, new_config_parents, nodes, goal_configs, num_goals, rng_states, NUM_NEW_CONFIGS);

        // collision check all the edges
        hipMemset(cc_result, 0, NUM_NEW_CONFIGS);
        validate_edges<Robot><<<NUM_NEW_CONFIGS, GRANULARITY>>>(new_configs, new_config_parents, cc_result, num_colliding_edges, env, nodes, GRANULARITY, NUM_NEW_CONFIGS);

        // add all the new edges to the tree
        grow_tree<Robot><<<numBlocks, blockSize>>>(new_configs, new_config_parents, cc_result, nodes, parents, num_colliding_edges, goal_configs, num_goals, NUM_NEW_CONFIGS);

        // update free index
        hipMemcpyFromSymbol(&free_index, HIP_SYMBOL(atomic_free_index), sizeof(int), 0, hipMemcpyDeviceToHost);
        hipMemcpyFromSymbol(&done, HIP_SYMBOL(reached_goal), sizeof(bool), 0, hipMemcpyDeviceToHost);
        if (done) break;
    }

    // retrieve data from gpu
    std::vector<int> h_parents(MAX_SAMPLES);
    std::vector<float> h_nodes(MAX_SAMPLES * dim);
    hipMemcpy(h_parents.data(), parents, MAX_SAMPLES * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_nodes.data(), nodes, MAX_SAMPLES * config_size, hipMemcpyDeviceToHost);

    std::vector<int> path;

    if (done) {
        std::cout << "Found Goal!" << std::endl;

        // get parent at position 0 in new_config_parents, because that will be the parent of the goal
        int parent_idx = -1;
        hipMemcpy(&parent_idx, new_config_parents, sizeof(int), hipMemcpyDeviceToHost);
        assert(parent_idx != -1);
        std::cout << parent_idx << std::endl;
        while (parent_idx != h_parents[parent_idx]) {
            std::cout << parent_idx << std::endl;
            path.emplace_back(parent_idx);
            parent_idx = h_parents[parent_idx];
        }
        path.emplace_back(parent_idx);
        std::reverse(path.begin(), path.end());
    }

    hipFree(start_config);
    hipFree(goal_configs);
    hipFree(nodes);
    hipFree(parents);
    hipFree(rng_states);
    hipFree(new_configs);
    hipFree(new_config_parents);
    hipFree(cc_result);
    hipFree(num_colliding_edges);
    hipFree(env);
}

template void solve<ppln::robots::Sphere>(std::array<float, 3>&, std::vector<std::array<float, 3>>&, ppln::collision::Environment<float>&);
template void solve<ppln::robots::Panda>(std::array<float, 7>&, std::vector<std::array<float, 7>>&, ppln::collision::Environment<float>&);