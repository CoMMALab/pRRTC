#include "hip/hip_runtime.h"
#include "Planners.hh"
#include "Robots.hh"
#include "utils.cuh"
#include "pRRTC_settings.hh"
#include "src/collision/environment.hh"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <float.h>

#include <vector>
#include <iostream>
#include <cassert>
#include <algorithm>
#include <numeric>

#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;


/*
Parallelized RRTC: Each block works to add a config to the tree (either start or goal depending on balance)
*/


namespace pRRTC {
    using namespace ppln;
    __device__ volatile int solved = 0;
    __device__ volatile int atomic_free_index[2]; // separate for tree_a and tree_b
    __device__ volatile int nodes_size[2];
    __device__ float path[2][500]; // solution path segments for tree_a, and tree_b
    __device__ int path_size[2] = {0, 0};
    __device__ float cost = 0.0;
    __device__ int reached_goal_idx = 0;
    __device__ int solved_iters = 0; // value of iters in the block that solves the problem
    __constant__ pRRTC_settings d_settings;
    __device__ int print_id = 0;
    // constexpr int MAX_SAMPLES = 1000000;
    // constexpr int MAX_ITERS = 1000000;
    // constexpr int NUM_NEW_CONFIGS = 600;
    constexpr int MAX_GRANULARITY = 256;
    // constexpr float RRT_RADIUS = 0.5;
    // constexpr float TREE_RATIO = 0.5;
    // constexpr bool balance = true;
    // constexpr bool dynamic_domain = true;
    // constexpr float ALPHA = 0.0001;
    // constexpr float dd_RADIUS = 4.0;
    // constexpr float dd_MIN_RADIUS = 1.0;
    // constexpr int NUM_SAMPLE_RETRY = 3;

    constexpr int BLOCK_SIZE = 64;
    constexpr float UNWRITTEN_VAL = -9999.0f;

    // Constants
    // __constant__ float primes[16] = {
    //     3.f, 5.f, 7.f, 11.f, 13.f, 17.f, 19.f, 23.f,
    //     29.f, 31.f, 37.f, 41.f, 43.f, 47.f, 53.f, 59.f
    // };

    template<typename Robot>
    struct HaltonState {
        float b[Robot::dimension];   // bases
        float n[Robot::dimension];   // numerators
        float d[Robot::dimension];   // denominators
    };

    void __device__ shuffle_array(float *array, int n, hiprandState &state) {
        for (int i = n - 1; i > 0; i--) {
            int j = hiprand(&state) % (i + 1);
            float temp = array[i];
            array[i] = array[j];
            array[j] = temp;
        }
    }

    template<typename Robot>
    __device__ void halton_initialize(HaltonState<Robot>& state, size_t skip_iterations, hiprandState& rng_state, int idx) {
        
        float primes[16] = {
            3.f, 5.f, 7.f, 11.f, 13.f, 17.f, 19.f, 23.f,
            29.f, 31.f, 37.f, 41.f, 43.f, 47.f, 53.f, 59.f
        };
        if (idx != 0) shuffle_array(primes, 16, rng_state);
        
        // Initialize bases from primes
        for (size_t i = 0; i < Robot::dimension; i++) {
            state.b[i] = primes[i];
            state.n[i] = 0.0f;
            state.d[i] = 1.0f;
        }
        
        // Skip iterations if requested
        volatile float temp_result[Robot::dimension];
        for (size_t i = 0; i < skip_iterations; i++) {
            halton_next(state, (float *)temp_result);
        }
    }

    template<typename Robot>
    __device__ void halton_next(HaltonState<Robot>& state, float* result) {
        for (size_t i = 0; i < Robot::dimension; i++) {
            float xf = state.d[i] - state.n[i];
            bool x_eq_1 = (xf == 1.0f);
            
            if (x_eq_1) {
                // x == 1 case
                state.d[i] = floorf(state.d[i] * state.b[i]);
                state.n[i] = 1.0f;
            } else {
                // x != 1 case
                float y = floorf(state.d[i] / state.b[i]);
                
                // Continue dividing by b until we find the right digit position
                while (xf <= y) {
                    y = floorf(y / state.b[i]);
                }
                
                state.n[i] = floorf((state.b[i] + 1.0f) * y) - xf;
            }
            
            result[i] = state.n[i] / state.d[i];
        }
    }

    __global__ void init_rng(hiprandState* states, unsigned long seed, int num_rng_states) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= num_rng_states) return;
        hiprand_init(seed + idx, idx, 0, &states[idx]);
    }

    template <typename Robot>
    __global__ void init_halton(HaltonState<Robot>* states, hiprandState* cr_states) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= d_settings.num_new_configs) return;
        // int skip = (hiprand_uniform(&cr_states[idx]) * 50000.0f);
        int skip = 0;
        if (idx == 0) skip = 0;
        halton_initialize(states[idx], skip, cr_states[idx], idx);
    }

    __device__ inline void print_config(volatile float *config, int dim) {
        for (int i = 0; i < dim; i++) {
            printf("%f ,", config[i]);
        }
        printf("\n");
    }

    inline void setup_environment_on_device(ppln::collision::Environment<float> *&d_env, 
                                      const ppln::collision::Environment<float> &h_env) {
        // First allocate the environment struct
        hipMalloc(&d_env, sizeof(ppln::collision::Environment<float>));
        
        // Initialize struct to zeros first
        hipMemset(d_env, 0, sizeof(ppln::collision::Environment<float>));

        // Handle each primitive type separately
        if (h_env.num_spheres > 0) {
            // Allocate and copy spheres array
            ppln::collision::Sphere<float> *d_spheres;
            hipMalloc(&d_spheres, sizeof(ppln::collision::Sphere<float>) * h_env.num_spheres);
            hipMemcpy(d_spheres, h_env.spheres, 
                    sizeof(ppln::collision::Sphere<float>) * h_env.num_spheres, 
                    hipMemcpyHostToDevice);
            
            // Update the struct fields directly
            hipMemcpy(&(d_env->spheres), &d_spheres, sizeof(ppln::collision::Sphere<float>*), 
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_spheres), &h_env.num_spheres, sizeof(unsigned int), 
                    hipMemcpyHostToDevice);
        }

        if (h_env.num_capsules > 0) {
            ppln::collision::Capsule<float> *d_capsules;
            hipMalloc(&d_capsules, sizeof(ppln::collision::Capsule<float>) * h_env.num_capsules);
            hipMemcpy(d_capsules, h_env.capsules,
                    sizeof(ppln::collision::Capsule<float>) * h_env.num_capsules,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->capsules), &d_capsules, sizeof(ppln::collision::Capsule<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_capsules), &h_env.num_capsules, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }

        // Repeat for each primitive type...
        if (h_env.num_z_aligned_capsules > 0) {
            ppln::collision::Capsule<float> *d_z_capsules;
            hipMalloc(&d_z_capsules, sizeof(ppln::collision::Capsule<float>) * h_env.num_z_aligned_capsules);
            hipMemcpy(d_z_capsules, h_env.z_aligned_capsules,
                    sizeof(ppln::collision::Capsule<float>) * h_env.num_z_aligned_capsules,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->z_aligned_capsules), &d_z_capsules, sizeof(ppln::collision::Capsule<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_z_aligned_capsules), &h_env.num_z_aligned_capsules, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }

        if (h_env.num_cylinders > 0) {
            ppln::collision::Cylinder<float> *d_cylinders;
            hipMalloc(&d_cylinders, sizeof(ppln::collision::Cylinder<float>) * h_env.num_cylinders);
            hipMemcpy(d_cylinders, h_env.cylinders,
                    sizeof(ppln::collision::Cylinder<float>) * h_env.num_cylinders,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->cylinders), &d_cylinders, sizeof(ppln::collision::Cylinder<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_cylinders), &h_env.num_cylinders, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }

        if (h_env.num_cuboids > 0) {
            ppln::collision::Cuboid<float> *d_cuboids;
            hipMalloc(&d_cuboids, sizeof(ppln::collision::Cuboid<float>) * h_env.num_cuboids);
            hipMemcpy(d_cuboids, h_env.cuboids,
                    sizeof(ppln::collision::Cuboid<float>) * h_env.num_cuboids,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->cuboids), &d_cuboids, sizeof(ppln::collision::Cuboid<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_cuboids), &h_env.num_cuboids, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }

        if (h_env.num_z_aligned_cuboids > 0) {
            ppln::collision::Cuboid<float> *d_z_cuboids;
            hipMalloc(&d_z_cuboids, sizeof(ppln::collision::Cuboid<float>) * h_env.num_z_aligned_cuboids);
            hipMemcpy(d_z_cuboids, h_env.z_aligned_cuboids,
                    sizeof(ppln::collision::Cuboid<float>) * h_env.num_z_aligned_cuboids,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->z_aligned_cuboids), &d_z_cuboids, sizeof(ppln::collision::Cuboid<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_z_aligned_cuboids), &h_env.num_z_aligned_cuboids, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }
    }


    inline void cleanup_environment_on_device(ppln::collision::Environment<float> *d_env, 
                                        const ppln::collision::Environment<float> &h_env) {
        // Get the pointers from device struct before freeing
        ppln::collision::Sphere<float> *d_spheres = nullptr;
        ppln::collision::Capsule<float> *d_capsules = nullptr;
        ppln::collision::Capsule<float> *d_z_capsules = nullptr;
        ppln::collision::Cylinder<float> *d_cylinders = nullptr;
        ppln::collision::Cuboid<float> *d_cuboids = nullptr;
        ppln::collision::Cuboid<float> *d_z_cuboids = nullptr;

        // Copy each pointer from device memory
        if (h_env.num_spheres > 0) {
            hipMemcpy(&d_spheres, &(d_env->spheres), sizeof(ppln::collision::Sphere<float>*), hipMemcpyDeviceToHost);
            hipFree(d_spheres);
        }
        
        if (h_env.num_capsules > 0) {
            hipMemcpy(&d_capsules, &(d_env->capsules), sizeof(ppln::collision::Capsule<float>*), hipMemcpyDeviceToHost);
            hipFree(d_capsules);
        }
        
        if (h_env.num_z_aligned_capsules > 0) {
            hipMemcpy(&d_z_capsules, &(d_env->z_aligned_capsules), sizeof(ppln::collision::Capsule<float>*), hipMemcpyDeviceToHost);
            hipFree(d_z_capsules);
        }
        
        if (h_env.num_cylinders > 0) {
            hipMemcpy(&d_cylinders, &(d_env->cylinders), sizeof(ppln::collision::Cylinder<float>*), hipMemcpyDeviceToHost);
            hipFree(d_cylinders);
        }
        
        if (h_env.num_cuboids > 0) {
            hipMemcpy(&d_cuboids, &(d_env->cuboids), sizeof(ppln::collision::Cuboid<float>*), hipMemcpyDeviceToHost);
            hipFree(d_cuboids);
        }
        
        if (h_env.num_z_aligned_cuboids > 0) {
            hipMemcpy(&d_z_cuboids, &(d_env->z_aligned_cuboids), sizeof(ppln::collision::Cuboid<float>*), hipMemcpyDeviceToHost);
            hipFree(d_z_cuboids);
        }

        // Finally free the environment struct itself
        hipFree(d_env);
    }

    __global__ void reset_device_variables_kernel() {
        solved = 0;
        
        atomic_free_index[0] = 0;
        atomic_free_index[1] = 0;
        nodes_size[0] = 0;
        nodes_size[1] = 0;
        
        path_size[0] = 0;
        path_size[1] = 0;
        
        for (int tree = 0; tree < 2; tree++) {
            for (int i = 0; i < 500; i++) {
                path[tree][i] = 0.0f;
            }
        }
        
        cost = 0.0f;
        reached_goal_idx = 0;
        }

    void reset_device_variables() {
        reset_device_variables_kernel<<<1, 1>>>();
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(error));
        }
    }

    __device__ __forceinline__ bool blockAnyTrue(bool localValue, int tid, int warpId, int laneId) {
        __shared__ bool shared[16];
        bool warpResult = __any_sync(0xffffffff, localValue);
        if (laneId == 0) shared[warpId] = warpResult;
        __syncthreads();
        if (warpId == 0) {
            bool blockResult = false;
            if (laneId < (blockDim.x + warpSize - 1) / warpSize) blockResult = shared[laneId];
            blockResult = __any_sync(0xffffffff, blockResult);
            if (laneId == 0) shared[0] = blockResult;
        }
        __syncthreads();
        return shared[0];
    }

    __device__ __forceinline__ bool check_partially_written(volatile float *node, int dim) {
        #pragma unroll
        for (int i = 0; i < dim; i++) {
            if (node[i] == UNWRITTEN_VAL) return true;
        }
        return false;
    }
    
    template <typename Robot>
    __global__ void
    __launch_bounds__(128, 8)
    rrtc(
        float **nodes,
        int **parents,
        float **radii,
        HaltonState<Robot> *halton_states,
        hiprandState *rng_states,
        ppln::collision::Environment<float> *env
    )
    {
        cg::grid_group grid = cg::this_grid();
        static constexpr auto dim = Robot::dimension;
        const int tid = threadIdx.x;
        const int bid = blockIdx.x; // 0 ... NUM_NEW_CONFIGS
        // const int lid = threadIdx.x%32;
        // const int wid = threadIdx.x/32;
        __shared__ int t_tree_id; // this tree
        __shared__ int o_tree_id; // the other tree
        __shared__ volatile float config[dim];
        __shared__ volatile float sdata[MAX_GRANULARITY];
        __shared__ volatile int sindex[MAX_GRANULARITY];
        __shared__ volatile unsigned int local_cc_result[1];
        __shared__ volatile float *t_nodes;
        __shared__ volatile float *o_nodes;
        __shared__ volatile int *t_parents;
        __shared__ volatile int *o_parents;
        __shared__ float scale;
        __shared__ volatile float *nearest_node;
        __shared__ float delta[dim];
        // __shared__ float var_cache[MAX_GRANULARITY][10];
        __shared__ volatile int index;
        __shared__ volatile float vec[dim];
        __shared__ unsigned int n_extensions;
        __shared__ bool should_skip;

        int iter = 0;

        while (true) {
            // if (solved != 0) return;
            // if (tid < dim) {
            //     config[tid] = hiprand_uniform(&rng_states[bid * dim + tid]);
            // }
            // __syncthreads();
            if (tid == 0) {
                // printf("iter: %d, bid: %d\n", iter, bid);
                iter++;
                if (iter > d_settings.max_iters) {
                    // printf("max iters reached from bid: %d\n", bid);
                    atomicCAS((int *)&solved, 0, -1);
                }

                if (d_settings.balance == 0 || iter == 1) {
                    t_tree_id = (bid < (d_settings.num_new_configs / 2))? 0 : 1;
                    o_tree_id = 1 - t_tree_id;
                }
                else if (d_settings.balance == 1 && abs(atomic_free_index[0]-atomic_free_index[1]) < 1.5 * d_settings.num_new_configs){
                    float ratio = atomic_free_index[0] / (float)(atomic_free_index[0]+atomic_free_index[1]);
                    float balance_factor = 1 - ratio;
                    t_tree_id = (bid < (d_settings.num_new_configs * balance_factor))? 0 : 1;
                    o_tree_id = 1 - t_tree_id;
                }
                else if (d_settings.balance == 1) {
                    float ratio = atomic_free_index[0] / (float)(atomic_free_index[0] + atomic_free_index[1]);
                    if (ratio < d_settings.tree_ratio) t_tree_id = 0;
                    else t_tree_id = 1;
                    o_tree_id = 1 - t_tree_id;
                }
                else if (d_settings.balance == 2) { // vamp balance algo
                    float ratio = abs(atomic_free_index[t_tree_id] - atomic_free_index[o_tree_id]) / (float) atomic_free_index[t_tree_id];
                    if (ratio < d_settings.tree_ratio)
                    {
                        t_tree_id = 1 - t_tree_id;
                        o_tree_id = 1 - t_tree_id;
                    }
                }

                t_nodes = nodes[t_tree_id];
                o_nodes = nodes[o_tree_id];
                t_parents = parents[t_tree_id];
                o_parents = parents[o_tree_id];
                
                halton_next(halton_states[bid], (float *)config);
                Robot::scale_cfg((float *)config);
                local_cc_result[0] = 0;
            }
            __syncthreads();
            // grid.sync();

            // if (solved != 0) {
            //     // if (tid == 0) printf("Exiting because solved = %d\n, bid: %d, iter: %d", solved, bid, iter);
            //     return;
            // }

            // divide up the work of finding nearest neighbor among the threads
            float local_min_dist = FLT_MAX;
            int local_near_idx = 0;
            float dist;
            // int size = nodes_size[t_tree_id];
            int size = atomic_free_index[t_tree_id];
            for (int i = tid; i < size; i += blockDim.x) {
                while (check_partially_written(&t_nodes[i * dim], dim)) {};
                dist = device_utils::sq_l2_dist((float *)&t_nodes[i * dim], (float *) config, dim);
                if (dist < local_min_dist) {
                    local_min_dist = dist;
                    local_near_idx = i;
                }
            }
            sdata[tid] = local_min_dist;
            sindex[tid] = local_near_idx;
            __syncthreads();

            for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
                if (tid < s){
                    if (sdata[tid + s] < sdata[tid]) {
                        sdata[tid] = sdata[tid + s];
                        sindex[tid] = sindex[tid + s];
                    }
                }
                __syncthreads();
            }

            // by this point NN dist = sdata[0], NN index = sindex[0]
            // now calculate the extension
            volatile float nearest_node_copy[dim];
            if (tid == 0) {
                sdata[0] = sqrt(sdata[0]);
                scale = min(1.0f, d_settings.range / (sdata[0]));
                nearest_node = &t_nodes[sindex[0] * dim];
                for (int i = 0; i < dim; i++) {
                    nearest_node_copy[i] = nearest_node[i];
                }
                // printf("nearest node copy: %f %f %f %f %f %f %f %f\n sindex[0]: %d, t_tree_id: %d\n",
                //     nearest_node_copy[0], nearest_node_copy[1], nearest_node_copy[2], nearest_node_copy[3], nearest_node_copy[4], nearest_node_copy[5], nearest_node_copy[6], nearest_node_copy[7]
                //     , sindex[0], t_tree_id
                // );
                should_skip = (d_settings.dynamic_domain && radii[t_tree_id][sindex[0]] < sdata[0]);
                // printf("radius: %f, dist: %f\n, should_skip: %d", radii[t_tree_id][sindex[0]], sdata[0], should_skip);
                // printf("calculate extension\n");
            }
            __syncthreads();

            if (should_skip) {
                // if (tid == 0) printf("skipping iter %d\n", iter);
                continue;
            }
            __syncthreads();

            if (tid < dim) {
                config[tid] = nearest_node[tid] + ((config[tid] - nearest_node[tid]) * scale);
                delta[tid] = (config[tid] - nearest_node[tid]) / (float) d_settings.granularity;
            }
            __syncthreads();

            // if (solved != 0) return;
            
            /* validate_edges */
            float interp_cfg[dim];
            for (int i = 0; i < dim; i++) {
                interp_cfg[i] = nearest_node[i] + ((tid + 1) * delta[i]);
            }
            // __shared__ bool each_thread_result[128];
            bool config_in_collision = not ppln::collision::fkcc<Robot>(interp_cfg, env, tid);
            // each_thread_result[tid] = config_in_collision;
            // __syncthreads();
            // bool edge_good = not blockAnyTrue(config_in_collision, tid, wid, lid);
            atomicOr((unsigned int *)&local_cc_result[0], config_in_collision ? 1u : 0u);
            __syncthreads();
            bool edge_good = local_cc_result[0] == 0;
            // bool repeated_node = (sdata[0]==0);
            // grid.sync();

            // if (tid == 0) {
            //     // check edge with loop to verify
            //     // printf("env debug: num_spheres: %d, num_capsules: %d, num_cylinders: %d, num_cuboids: %d\n", env->num_spheres, env->num_capsules, env->num_cylinders, env->num_cuboids);
            //     bool debug_collision = false;
            //     float debug_delta[dim];
            //     float debug_interp_cfg[dim];
            //     for (int i = 0; i < dim; i++) {
            //         debug_delta[i] = (config[i] - nearest_node[i]) / (float) d_settings.granularity;
            //     }
            //     for (int i = 0; i < d_settings.granularity; i++) {
            //         for (int j = 0; j < dim; j++) {
            //             debug_interp_cfg[j] = nearest_node[j] + ((i + 1) * debug_delta[j]);
            //         }
            //         config_in_collision = not ppln::collision::fkcc<Robot>(debug_interp_cfg, env, tid);
            //         if (config_in_collision) {
            //             debug_collision = true;
            //             break;
            //         }
            //     }

            //     // bool debug_collision_2 = false;
            //     // for (int i = 0; i < 128; i++) {
            //     //     if (each_thread_result[i]) {
            //     //         debug_collision_2 = true;
            //     //         break;
            //     //     }
            //     // }

            //     // int l_print_id = atomicAdd(&print_id, 1);
            //     // printf("%d checked edge between: %f %f %f %f %f %f %f %f\n and %f %f %f %f %f %f %f %f. Result: %d, Debug1: %d, Debug2: %d, sindex[0]: %d\n, Nearest Node copy: %f %f %f %f %f %f %f %f\n", 
            //     //     l_print_id,
            //     //     nearest_node[0], nearest_node[1], nearest_node[2], nearest_node[3], nearest_node[4], nearest_node[5], nearest_node[6], nearest_node[7],
            //     //     config[0], config[1], config[2], config[3], config[4], config[5], config[6], config[7], edge_good, !debug_collision, !debug_collision_2, sindex[0],
            //     //     nearest_node_copy[0], nearest_node_copy[1], nearest_node_copy[2], nearest_node_copy[3], nearest_node_copy[4], nearest_node_copy[5], nearest_node_copy[6], nearest_node_copy[7]
            //     // );
                
            //     // printf("%d Debug delta: %f %f %f %f %f %f %f %f\n %f %f %f %f %f %f %f %f\n",
            //     // l_print_id,
            //     // delta[0], delta[1], delta[2], delta[3], delta[4], delta[5], delta[6], delta[7],
            //     // debug_delta[0], debug_delta[1], debug_delta[2], debug_delta[3], debug_delta[4], debug_delta[5], debug_delta[6], debug_delta[7]
            //     // );
                
            // }
            __syncthreads();
            if (edge_good) {
                /* grow tree */
                if (tid == 0) {
                    
                    index = atomicAdd((int *)&atomic_free_index[t_tree_id], 1);
                    if (index >= d_settings.max_samples) solved = -1;
                    
                    t_parents[index] = sindex[0];
                    
                    if (d_settings.dynamic_domain) {
                        radii[t_tree_id][index] = FLT_MAX;
                        volatile float *radius_ptr = &radii[t_tree_id][sindex[0]];
                        float old_radius, new_radius;
                        int expected, desired;
                        do {
                            // printf("dynamic domain loop 1\n");
                            old_radius = *radius_ptr;
                            if (old_radius == FLT_MAX) break;
                            new_radius = old_radius * (1 + d_settings.dd_alpha);
                            expected = __float_as_int(old_radius);
                            desired = __float_as_int(new_radius);
                        } while (atomicCAS((int *)radius_ptr, expected, desired) != expected);
                    }
                }
                __syncthreads();

                if (tid < dim) {
                    t_nodes[index * dim + tid] = config[tid];
                }
                __syncthreads();
                __threadfence_system();
            }
            // grid.sync();
            if (edge_good) {
                // if (tid == 0) {
                //     atomicAdd((int *)&nodes_size[t_tree_id], 1);
                //     // printf("added config to tree %d at index %d (bid %d, parent %d): %f %f %f %f %f %f %f %f\n parent: %f %f %f %f %f %f %f %f\n", 
                //     //         t_tree_id, index, bid, sindex[0], config[0], config[1], config[2], config[3], config[4], config[5], config[6], config[7],
                //     //         nearest_node[0], nearest_node[1], nearest_node[2], nearest_node[3], nearest_node[4], nearest_node[5], nearest_node[6], nearest_node[7]
                //     //     );
                // }
                // __syncthreads();
                /* connect */
                local_min_dist = INFINITY;
                local_near_idx = 0;
                // int size = nodes_size[o_tree_id];
                int size = atomic_free_index[o_tree_id];
                for (unsigned int i = tid; i < size; i += blockDim.x) {
                    while (check_partially_written(&o_nodes[i * dim], dim)) {};
                    dist = device_utils::sq_l2_dist((float *)&o_nodes[i * dim], (float *)config, dim);
                    if (dist < local_min_dist) {
                        local_min_dist = dist;
                        local_near_idx = i;
                    }
                }
                sdata[tid] = local_min_dist;
                sindex[tid] = local_near_idx;
                __syncthreads();
                
                for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
                    if (tid < s) {
                        if (sdata[tid + s] < sdata[tid]) {
                            sdata[tid] = sdata[tid + s];
                            sindex[tid] = sindex[tid + s];
                        }
                    }
                    __syncthreads();
                }
                
                
                if (tid == 0) {
                    sdata[0] = sqrt(sdata[0]);
                    nearest_node = &o_nodes[sindex[0] * dim];
                    n_extensions = ceil(sdata[0] / d_settings.range);
                    local_cc_result[0] = 0;
                    // printf("found closest in other tree\n");
                }
                __syncthreads();

                if (tid < dim) {
                    vec[tid] = (nearest_node[tid] - config[tid]) / (float) n_extensions;
                }
                __syncthreads();
            }
            // grid.sync();
            if (edge_good) {
                // validate the edge to the nearest neighbor in opposite tree, go as far as we can
                int i_extensions = 0;
                int extension_parent_idx = index;
                // printf("here6\n");
                while (i_extensions < n_extensions) {
                    /* each thread checking an interpolated config along the extension vector*/
                    for (int i = 0; i < dim; i++) {
                        interp_cfg[i] = config[i] + ((tid + 1) * (vec[i] / (float) d_settings.granularity));
                    }
                    __syncthreads();
                    bool config_in_collision = not ppln::collision::fkcc<Robot>(interp_cfg, env, tid);
                    // bool edge_good = not blockAnyTrue(config_in_collision, tid, wid, lid);
                    atomicOr((unsigned int *)&local_cc_result[0], config_in_collision ? 1u : 0u);
                    __syncthreads();
                    bool ext_edge_good = local_cc_result[0] == 0;
                    if (!ext_edge_good) break;
                    // if (local_cc_result[0] != 0) break;
                    /* add extension to tree */
                    if (tid == 0) {
                        index = atomicAdd((int *)&atomic_free_index[t_tree_id], 1);
                        if (index >= d_settings.max_samples) solved = -1;
                        t_parents[index] = extension_parent_idx;
                        radii[t_tree_id][index] = FLT_MAX;
                        extension_parent_idx = index;
                        local_cc_result[0] = 0;
                        // printf("in extension loop\n");
                    }
                    __syncthreads();
                    if (tid < dim) {
                        config[tid] = config[tid] + vec[tid];
                        t_nodes[index * dim + tid] = config[tid];
                    }
                    __syncthreads();
                    __threadfence_system();
                    // if (tid == 0) {
                    //     atomicAdd((int *)&nodes_size[t_tree_id], 1);
                    //     // printf("added config from extension to tree %d at index %d (bid %d, parent %d): %f %f %f %f %f %f %f %f\n", t_tree_id, index, bid, t_parents[index], config[0], config[1], config[2], config[3], config[4], config[5], config[6], config[7]);
                    //     // print_config(config, dim);
                    // }
                    i_extensions++;
                    __syncthreads();
                }
                if (i_extensions == n_extensions) { // connected
                    if (tid == 0 && atomicCAS((int *)&solved, 0, 1) == 0) {
                        // printf("in connected\n");
                        // printf("config at connection: %f %f %f %f %f %f %f %f\n nearest node: %f %f %f %f %f %f %f %f\n", 
                        //     config[0], config[1], config[2], config[3], config[4], config[5], config[6], config[7],
                        //     nearest_node[0], nearest_node[1], nearest_node[2], nearest_node[3], nearest_node[4], nearest_node[5], nearest_node[6], nearest_node[7]
                        // );
                        // trace back to the start and goal.
                        int current = index;
                        int parent;
                        int t_path_size = 0;
                        int o_path_size = 0;
                        while (t_parents[current] != current) {
                            parent = t_parents[current];
                            cost += device_utils::l2_dist((float *)&t_nodes[current * dim], (float *)&t_nodes[parent * dim], dim);
                            for (int i = 0; i < dim; i++) path[t_tree_id][t_path_size * dim + i] = t_nodes[current * dim + i];
                            // printf("added to path[%d]: %f %f %f %f %f %f %f %f\n", t_tree_id, path[t_tree_id][t_path_size * dim], path[t_tree_id][t_path_size * dim + 1], path[t_tree_id][t_path_size * dim + 2], path[t_tree_id][t_path_size * dim + 3], path[t_tree_id][t_path_size * dim + 4], path[t_tree_id][t_path_size * dim + 5], path[t_tree_id][t_path_size * dim + 6], path[t_tree_id][t_path_size * dim + 7]);
                            //print_config(&t_nodes[current * dim], dim);
                            t_path_size++;
                            current = parent;
                            
                        }
                        //printf("GPU path above");
                        if (t_tree_id == 1) reached_goal_idx = current;
                        current = sindex[0];
                        // printf("sindex[0]: %d\n", sindex[0]);
                        // printf("entered here2\n");
                        while(o_parents[current] != current) {
                            parent = o_parents[current];
                            cost += device_utils::l2_dist((float *)&o_nodes[current * dim], (float *)&o_nodes[parent * dim], dim);
                            for (int i = 0; i < dim; i++) path[o_tree_id][o_path_size * dim + i] = o_nodes[current * dim + i];
                            // printf("added to path[%d]: %f %f %f %f %f %f %f %f\n", o_tree_id, path[o_tree_id][o_path_size * dim], path[o_tree_id][o_path_size * dim + 1], path[o_tree_id][o_path_size * dim + 2], path[o_tree_id][o_path_size * dim + 3], path[o_tree_id][o_path_size * dim + 4], path[o_tree_id][o_path_size * dim + 5], path[o_tree_id][o_path_size * dim + 6], path[o_tree_id][o_path_size * dim + 7]);
                            //print_config(&o_nodes[current * dim], dim);
                            o_path_size++;
                            current = parent;
                        }
                        //printf("GPU path above 2");
                        if (t_tree_id == 0) reached_goal_idx = current;
                        path_size[t_tree_id] = t_path_size;
                        path_size[o_tree_id] = o_path_size;
                        solved_iters = iter;
                        
                        // printf("t_tree_id %d, t_path_size: %d, o_path_size: %d\n", t_tree_id, t_path_size, o_path_size);
                        // return;
                    }
                    __syncthreads();
                }
                // printf("here8\n");
            }
            else if (d_settings.dynamic_domain && tid == 0) {         
                volatile float *radius_ptr = &radii[t_tree_id][sindex[0]];
                float old_radius, new_radius;
                int expected, desired;
                do {
                    // printf("in d settings end\n");
                    old_radius = *radius_ptr;
                    if (old_radius == FLT_MAX) {
                        new_radius = d_settings.dd_radius;
                    } else {
                        new_radius = fmaxf(old_radius * (1.f - d_settings.dd_alpha), d_settings.dd_min_radius);
                    }
                    expected = __float_as_int(old_radius);
                    desired = __float_as_int(new_radius);
                } while (atomicCAS((int *)radius_ptr, expected, desired) != expected);
            }
            __syncthreads();
            // grid.sync();
            if (solved != 0) return;
        }
    }




    template <typename Robot>
    PlannerResult<Robot> solve(
        typename Robot::Configuration &start,
        std::vector<typename Robot::Configuration> &goals,
        ppln::collision::Environment<float> &h_environment,
        pRRTC_settings &settings
    ) 
    {
        // std::cout << "here" << std::endl;
        hipSetDevice(1);
        // std::cout << "here1" << std::endl;
        auto start_time = std::chrono::steady_clock::now();
        static constexpr auto dim = Robot::dimension;
        // std::size_t iter = 0;
        std::size_t start_index = 0;

        
        PlannerResult<Robot> res;
        // copy data to GPU
        hipMemcpyToSymbol(HIP_SYMBOL(d_settings), &settings, sizeof(settings));

        // std::cout << "here2" << std::endl;
        int num_goals = goals.size();
        float *nodes[2];
        int *parents[2];
        float *radii[2];
        float **d_nodes;
        int **d_parents;
        float **d_radii;
        hipMalloc(&d_nodes, 2 * sizeof(float*));
        hipMalloc(&d_parents, 2 * sizeof(int*));
        hipMalloc(&d_radii, 2 * sizeof(float*));
        const std::size_t config_size = dim * sizeof(float);

        for (int i = 0; i < 2; i++) {
            hipMalloc(&nodes[i], settings.max_samples * config_size);
            hipMalloc(&parents[i], settings.max_samples * sizeof(int));
            hipMalloc(&radii[i], settings.max_samples * sizeof(float));
        }
        hipMemcpy(d_nodes, nodes, 2 * sizeof(float*), hipMemcpyHostToDevice);
        hipMemcpy(d_parents, parents, 2 * sizeof(int*), hipMemcpyHostToDevice);
        hipMemcpy(d_radii, radii, 2 * sizeof(float*), hipMemcpyHostToDevice);

        // set nodes to unitialized
        std::vector<float> nodes_init(settings.max_samples * dim, UNWRITTEN_VAL);
        hipMemcpy((void *)nodes[0], nodes_init.data(), config_size * settings.max_samples, hipMemcpyHostToDevice);
        hipMemcpy((void *)nodes[1], nodes_init.data(), config_size * settings.max_samples, hipMemcpyHostToDevice);
        // add start to tree_a and goals to tree_b
        hipMemcpy((void *)nodes[0], start.data(), config_size, hipMemcpyHostToDevice);
        hipMemcpy((void *)parents[0], &start_index, sizeof(int), hipMemcpyHostToDevice);

        hipMemcpy((void *)nodes[1], goals.data(), config_size * num_goals, hipMemcpyHostToDevice);
        std::vector<int> parents_b_init(num_goals);
        iota(parents_b_init.begin(), parents_b_init.end(), 0); // consecutive integers from 0 ... num_goals - 1
        hipMemcpy((void *)parents[1], parents_b_init.data(), sizeof(int) * num_goals, hipMemcpyHostToDevice);

        // initialize radii
        std::vector<float> radii_init(num_goals, FLT_MAX);
        hipMemcpy((void *)radii[0], radii_init.data(), sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy((void *)radii[1], radii_init.data(), sizeof(float) * num_goals, hipMemcpyHostToDevice);
        // std::cout << "here4" << std::endl;
        // create a hiprandState for each thread -> holds state of RNG for each thread seperately
        // For growing the tree we will create NUM_NEW_CONFIGS threads
        hiprandState *rng_states;
        int num_rng_states = settings.num_new_configs * dim;
        hipMalloc(&rng_states, num_rng_states * sizeof(hiprandState));
        int numBlocks = (num_rng_states + BLOCK_SIZE - 1) / BLOCK_SIZE;
        init_rng<<<numBlocks, BLOCK_SIZE>>>(rng_states, 1, num_rng_states);

        HaltonState<Robot> *halton_states;
        hipMalloc(&halton_states, settings.num_new_configs * sizeof(HaltonState<Robot>));
        int numBlocks1 = (settings.num_new_configs + BLOCK_SIZE - 1) / BLOCK_SIZE;
        init_halton<Robot><<<numBlocks1, BLOCK_SIZE>>>(halton_states, rng_states);

        // free index for next available position in tree_a and tree_b
        int h_free_index[2] = {1, num_goals};
        hipMemcpyToSymbol(HIP_SYMBOL(atomic_free_index), &h_free_index, sizeof(int) * 2);
        hipMemcpyToSymbol(HIP_SYMBOL(nodes_size), &h_free_index, sizeof(int) * 2);
        // std::cout << "here5" << std::endl;
        // allocate for obstacles
        ppln::collision::Environment<float> *env;
        setup_environment_on_device(env, h_environment);
        // std::cout << "here6" << std::endl;
        cudaCheckError(hipGetLastError());
        // Setup pinned memory for signaling
        int *h_solved;
        int current_samples[2];
        int h_solved_iters = -1;
        hipHostMalloc(&h_solved, sizeof(int));  // Pinned memory
        *h_solved = -1;


        auto kernel_start_time = std::chrono::steady_clock::now();
        rrtc<Robot><<<settings.num_new_configs, settings.granularity>>> (
            d_nodes,
            d_parents,
            d_radii,
            halton_states,
            rng_states,
            env
        );
        hipDeviceSynchronize();
        res.kernel_ns = get_elapsed_nanoseconds(kernel_start_time);
        cudaCheckError(hipGetLastError());

        // void* kernelArgs[] = {
        //     (void*)&d_nodes,
        //     (void*)&d_parents,
        //     (void*)&d_radii,
        //     (void*)&halton_states,
        //     (void*)&rng_states,
        //     (void*)&env
        // };
        // auto kernel_start_time = std::chrono::steady_clock::now();
        // hipError_t err = hipLaunchCooperativeKernel(
        //     (void*)rrtc<Robot>,  // Kernel function
        //     settings.num_new_configs, settings.granularity,  // Grid and block dimensions
        //     kernelArgs,  // Kernel arguments
        //     0  // Shared memory per block (set to 0 so the compiler and auto compute)
        // );
        // if (err != hipSuccess) {
        //     std::cerr << "CUDA Kernel launch failed: " << hipGetErrorString(err) << "\n";
        // }
        // hipDeviceSynchronize();
        // cudaCheckError(hipGetLastError());
        // res.kernel_ns = get_elapsed_nanoseconds(kernel_start_time);
        
        
        hipMemcpyFromSymbol(current_samples, HIP_SYMBOL(atomic_free_index), sizeof(int) * 2, 0, hipMemcpyDeviceToHost);
        // hipMemcpyFromSymbol(current_samples, HIP_SYMBOL(nodes_size), sizeof(int) * 2, 0, hipMemcpyDeviceToHost);
        hipMemcpyFromSymbol(h_solved, HIP_SYMBOL(solved), sizeof(int), 0, hipMemcpyDeviceToHost);
        hipMemcpyFromSymbol(&h_solved_iters, HIP_SYMBOL(solved_iters), sizeof(int), 0, hipMemcpyDeviceToHost);

        // currently, iteration count is not copied because each block may have different iteration count
        if (*h_solved!=1) *h_solved=0;
        std::cout << "current_samples: start: " << current_samples[0] << ", goal: " << current_samples[1] << "\n";
        std::cout << "solved iters: " << h_solved_iters << "\n";
        res.start_tree_size = current_samples[0];
        res.goal_tree_size = current_samples[1];
        Robot::print_robot_config(start);
        Robot::print_robot_config(goals[0]);
        if (*h_solved) {
            std::cout << "solved!\n";
            int h_path_size[2];
            float h_paths[2][500];
            float h_cost;
            int h_reached_goal_idx;
            hipMemcpyFromSymbol(h_path_size, HIP_SYMBOL(path_size), sizeof(int) * 2, 0, hipMemcpyDeviceToHost);
            hipMemcpyFromSymbol(h_paths, HIP_SYMBOL(path), sizeof(float) * 2 * 500, 0, hipMemcpyDeviceToHost);
            hipMemcpyFromSymbol(&h_cost, HIP_SYMBOL(cost), sizeof(float), 0, hipMemcpyDeviceToHost);
            hipMemcpyFromSymbol(&h_reached_goal_idx, HIP_SYMBOL(reached_goal_idx), sizeof(int), 0, hipMemcpyDeviceToHost);
            cudaCheckError(hipGetLastError());
            // Robot::print_robot_config(goals[h_reached_goal_idx]);
            res.path.emplace_back(goals[h_reached_goal_idx]);
            typename Robot::Configuration config;
            for (int i = h_path_size[1] - 1; i >= 0; i--) {
                std::copy_n(h_paths[1] + i * dim, dim, config.begin());
                res.path.emplace_back(config);
                // print_cfg_ptr<Robot>(&h_paths[1][i * dim]);
            }
            for (int i = 0; i < h_path_size[0]; i++) {
                std::copy_n(h_paths[0] + i * dim, dim, config.begin());
                res.path.emplace_back(config);
                // print_cfg_ptr<Robot>(&h_paths[0][i * dim]);
            }
            // Robot::print_robot_config(start);
            res.path.emplace_back(start);
            res.cost = h_cost;
            res.path_length = (h_path_size[0] + h_path_size[1]);
            std::cout << "cost: " << res.cost << "\n";
        }
        res.solved = (*h_solved) != 0;
        res.iters = h_solved_iters;
        
        cleanup_environment_on_device(env, h_environment);
        reset_device_variables();
        hipFree((void *)nodes[0]);
        hipFree((void *)nodes[1]);
        hipFree((void *)parents[0]);
        hipFree((void *)parents[1]);
        hipFree((void *)radii[0]);
        hipFree((void *)radii[1]);
        hipFree(rng_states);
        hipFree(halton_states);
        hipFree(d_nodes);
        hipFree(d_parents);
        hipFree(d_radii);
        hipHostFree(h_solved);
        cudaCheckError(hipGetLastError());
        res.wall_ns = get_elapsed_nanoseconds(start_time);
        hipDeviceReset();
        return res;
    }

    template PlannerResult<typename ppln::robots::Sphere> solve<ppln::robots::Sphere>(std::array<float, 3>&, std::vector<std::array<float, 3>>&, ppln::collision::Environment<float>&, pRRTC_settings&);
    template PlannerResult<typename ppln::robots::Panda> solve<ppln::robots::Panda>(std::array<float, 7>&, std::vector<std::array<float, 7>>&, ppln::collision::Environment<float>&, pRRTC_settings&);
    template PlannerResult<typename ppln::robots::Fetch> solve<ppln::robots::Fetch>(std::array<float, 8>&, std::vector<std::array<float, 8>>&, ppln::collision::Environment<float>&, pRRTC_settings&);
}
