#include "hip/hip_runtime.h"
#include "Planners.hh"
#include "Robots.hh"
#include "utils.cuh"
#include "pRRTC_settings.hh"
#include "src/collision/environment.hh"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <float.h>

#include <vector>
#include <iostream>
#include <cassert>
#include <algorithm>
#include <numeric>

/*
Parallelized RRTC: Each block works to add a config to the tree (either start or goal depending on balance)
*/


namespace pRRTC {
    using namespace ppln;
    __device__ volatile int solved = 0;
    __device__ volatile int atomic_free_index[2]; // separate for tree_a and tree_b
    __device__ float path[2][500]; // solution path segments for tree_a, and tree_b
    __device__ int path_size[2] = {0, 0};
    __device__ float cost = 0.0;
    __device__ int reached_goal_idx = 0;
    __constant__ pRRTC_settings d_settings;

    // constexpr int MAX_SAMPLES = 1000000;
    // constexpr int MAX_ITERS = 1000000;
    // constexpr int NUM_NEW_CONFIGS = 600;
    constexpr int MAX_GRANULARITY = 256;
    // constexpr float RRT_RADIUS = 0.5;
    // constexpr float TREE_RATIO = 0.5;
    // constexpr bool balance = true;
    // constexpr bool dynamic_domain = true;
    // constexpr float ALPHA = 0.0001;
    // constexpr float dd_RADIUS = 4.0;
    // constexpr float dd_MIN_RADIUS = 1.0;
    // constexpr int NUM_SAMPLE_RETRY = 3;

    constexpr int BLOCK_SIZE = 64;

    // Constants
    __constant__ float primes[16] = {
        3.f, 5.f, 7.f, 11.f, 13.f, 17.f, 19.f, 23.f,
        29.f, 31.f, 37.f, 41.f, 43.f, 47.f, 53.f, 59.f
    };

    template<typename Robot>
    struct HaltonState {
        float b[Robot::dimension];   // bases
        float n[Robot::dimension];   // numerators
        float d[Robot::dimension];   // denominators
    };

    template<typename Robot>
    __device__ void halton_initialize(HaltonState<Robot>& state, size_t skip_iterations) {
        // Initialize bases from primes
        for (size_t i = 0; i < Robot::dimension; i++) {
            state.b[i] = primes[i];
            state.n[i] = 0.0f;
            state.d[i] = 1.0f;
        }
        
        // Skip iterations if requested
        volatile float temp_result[Robot::dimension];
        for (size_t i = 0; i < skip_iterations; i++) {
            halton_next(state, (float *)temp_result);
        }
    }

    template<typename Robot>
    __device__ void halton_next(HaltonState<Robot>& state, float* result) {
        for (size_t i = 0; i < Robot::dimension; i++) {
            float xf = state.d[i] - state.n[i];
            bool x_eq_1 = (xf == 1.0f);
            
            if (x_eq_1) {
                // x == 1 case
                state.d[i] = floorf(state.d[i] * state.b[i]);
                state.n[i] = 1.0f;
            } else {
                // x != 1 case
                float y = floorf(state.d[i] / state.b[i]);
                
                // Continue dividing by b until we find the right digit position
                while (xf <= y) {
                    y = floorf(y / state.b[i]);
                }
                
                state.n[i] = floorf((state.b[i] + 1.0f) * y) - xf;
            }
            
            result[i] = state.n[i] / state.d[i];
        }
    }

    __global__ void init_rng(hiprandState* states, unsigned long seed, int num_rng_states) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= num_rng_states) return;
        hiprand_init(seed + idx, idx, 0, &states[idx]);
    }

    template <typename Robot>
    __global__ void init_halton(HaltonState<Robot>* states, hiprandState* cr_states) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= d_settings.num_new_configs) return;
        int skip = (hiprand_uniform(&cr_states[idx]) * 50000.0f);
        if (idx == 0) skip = 0;
        if (idx == 1) skip = 100000;
        halton_initialize(states[idx], skip);
    }

    __device__ inline void print_config(float *config, int dim) {
        for (int i = 0; i < dim; i++) {
            printf("%f ,", config[i]);
        }
        printf("\n");
    }

    inline void setup_environment_on_device(ppln::collision::Environment<float> *&d_env, 
                                      const ppln::collision::Environment<float> &h_env) {
        // First allocate the environment struct
        hipMalloc(&d_env, sizeof(ppln::collision::Environment<float>));
        
        // Initialize struct to zeros first
        hipMemset(d_env, 0, sizeof(ppln::collision::Environment<float>));

        // Handle each primitive type separately
        if (h_env.num_spheres > 0) {
            // Allocate and copy spheres array
            ppln::collision::Sphere<float> *d_spheres;
            hipMalloc(&d_spheres, sizeof(ppln::collision::Sphere<float>) * h_env.num_spheres);
            hipMemcpy(d_spheres, h_env.spheres, 
                    sizeof(ppln::collision::Sphere<float>) * h_env.num_spheres, 
                    hipMemcpyHostToDevice);
            
            // Update the struct fields directly
            hipMemcpy(&(d_env->spheres), &d_spheres, sizeof(ppln::collision::Sphere<float>*), 
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_spheres), &h_env.num_spheres, sizeof(unsigned int), 
                    hipMemcpyHostToDevice);
        }

        if (h_env.num_capsules > 0) {
            ppln::collision::Capsule<float> *d_capsules;
            hipMalloc(&d_capsules, sizeof(ppln::collision::Capsule<float>) * h_env.num_capsules);
            hipMemcpy(d_capsules, h_env.capsules,
                    sizeof(ppln::collision::Capsule<float>) * h_env.num_capsules,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->capsules), &d_capsules, sizeof(ppln::collision::Capsule<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_capsules), &h_env.num_capsules, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }

        // Repeat for each primitive type...
        if (h_env.num_z_aligned_capsules > 0) {
            ppln::collision::Capsule<float> *d_z_capsules;
            hipMalloc(&d_z_capsules, sizeof(ppln::collision::Capsule<float>) * h_env.num_z_aligned_capsules);
            hipMemcpy(d_z_capsules, h_env.z_aligned_capsules,
                    sizeof(ppln::collision::Capsule<float>) * h_env.num_z_aligned_capsules,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->z_aligned_capsules), &d_z_capsules, sizeof(ppln::collision::Capsule<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_z_aligned_capsules), &h_env.num_z_aligned_capsules, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }

        if (h_env.num_cylinders > 0) {
            ppln::collision::Cylinder<float> *d_cylinders;
            hipMalloc(&d_cylinders, sizeof(ppln::collision::Cylinder<float>) * h_env.num_cylinders);
            hipMemcpy(d_cylinders, h_env.cylinders,
                    sizeof(ppln::collision::Cylinder<float>) * h_env.num_cylinders,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->cylinders), &d_cylinders, sizeof(ppln::collision::Cylinder<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_cylinders), &h_env.num_cylinders, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }

        if (h_env.num_cuboids > 0) {
            ppln::collision::Cuboid<float> *d_cuboids;
            hipMalloc(&d_cuboids, sizeof(ppln::collision::Cuboid<float>) * h_env.num_cuboids);
            hipMemcpy(d_cuboids, h_env.cuboids,
                    sizeof(ppln::collision::Cuboid<float>) * h_env.num_cuboids,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->cuboids), &d_cuboids, sizeof(ppln::collision::Cuboid<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_cuboids), &h_env.num_cuboids, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }

        if (h_env.num_z_aligned_cuboids > 0) {
            ppln::collision::Cuboid<float> *d_z_cuboids;
            hipMalloc(&d_z_cuboids, sizeof(ppln::collision::Cuboid<float>) * h_env.num_z_aligned_cuboids);
            hipMemcpy(d_z_cuboids, h_env.z_aligned_cuboids,
                    sizeof(ppln::collision::Cuboid<float>) * h_env.num_z_aligned_cuboids,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->z_aligned_cuboids), &d_z_cuboids, sizeof(ppln::collision::Cuboid<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_z_aligned_cuboids), &h_env.num_z_aligned_cuboids, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }
    }


    inline void cleanup_environment_on_device(ppln::collision::Environment<float> *d_env, 
                                        const ppln::collision::Environment<float> &h_env) {
        // Get the pointers from device struct before freeing
        ppln::collision::Sphere<float> *d_spheres = nullptr;
        ppln::collision::Capsule<float> *d_capsules = nullptr;
        ppln::collision::Capsule<float> *d_z_capsules = nullptr;
        ppln::collision::Cylinder<float> *d_cylinders = nullptr;
        ppln::collision::Cuboid<float> *d_cuboids = nullptr;
        ppln::collision::Cuboid<float> *d_z_cuboids = nullptr;

        // Copy each pointer from device memory
        if (h_env.num_spheres > 0) {
            hipMemcpy(&d_spheres, &(d_env->spheres), sizeof(ppln::collision::Sphere<float>*), hipMemcpyDeviceToHost);
            hipFree(d_spheres);
        }
        
        if (h_env.num_capsules > 0) {
            hipMemcpy(&d_capsules, &(d_env->capsules), sizeof(ppln::collision::Capsule<float>*), hipMemcpyDeviceToHost);
            hipFree(d_capsules);
        }
        
        if (h_env.num_z_aligned_capsules > 0) {
            hipMemcpy(&d_z_capsules, &(d_env->z_aligned_capsules), sizeof(ppln::collision::Capsule<float>*), hipMemcpyDeviceToHost);
            hipFree(d_z_capsules);
        }
        
        if (h_env.num_cylinders > 0) {
            hipMemcpy(&d_cylinders, &(d_env->cylinders), sizeof(ppln::collision::Cylinder<float>*), hipMemcpyDeviceToHost);
            hipFree(d_cylinders);
        }
        
        if (h_env.num_cuboids > 0) {
            hipMemcpy(&d_cuboids, &(d_env->cuboids), sizeof(ppln::collision::Cuboid<float>*), hipMemcpyDeviceToHost);
            hipFree(d_cuboids);
        }
        
        if (h_env.num_z_aligned_cuboids > 0) {
            hipMemcpy(&d_z_cuboids, &(d_env->z_aligned_cuboids), sizeof(ppln::collision::Cuboid<float>*), hipMemcpyDeviceToHost);
            hipFree(d_z_cuboids);
        }

        // Finally free the environment struct itself
        hipFree(d_env);
    }

    __global__ void reset_device_variables_kernel() {
        solved = 0;
        
        atomic_free_index[0] = 0;
        atomic_free_index[1] = 0;
        
        path_size[0] = 0;
        path_size[1] = 0;
        
        for (int tree = 0; tree < 2; tree++) {
            for (int i = 0; i < 500; i++) {
                path[tree][i] = 0.0f;
            }
        }
        
        cost = 0.0f;
        reached_goal_idx = 0;
        }

    void reset_device_variables() {
        reset_device_variables_kernel<<<1, 1>>>();
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(error));
        }
    }
    
    template <typename Robot>
    __global__ void rrtc(
        float **nodes,
        int **parents,
        float **radii,
        HaltonState<Robot> *halton_states,
        hiprandState *rng_states,
        ppln::collision::Environment<float> *env
    )
    {
        
        // printf("rrtc\n");
        static constexpr auto dim = Robot::dimension;
        const int tid = threadIdx.x;
        const int bid = blockIdx.x; // 0 ... NUM_NEW_CONFIGS
        __shared__ int t_tree_id; // this tree
        __shared__ int o_tree_id; // the other tree
        __shared__ volatile float config[dim];
        __shared__ volatile float sdata[MAX_GRANULARITY];
        __shared__ volatile unsigned int sindex[MAX_GRANULARITY];
        __shared__ volatile unsigned int local_cc_result[1];
        __shared__ float *t_nodes;
        __shared__ float *o_nodes;
        __shared__ int *t_parents;
        __shared__ int *o_parents;
        __shared__ float scale;
        __shared__ volatile float *nearest_node;
        __shared__ volatile float delta[dim];
        __shared__ float var_cache[MAX_GRANULARITY][10];
        __shared__ volatile int index;
        __shared__ volatile float vec[dim];
        __shared__ unsigned int n_extensions;

        // printf("here1\n");
        /* sample_edges */
        // if (tid < dim) {
        //     config[tid] = hiprand_uniform(&rng_states[bid * dim + tid]);
        // }
        // __syncthreads();

        int iter=0;

        while (true){

            
            if (tid == 0) {
                iter++;
                if (iter > d_settings.max_iters) atomicCAS((int *)&solved, 0, -1);
                // t_tree_id = 0;

                if (!d_settings.balance || iter==1){
                    t_tree_id = (bid < (d_settings.num_new_configs / 2))? 0 : 1;
                    o_tree_id = 1 - t_tree_id;
                }
                else if (d_settings.balance && abs(atomic_free_index[0]-atomic_free_index[1]) < 1.5 * d_settings.num_new_configs){
                    float ratio = atomic_free_index[0] / (float)(atomic_free_index[0]+atomic_free_index[1]);
                    float balance_factor = 1 - ratio;
                    t_tree_id = (bid < (d_settings.num_new_configs * balance_factor))? 0 : 1;
                    o_tree_id = 1 - t_tree_id;
                }
                else if (d_settings.balance) {
                    float ratio = atomic_free_index[0] / (float)(atomic_free_index[0] + atomic_free_index[1]);
                    if (ratio < d_settings.tree_ratio) t_tree_id = 0;
                    else t_tree_id = 1;
                    o_tree_id = 1 - t_tree_id;
                }

                t_nodes = nodes[t_tree_id];
                o_nodes = nodes[o_tree_id];
                t_parents = parents[t_tree_id];
                o_parents = parents[o_tree_id];
                
                halton_next(halton_states[bid], (float *)config);
                Robot::scale_cfg((float *)config);
                local_cc_result[0] = 0;
            }
            __syncthreads();
            

            // if (tid == 0 && bid == 1) {
            //     printf("sample: ");
            //     print_config(config, dim);
            // }
            // __syncthreads();

            // divide up the work of finding nearest neighbor among the threads
            float local_min_dist = INFINITY;
            unsigned int local_near_idx = 0;
            float dist;
            for (unsigned int i = 0; i < atomic_free_index[t_tree_id]; i += blockDim.x) {
                dist = device_utils::sq_l2_dist(&t_nodes[i * dim], (float *) config, dim);
                if (dist < local_min_dist) {
                    local_min_dist = dist;
                    local_near_idx = i;
                }
            }
            sdata[tid] = local_min_dist;
            sindex[tid] = local_near_idx;
            __syncthreads();

            for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
                
                float sdata_tid_s = sdata[tid + s];
                float sdata_tid = sdata[tid];
                
                __syncthreads();
                if (tid < s && ((tid + s) < atomic_free_index[t_tree_id])){
                    if (sdata_tid_s < sdata_tid) {
                        sdata[tid] = sdata[tid + s];
                        sindex[tid] = sindex[tid + s];
                    }
                }
                __syncthreads();
            }

            // by this point NN dist = sdata[0], NN index = sindex[0]
            // now calculate the extension

            if (tid == 0) {
                sdata[0] = sqrt(sdata[0]);
                scale = min(1.0f, d_settings.range / (sdata[0]));
                nearest_node = &t_nodes[sindex[0] * dim];
            }
            __syncthreads();

            float nearest_radius = radii[t_tree_id][sindex[0]];
            if (d_settings.dynamic_domain && nearest_radius < sdata[0]) {
                continue;
            }

            if (tid < dim) {
                config[tid] = nearest_node[tid] + ((config[tid] - nearest_node[tid]) * scale);
                delta[tid] = (config[tid] - nearest_node[tid]) / (float) d_settings.granularity;
            }
            __syncthreads();

            if (solved!=0) return;
            
            /* validate_edges */
            float interp_cfg[dim];
            for (int i = 0; i < dim; i++) {
                interp_cfg[i] = nearest_node[i] + ((tid + 1) * delta[i]);

            }
            
            bool config_in_collision = not ppln::collision::fkcc<Robot>(interp_cfg, env, var_cache, tid, local_cc_result);
            atomicOr((unsigned int *)&local_cc_result[0], config_in_collision ? 1u : 0u);
            __syncthreads();
            if (local_cc_result[0] == 0 && sdata[0] > 0) {
                /* grow tree */
                if (tid == 0) {
                    index = atomicAdd((int *)&atomic_free_index[t_tree_id], 1);
                    if (index >= d_settings.max_samples) solved=-1;
                    t_parents[index] = sindex[0];
                    radii[t_tree_id][index] = FLT_MAX;
                    if (d_settings.dynamic_domain && nearest_radius != FLT_MAX) {
                        radii[t_tree_id][sindex[0]] *= (1 + d_settings.dd_alpha);
                    }
                }
                __syncthreads();

                if (tid < dim) {
                    t_nodes[index * dim + tid] = config[tid];
                }
                __syncthreads();

                /* connect */
                local_min_dist = INFINITY;
                local_near_idx = 0;
                for (unsigned int i = 0; i < atomic_free_index[o_tree_id]; i += blockDim.x) {
                    dist = device_utils::sq_l2_dist(&o_nodes[i * dim], (float *)config, dim);
                    if (dist < local_min_dist) {
                        local_min_dist = dist;
                        local_near_idx = i;
                    }
                }
                sdata[tid] = local_min_dist;
                sindex[tid] = local_near_idx;
                __syncthreads();
                
                for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
                    if (tid < s) {
                        if (sdata[tid + s] < sdata[tid]) {
                            sdata[tid] = sdata[tid + s];
                            sindex[tid] = sindex[tid + s];
                        }
                    }
                    __syncthreads();
                }
                
                
                if (tid == 0) {
                    sdata[0] = sqrt(sdata[0]);
                    // scale = min(1.0f, RRT_RADIUS / sdata[0]);
                    nearest_node = &o_nodes[sindex[0] * dim];
                    n_extensions = ceil(sdata[0] / d_settings.range);
                    local_cc_result[0] = 0;
                }
                __syncthreads();

                if (tid < dim) {
                    vec[tid] = (nearest_node[tid] - config[tid]) / (float) n_extensions;
                }
                __syncthreads();

                // validate the edge to the nearest neighbor in opposite tree, go as far as we can
                int i_extensions = 0;
                int extension_parent_idx = index;
                // printf("here6\n");
                while (i_extensions < n_extensions) {
                    /* each thread checking an interpolated config along the extension vector*/
                    for (int i = 0; i < dim; i++) {
                        interp_cfg[i] = config[i] + ((tid + 1) * (vec[i] / d_settings.granularity));
                    }
                    bool config_in_collision = not ppln::collision::fkcc<Robot>(interp_cfg, env, var_cache, tid, local_cc_result);
                    atomicOr((unsigned int *)&local_cc_result[0], config_in_collision ? 1u : 0u);
                    __syncthreads();
                    if (local_cc_result[0] != 0) break;
                    /* add extension to tree */
                    if (tid == 0) {
                        index = atomicAdd((int *)&atomic_free_index[t_tree_id], 1);
                        if (index >= d_settings.max_samples) solved=-1;
                        t_parents[index] = extension_parent_idx;
                        radii[t_tree_id][index] = FLT_MAX;
                        extension_parent_idx = index;
                        local_cc_result[0] = 0;
                    }
                    __syncthreads();
                    if (tid < dim) {
                        config[tid] = config[tid] + vec[tid];
                        t_nodes[index * dim + tid] = config[tid];
                    }
                    
                    i_extensions++;
                    __syncthreads();
                }
                if (i_extensions == n_extensions) { // connected
                    if (tid == 0 && atomicCAS((int *)&solved, 0, 1) == 0) {
                        // trace back to the start and goal.
                        int current = index;
                        int parent;
                        int t_path_size = 0;
                        int o_path_size = 0;
                        while (t_parents[current] != current) {
                            // printf("entered here1\n");
                            // printf("path config: ");
                            // print_config(&t_nodes[current*dim], dim);
                            parent = t_parents[current];
                            cost += device_utils::l2_dist(&t_nodes[current * dim], &t_nodes[parent * dim], dim);
                            for (int i = 0; i < dim; i++) path[t_tree_id][t_path_size * dim + i] = t_nodes[current * dim + i];
                            
                            //print_config(&t_nodes[current * dim], dim);
                            t_path_size++;
                            current = parent;
                            
                        }
                        //printf("GPU path above");
                        if (t_tree_id == 1) reached_goal_idx = current;
                        current = sindex[0];
                        // printf("entered here2\n");
                        while(o_parents[current] != current) {
                            parent = o_parents[current];
                            cost += device_utils::l2_dist(&o_nodes[current * dim], &o_nodes[parent * dim], dim);
                            for (int i = 0; i < dim; i++) path[o_tree_id][o_path_size * dim + i] = o_nodes[current * dim + i];
                            //print_config(&o_nodes[current * dim], dim);
                            o_path_size++;
                            current = parent;
                        }
                        //printf("GPU path above 2");
                        if (t_tree_id == 0) reached_goal_idx = current;
                        path_size[t_tree_id] = t_path_size;
                        path_size[o_tree_id] = o_path_size;
                        //printf("path_size: {%d, %d}; cost: %f\n", path_size[0], path_size[1], cost);
                        // printf("entered here3\n");
                        return;
                    }
                    __syncthreads();
                }
                // printf("here8\n");
            }
            else if (d_settings.dynamic_domain) {
                if (nearest_radius == FLT_MAX)
                {
                    radii[t_tree_id][sindex[0]] = d_settings.dd_radius;
                }
                else
                {
                    radii[t_tree_id][sindex[0]] = max(radii[t_tree_id][sindex[0]] * (1.F - d_settings.dd_alpha), d_settings.dd_min_radius);
                }
            }
            __syncthreads();
            if (solved!=0) return;
        }
        
        
    }




    template <typename Robot>
    PlannerResult<Robot> solve(
        typename Robot::Configuration &start,
        std::vector<typename Robot::Configuration> &goals,
        ppln::collision::Environment<float> &h_environment,
        pRRTC_settings &settings
    ) 
    {
        static constexpr auto dim = Robot::dimension;
        std::size_t iter = 0;
        std::size_t start_index = 0;

        
        PlannerResult<Robot> res;
        // copy data to GPU

        // pRRTC_settings *d_settings;
        // hipMalloc(&d_settings, sizeof(pRRTC_settings));
        hipMemcpyToSymbol(HIP_SYMBOL(d_settings), &settings, sizeof(settings));

        float *start_config;
        float *goal_configs;
        int num_goals = goals.size();
        float *nodes[2];
        int *parents[2];
        float *radii[2];
        float **d_nodes;
        int **d_parents;
        float **d_radii;
        hipMalloc(&d_nodes, 2 * sizeof(float*));
        hipMalloc(&d_parents, 2 * sizeof(int*));
        hipMalloc(&d_radii, 2 * sizeof(float*));
        const std::size_t config_size = dim * sizeof(float);
        hipMalloc(&start_config, config_size);
        hipMalloc(&goal_configs, config_size * num_goals);
        for (int i = 0; i < 2; i++) {
            hipMalloc(&nodes[i], settings.max_samples * config_size);
            hipMalloc(&parents[i], settings.max_samples * sizeof(int));
            hipMalloc(&radii[i], settings.max_samples * sizeof(float));
        }
        hipMemcpy(d_nodes, nodes, 2 * sizeof(float*), hipMemcpyHostToDevice);
        hipMemcpy(d_parents, parents, 2 * sizeof(int*), hipMemcpyHostToDevice);
        hipMemcpy(d_radii, radii, 2 * sizeof(float*), hipMemcpyHostToDevice);

        hipMemcpy(start_config, start.data(), config_size, hipMemcpyHostToDevice);
        hipMemcpy(goal_configs, goals.data(), config_size, hipMemcpyHostToDevice);

        // add start to tree_a and goals to tree_b
        hipMemcpy(nodes[0], start.data(), config_size, hipMemcpyHostToDevice);
        hipMemcpy(parents[0], &start_index, sizeof(int), hipMemcpyHostToDevice);

        hipMemcpy(nodes[1], goals.data(), config_size * num_goals, hipMemcpyHostToDevice);
        std::vector<int> parents_b_init(num_goals);
        iota(parents_b_init.begin(), parents_b_init.end(), 0); // consecutive integers from 0 ... num_goals - 1
        hipMemcpy(parents[1], parents_b_init.data(), sizeof(int) * num_goals, hipMemcpyHostToDevice);

        // initialize radii
        std::vector<float> radii_init(num_goals, FLT_MAX);
        hipMemcpy(radii[0], radii_init.data(), sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(radii[1], radii_init.data(), sizeof(float) * num_goals, hipMemcpyHostToDevice);

        // create a hiprandState for each thread -> holds state of RNG for each thread seperately
        // For growing the tree we will create NUM_NEW_CONFIGS threads
        hiprandState *rng_states;
        int num_rng_states = settings.num_new_configs * dim;
        hipMalloc(&rng_states, num_rng_states * sizeof(hiprandState));
        int numBlocks = (num_rng_states + BLOCK_SIZE - 1) / BLOCK_SIZE;
        init_rng<<<numBlocks, BLOCK_SIZE>>>(rng_states, 1, num_rng_states);

        HaltonState<Robot> *halton_states;
        hipMalloc(&halton_states, settings.num_new_configs * sizeof(HaltonState<Robot>));
        int numBlocks1 = (settings.num_new_configs + BLOCK_SIZE - 1) / BLOCK_SIZE;
        init_halton<Robot><<<numBlocks1, BLOCK_SIZE>>>(halton_states, rng_states);

        // free index for next available position in tree_a and tree_b
        int h_free_index[2] = {1, num_goals};
        hipMemcpyToSymbol(HIP_SYMBOL(atomic_free_index), &h_free_index, sizeof(int) * 2);

        // allocate for obstacles
        ppln::collision::Environment<float> *env;
        setup_environment_on_device(env, h_environment);

        cudaCheckError(hipGetLastError());
        // Setup pinned memory for signaling
        int *h_solved;
        int current_samples[2];
        hipHostMalloc(&h_solved, sizeof(int));  // Pinned memory
        *h_solved = 0;
        
        auto start_time = std::chrono::steady_clock::now();
        
        // std::cout << "iter: " << iter << "\n";
        rrtc<Robot><<<settings.num_new_configs, settings.granularity>>> (
            d_nodes,
            d_parents,
            d_radii,
            halton_states,
            rng_states,
            env
        );
        hipDeviceSynchronize();
        // cudaCheckError(hipGetLastError());

        res.nanoseconds = get_elapsed_nanoseconds(start_time);

        // int current_samples[2];
        hipMemcpyFromSymbol(current_samples, HIP_SYMBOL(atomic_free_index), sizeof(int) * 2, 0, hipMemcpyDeviceToHost);
        

        hipMemcpyFromSymbol(h_solved, HIP_SYMBOL(solved), sizeof(int), 0, hipMemcpyDeviceToHost);

        // currently, iteration count is not copied because each block may have different iteration count

        if (*h_solved!=1) *h_solved=0;
        
        std::cout << "current_samples: start: " << current_samples[0] << ", goal: " << current_samples[1] << "\n";
        // printf("current_samples: %d, %d\n", current_samples[0], current_samples[1]);
        std::cout << "iters: " << iter << "\n";
        std::cout << "exited loop\n";
        Robot::print_robot_config(start);
        Robot::print_robot_config(goals[0]);
        if (*h_solved) {
            std::cout << "solved!\n";
            int h_path_size[2];
            float h_paths[2][500];
            float h_cost;
            int h_reached_goal_idx;
            hipMemcpyFromSymbol(h_path_size, HIP_SYMBOL(path_size), sizeof(int) * 2, 0, hipMemcpyDeviceToHost);
            hipMemcpyFromSymbol(h_paths, HIP_SYMBOL(path), sizeof(float) * 2 * 500, 0, hipMemcpyDeviceToHost);
            hipMemcpyFromSymbol(&h_cost, HIP_SYMBOL(cost), sizeof(float), 0, hipMemcpyDeviceToHost);
            hipMemcpyFromSymbol(&h_reached_goal_idx, HIP_SYMBOL(reached_goal_idx), sizeof(int), 0, hipMemcpyDeviceToHost);
            Robot::print_robot_config(goals[h_reached_goal_idx]);
            for (int i = h_path_size[1] - 1; i >= 0; i--) print_cfg<Robot>(&h_paths[1][i * dim]);
            for (int i = 0; i < h_path_size[0]; i++) print_cfg<Robot>(&h_paths[0][i * dim]);
            Robot::print_robot_config(start);
            res.cost = h_cost;
            std::cout << "cost: " << res.cost << "\n";
        }
        // cudaCheckError(hipGetLastError());
        res.solved = (*h_solved) != 0;
        res.iters = iter;
        
        cleanup_environment_on_device(env, h_environment);
        reset_device_variables();
        // cudaCheckError(hipGetLastError());
        hipFree(start_config);
        hipFree(goal_configs);
        // cudaCheckError(hipGetLastError());
        hipFree(nodes[0]);
        hipFree(nodes[1]);
        // cudaCheckError(hipGetLastError());
        hipFree(parents[0]);
        hipFree(parents[1]);
        // cudaCheckError(hipGetLastError());
        hipFree(rng_states);
        hipFree(halton_states);
        hipFree(d_nodes);
        hipFree(d_parents);
        // hipFree(h_solved);
        cudaCheckError(hipGetLastError());
        return res;
    }

    template PlannerResult<typename ppln::robots::Sphere> solve<ppln::robots::Sphere>(std::array<float, 3>&, std::vector<std::array<float, 3>>&, ppln::collision::Environment<float>&, pRRTC_settings&);
    template PlannerResult<typename ppln::robots::Panda> solve<ppln::robots::Panda>(std::array<float, 7>&, std::vector<std::array<float, 7>>&, ppln::collision::Environment<float>&, pRRTC_settings&);
    template PlannerResult<typename ppln::robots::Fetch> solve<ppln::robots::Fetch>(std::array<float, 8>&, std::vector<std::array<float, 8>>&, ppln::collision::Environment<float>&, pRRTC_settings&);
}
