#include "hip/hip_runtime.h"
#include "Planners.hh"
#include "Robots.hh"
#include "utils.cuh"
#include "pRRTC_settings.hh"
#include "src/collision/environment.hh"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <float.h>

#include <vector>
#include <iostream>
#include <cassert>
#include <algorithm>
#include <numeric>

#define FULL_MASK 0xffffffff


/*
Parallelized RRTC: Each warp works to add a config to the tree (either start or goal depending on balance)
*/


namespace pwRRTC {
    using namespace ppln;
    __device__ volatile int solved = 0;
    __device__ volatile int atomic_free_index[2]; // separate for tree_a and tree_b
    __device__ volatile int nodes_size[2];
    __device__ float path[2][500]; // solution path segments for tree_a, and tree_b
    __device__ int path_size[2] = {0, 0};
    __device__ float cost = 0.0;
    __device__ int reached_goal_idx = 0;
    __device__ int solved_iters = 0; // value of iters in the block that solves the problem
    __constant__ pRRTC_settings d_settings;
    __device__ int print_id = 0;

    constexpr int MAX_GRANULARITY = 32;

    constexpr int BLOCK_SIZE = 64;
    constexpr float UNWRITTEN_VAL = -9999.0f;

    // Constants
    // __constant__ float primes[16] = {
    //     3.f, 5.f, 7.f, 11.f, 13.f, 17.f, 19.f, 23.f,
    //     29.f, 31.f, 37.f, 41.f, 43.f, 47.f, 53.f, 59.f
    // };

    template<typename Robot>
    struct HaltonState {
        float b[Robot::dimension];   // bases
        float n[Robot::dimension];   // numerators
        float d[Robot::dimension];   // denominators
    };

    void __device__ shuffle_array(float *array, int n, hiprandState &state) {
        for (int i = n - 1; i > 0; i--) {
            int j = hiprand(&state) % (i + 1);
            float temp = array[i];
            array[i] = array[j];
            array[j] = temp;
        }
    }

    template<typename Robot>
    __device__ void halton_initialize(HaltonState<Robot>& state, size_t skip_iterations, hiprandState& rng_state, int idx) {
        
        float primes[16] = {
            3.f, 5.f, 7.f, 11.f, 13.f, 17.f, 19.f, 23.f,
            29.f, 31.f, 37.f, 41.f, 43.f, 47.f, 53.f, 59.f
        };
        if (idx != 0) shuffle_array(primes, 16, rng_state);
        
        // Initialize bases from primes
        for (size_t i = 0; i < Robot::dimension; i++) {
            state.b[i] = primes[i];
            state.n[i] = 0.0f;
            state.d[i] = 1.0f;
        }
        
        // Skip iterations if requested
        volatile float temp_result[Robot::dimension];
        for (size_t i = 0; i < skip_iterations; i++) {
            halton_next(state, (float *)temp_result);
        }
    }

    template<typename Robot>
    __device__ void halton_next(HaltonState<Robot>& state, float* result) {
        for (size_t i = 0; i < Robot::dimension; i++) {
            float xf = state.d[i] - state.n[i];
            bool x_eq_1 = (xf == 1.0f);
            
            if (x_eq_1) {
                // x == 1 case
                state.d[i] = floorf(state.d[i] * state.b[i]);
                state.n[i] = 1.0f;
            } else {
                // x != 1 case
                float y = floorf(state.d[i] / state.b[i]);
                
                // Continue dividing by b until we find the right digit position
                while (xf <= y) {
                    y = floorf(y / state.b[i]);
                }
                
                state.n[i] = floorf((state.b[i] + 1.0f) * y) - xf;
            }
            
            result[i] = state.n[i] / state.d[i];
        }
    }

    __global__ void init_rng(hiprandState* states, unsigned long seed, int num_rng_states) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= num_rng_states) return;
        hiprand_init(seed + idx, idx, 0, &states[idx]);
    }

    template <typename Robot>
    __global__ void init_halton(HaltonState<Robot>* states, hiprandState* cr_states) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= d_settings.num_new_configs) return;
        // int skip = (hiprand_uniform(&cr_states[idx]) * 50000.0f);
        int skip = 0;
        if (idx == 0) skip = 0;
        halton_initialize(states[idx], skip, cr_states[idx], idx);
    }

    __device__ inline void print_config(volatile float *config, int dim) {
        for (int i = 0; i < dim; i++) {
            printf("%f ,", config[i]);
        }
        printf("\n");
    }

    inline void setup_environment_on_device(ppln::collision::Environment<float> *&d_env, 
                                      const ppln::collision::Environment<float> &h_env) {
        // First allocate the environment struct
        hipMalloc(&d_env, sizeof(ppln::collision::Environment<float>));
        
        // Initialize struct to zeros first
        hipMemset(d_env, 0, sizeof(ppln::collision::Environment<float>));

        // Handle each primitive type separately
        if (h_env.num_spheres > 0) {
            // Allocate and copy spheres array
            ppln::collision::Sphere<float> *d_spheres;
            hipMalloc(&d_spheres, sizeof(ppln::collision::Sphere<float>) * h_env.num_spheres);
            hipMemcpy(d_spheres, h_env.spheres, 
                    sizeof(ppln::collision::Sphere<float>) * h_env.num_spheres, 
                    hipMemcpyHostToDevice);
            
            // Update the struct fields directly
            hipMemcpy(&(d_env->spheres), &d_spheres, sizeof(ppln::collision::Sphere<float>*), 
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_spheres), &h_env.num_spheres, sizeof(unsigned int), 
                    hipMemcpyHostToDevice);
        }

        if (h_env.num_capsules > 0) {
            ppln::collision::Capsule<float> *d_capsules;
            hipMalloc(&d_capsules, sizeof(ppln::collision::Capsule<float>) * h_env.num_capsules);
            hipMemcpy(d_capsules, h_env.capsules,
                    sizeof(ppln::collision::Capsule<float>) * h_env.num_capsules,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->capsules), &d_capsules, sizeof(ppln::collision::Capsule<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_capsules), &h_env.num_capsules, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }

        // Repeat for each primitive type...
        if (h_env.num_z_aligned_capsules > 0) {
            ppln::collision::Capsule<float> *d_z_capsules;
            hipMalloc(&d_z_capsules, sizeof(ppln::collision::Capsule<float>) * h_env.num_z_aligned_capsules);
            hipMemcpy(d_z_capsules, h_env.z_aligned_capsules,
                    sizeof(ppln::collision::Capsule<float>) * h_env.num_z_aligned_capsules,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->z_aligned_capsules), &d_z_capsules, sizeof(ppln::collision::Capsule<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_z_aligned_capsules), &h_env.num_z_aligned_capsules, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }

        if (h_env.num_cylinders > 0) {
            ppln::collision::Cylinder<float> *d_cylinders;
            hipMalloc(&d_cylinders, sizeof(ppln::collision::Cylinder<float>) * h_env.num_cylinders);
            hipMemcpy(d_cylinders, h_env.cylinders,
                    sizeof(ppln::collision::Cylinder<float>) * h_env.num_cylinders,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->cylinders), &d_cylinders, sizeof(ppln::collision::Cylinder<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_cylinders), &h_env.num_cylinders, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }

        if (h_env.num_cuboids > 0) {
            ppln::collision::Cuboid<float> *d_cuboids;
            hipMalloc(&d_cuboids, sizeof(ppln::collision::Cuboid<float>) * h_env.num_cuboids);
            hipMemcpy(d_cuboids, h_env.cuboids,
                    sizeof(ppln::collision::Cuboid<float>) * h_env.num_cuboids,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->cuboids), &d_cuboids, sizeof(ppln::collision::Cuboid<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_cuboids), &h_env.num_cuboids, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }

        if (h_env.num_z_aligned_cuboids > 0) {
            ppln::collision::Cuboid<float> *d_z_cuboids;
            hipMalloc(&d_z_cuboids, sizeof(ppln::collision::Cuboid<float>) * h_env.num_z_aligned_cuboids);
            hipMemcpy(d_z_cuboids, h_env.z_aligned_cuboids,
                    sizeof(ppln::collision::Cuboid<float>) * h_env.num_z_aligned_cuboids,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->z_aligned_cuboids), &d_z_cuboids, sizeof(ppln::collision::Cuboid<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_z_aligned_cuboids), &h_env.num_z_aligned_cuboids, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }
    }


    inline void cleanup_environment_on_device(ppln::collision::Environment<float> *d_env, 
                                        const ppln::collision::Environment<float> &h_env) {
        // Get the pointers from device struct before freeing
        ppln::collision::Sphere<float> *d_spheres = nullptr;
        ppln::collision::Capsule<float> *d_capsules = nullptr;
        ppln::collision::Capsule<float> *d_z_capsules = nullptr;
        ppln::collision::Cylinder<float> *d_cylinders = nullptr;
        ppln::collision::Cuboid<float> *d_cuboids = nullptr;
        ppln::collision::Cuboid<float> *d_z_cuboids = nullptr;

        // Copy each pointer from device memory
        if (h_env.num_spheres > 0) {
            hipMemcpy(&d_spheres, &(d_env->spheres), sizeof(ppln::collision::Sphere<float>*), hipMemcpyDeviceToHost);
            hipFree(d_spheres);
        }
        
        if (h_env.num_capsules > 0) {
            hipMemcpy(&d_capsules, &(d_env->capsules), sizeof(ppln::collision::Capsule<float>*), hipMemcpyDeviceToHost);
            hipFree(d_capsules);
        }
        
        if (h_env.num_z_aligned_capsules > 0) {
            hipMemcpy(&d_z_capsules, &(d_env->z_aligned_capsules), sizeof(ppln::collision::Capsule<float>*), hipMemcpyDeviceToHost);
            hipFree(d_z_capsules);
        }
        
        if (h_env.num_cylinders > 0) {
            hipMemcpy(&d_cylinders, &(d_env->cylinders), sizeof(ppln::collision::Cylinder<float>*), hipMemcpyDeviceToHost);
            hipFree(d_cylinders);
        }
        
        if (h_env.num_cuboids > 0) {
            hipMemcpy(&d_cuboids, &(d_env->cuboids), sizeof(ppln::collision::Cuboid<float>*), hipMemcpyDeviceToHost);
            hipFree(d_cuboids);
        }
        
        if (h_env.num_z_aligned_cuboids > 0) {
            hipMemcpy(&d_z_cuboids, &(d_env->z_aligned_cuboids), sizeof(ppln::collision::Cuboid<float>*), hipMemcpyDeviceToHost);
            hipFree(d_z_cuboids);
        }

        // Finally free the environment struct itself
        hipFree(d_env);
    }

    __global__ void reset_device_variables_kernel() {
        solved = 0;
        
        atomic_free_index[0] = 0;
        atomic_free_index[1] = 0;
        nodes_size[0] = 0;
        nodes_size[1] = 0;
        
        path_size[0] = 0;
        path_size[1] = 0;
        
        for (int tree = 0; tree < 2; tree++) {
            for (int i = 0; i < 500; i++) {
                path[tree][i] = 0.0f;
            }
        }
        
        cost = 0.0f;
        reached_goal_idx = 0;
        }

    void reset_device_variables() {
        reset_device_variables_kernel<<<1, 1>>>();
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(error));
        }
    }

    __device__ __forceinline__ bool check_partially_written(float *node, int dim) {
        #pragma unroll
        for (int i = 0; i < dim; i++) {
            if (node[i] == UNWRITTEN_VAL) return true;
        }
        return false;
    }
    
    template <typename Robot>
    __global__ void
    __launch_bounds__(128, 8)
    rrtc(
        float **nodes,
        int **parents,
        float **radii,
        HaltonState<Robot> *halton_states,
        hiprandState *rng_states,
        ppln::collision::Environment<float> *env
    )
    {
        static constexpr auto dim = Robot::dimension;
        // const int tid = threadIdx.x;
        const int bid = blockIdx.x; // 0 ... NUM_NEW_CONFIGS
        const int lid = threadIdx.x%32;
        const int wid = threadIdx.x/32;
        const int global_wid = bid * 4 + wid;
        __shared__ float config[4][dim];
        __shared__ float delta[4][dim];
        __shared__ int index[4];

        int iter = 0;
        int t_tree_id = 0;
        int o_tree_id = 1;
        float *t_nodes = nodes[t_tree_id];
        float *o_nodes = nodes[o_tree_id];
        int *t_parents = parents[t_tree_id];
        int *o_parents = parents[o_tree_id];
        float *t_radii = radii[t_tree_id];

        while (true) {
            if (solved != 0) return;
            // if (tid < dim) {
            //     config[tid] = hiprand_uniform(&rng_states[bid * dim + tid]);
            // }
            // __syncthreads();
            printf("iter: %d, global_wid: %d\n", iter, global_wid);
            iter++;
            if (iter > d_settings.max_iters) {
                // printf("max iters reached from bid: %d\n", bid);
                atomicCAS((int *)&solved, 0, -1);
            }

            if (d_settings.balance == 0 || iter == 1) {
                t_tree_id = (bid < (d_settings.num_new_configs / 2))? 0 : 1;
                o_tree_id = 1 - t_tree_id;
            }
            else if (d_settings.balance == 1 && abs(atomic_free_index[0]-atomic_free_index[1]) < 1.5 * d_settings.num_new_configs) {
                float ratio = atomic_free_index[0] / (float)(atomic_free_index[0] + atomic_free_index[1]);
                float balance_factor = 1 - ratio;
                t_tree_id = (bid < (d_settings.num_new_configs * balance_factor))? 0 : 1;
                o_tree_id = 1 - t_tree_id;
            }
            else if (d_settings.balance == 1) {
                float ratio = atomic_free_index[0] / (float)(atomic_free_index[0] + atomic_free_index[1]);
                if (ratio < d_settings.tree_ratio) t_tree_id = 0;
                else t_tree_id = 1;
                o_tree_id = 1 - t_tree_id;
            }
            else if (d_settings.balance == 2) { // vamp balance algo
                float ratio = abs(atomic_free_index[t_tree_id] - atomic_free_index[o_tree_id]) / (float) atomic_free_index[t_tree_id];
                if (ratio < d_settings.tree_ratio)
                {
                    t_tree_id = 1 - t_tree_id;
                    o_tree_id = 1 - t_tree_id;
                }
            }

            t_nodes = nodes[t_tree_id];
            o_nodes = nodes[o_tree_id];
            t_parents = parents[t_tree_id];
            o_parents = parents[o_tree_id];
            t_radii = radii[t_tree_id];

            if (lid == 0) {
                halton_next(halton_states[global_wid], (float *)config[wid]);
                Robot::scale_cfg((float *)config[wid]);
            }
            __syncwarp();

            // divide up the work of finding nearest neighbor among the warp
            float min_dist = FLT_MAX;
            int min_index = 0;
            float dist;
            // int size = nodes_size[t_tree_id];
            int size = atomic_free_index[t_tree_id];
            for (int i = lid; i < size; i += 32) {
                // while (check_partially_written(&t_nodes[i * dim], dim)) {};
                if (check_partially_written(&t_nodes[i * dim], dim)) break;
                dist = device_utils::sq_l2_dist((float *)&t_nodes[i * dim], (float *) config, dim);
                if (dist < min_dist) {
                    min_dist = dist;
                    min_index = i;
                }
            }
            
            for (unsigned int offset = 16; offset > 0; offset /= 2) {
                float other_dist = __shfl_down_sync(FULL_MASK, min_dist, offset);
                int other_index = __shfl_down_sync(FULL_MASK, min_index, offset);
                if (other_dist < min_dist) {
                    min_dist = other_dist;
                    min_index = other_index;
                }
            }
            __syncwarp();
            min_dist = __shfl_sync(FULL_MASK, min_dist, 0);
            min_index = __shfl_sync(FULL_MASK, min_index, 0);
            min_dist = sqrt(min_dist);


            float scale = min(1.0f, d_settings.range / min_dist);
            float *nearest_node = &t_nodes[min_index * dim];
            if (d_settings.dynamic_domain && t_radii[min_index] < min_dist) {
                continue;
            }

            if (lid < dim) {
                config[wid][lid] = nearest_node[lid] + ((config[wid][lid] - nearest_node[lid]) * scale);
                delta[wid][lid] = (config[wid][lid] - nearest_node[lid]) / 32.0f;
            }
            __syncwarp();

            // if (solved != 0) return;
            
            /* validate_edges */
            float interp_cfg[dim];
            for (int i = 0; i < dim; i++) {
                interp_cfg[i] = nearest_node[i] + ((lid + 1) * delta[wid][i]);
            }
            bool config_in_collision = not ppln::collision::fkcc<Robot>(interp_cfg, env, lid);
            __syncwarp();
            for (unsigned int offset = 16; offset > 0; offset /= 2) {
                // if (lid == 0) printf("here7\n");
                bool other_config_in_collision = __shfl_down_sync(FULL_MASK, config_in_collision, offset);
                // printf("other_config_in_collision: %d\n", other_config_in_collision);
                config_in_collision = config_in_collision || other_config_in_collision;
            }
            // if (lid == 0) printf("config_in_collision: %d\n", config_in_collision);
            config_in_collision = __shfl_sync(FULL_MASK, config_in_collision, 0);
            
            if (!config_in_collision) {
                /* grow tree */
                if (lid == 0) {
                    
                    index[wid] = atomicAdd((int *)&atomic_free_index[t_tree_id], 1);
                    if (index[wid] >= d_settings.max_samples) solved = -1;
                    
                    t_parents[index[wid]] = min_index;
                    
                    if (d_settings.dynamic_domain) {
                        t_radii[index[wid]] = FLT_MAX;
                        volatile float *radius_ptr = &t_radii[min_index];
                        float old_radius, new_radius;
                        int expected, desired;
                        do {
                            // printf("dynamic domain loop 1\n");
                            old_radius = *radius_ptr;
                            if (old_radius == FLT_MAX) break;
                            new_radius = old_radius * (1 + d_settings.dd_alpha);
                            expected = __float_as_int(old_radius);
                            desired = __float_as_int(new_radius);
                        } while (atomicCAS((int *)radius_ptr, expected, desired) != expected);
                    }
                }
                __syncwarp();
                if (lid < dim) {
                    t_nodes[index[wid] * dim + lid] = config[wid][lid];
                }
                __syncwarp();
               
                /* connect */
                float min_dist = FLT_MAX;
                int min_index = 0;
                int size = atomic_free_index[o_tree_id];
                for (unsigned int i = lid; i < size; i += 32) {
                    if (check_partially_written(&o_nodes[i * dim], dim)) break;
                    dist = device_utils::sq_l2_dist((float *)&o_nodes[i * dim], (float *)config[wid], dim);
                    if (dist < min_dist) {
                        min_dist = dist;
                        min_index = i;
                    }
                }
                __syncwarp();
                for (unsigned int offset = 16; offset > 0; offset /= 2) {
                    float other_dist = __shfl_down_sync(FULL_MASK, min_dist, offset);
                    int other_index = __shfl_down_sync(FULL_MASK, min_index, offset);
                    if (other_dist < min_dist) {
                        min_dist = other_dist;
                        min_index = other_index;
                    }
                }
                min_dist = __shfl_sync(FULL_MASK, min_dist, 0);
                min_index = __shfl_sync(FULL_MASK, min_index, 0);
                min_dist = sqrt(min_dist);
                int n_extensions = ceil(min_dist / d_settings.range);

                if (lid < dim) {
                    delta[wid][lid] = (o_nodes[min_index * dim + lid] - config[wid][lid]) / (float) n_extensions;
                }
                __syncwarp();

                // validate the edge to the nearest neighbor in opposite tree, go as far as we can
                int i_extensions = 0;
                int extension_parent_idx = index[wid];
                // printf("here6\n");
                while (i_extensions < n_extensions) {
                    /* each thread checking an interpolated config along the extension vector*/
                    for (int i = 0; i < dim; i++) {
                        interp_cfg[i] = config[wid][i] + ((lid + 1) * (delta[wid][i] / 32.0f));
                    }
                    config_in_collision = not ppln::collision::fkcc<Robot>(interp_cfg, env, threadIdx.x);
                    __syncwarp();
                    for (unsigned int offset = 16; offset > 0; offset /= 2) {
                        bool other_config_in_collision = __shfl_down_sync(FULL_MASK, config_in_collision, offset);
                        config_in_collision = config_in_collision || other_config_in_collision;
                    }
                    config_in_collision = __shfl_sync(FULL_MASK, config_in_collision, 0);
                    if (config_in_collision) break;
                    // if (local_cc_result[0] != 0) break;

                    /* add extension to tree */
                    if (lid == 0) {
                        index[wid] = atomicAdd((int *)&atomic_free_index[t_tree_id], 1);
                        if (index[wid] >= d_settings.max_samples) solved = -1;
                        t_parents[index[wid]] = extension_parent_idx;
                        t_radii[index[wid]] = FLT_MAX;
                        extension_parent_idx = index[wid];
                        // printf("in extension loop\n");
                    }
                    __syncwarp();
                    if (lid < dim) {
                        config[wid][lid] = config[wid][lid] + delta[wid][lid];
                        t_nodes[index[wid] * dim + lid] = config[wid][lid];
                    }
                    __syncwarp();
                    // __threadfence_system();
                    // if (tid == 0) {
                    //     atomicAdd((int *)&nodes_size[t_tree_id], 1);
                    //     // printf("added config from extension to tree %d at index %d (bid %d, parent %d): %f %f %f %f %f %f %f %f\n", t_tree_id, index, bid, t_parents[index], config[0], config[1], config[2], config[3], config[4], config[5], config[6], config[7]);
                    //     // print_config(config, dim);
                    // }
                    i_extensions++;
                }
                if (i_extensions == n_extensions) { // connected
                    if (lid == 0 && atomicCAS((int *)&solved, 0, 1) == 0) {
                        // printf("in connected\n");
                        // printf("config at connection: %f %f %f %f %f %f %f %f\n nearest node: %f %f %f %f %f %f %f %f\n", 
                        //     config[0], config[1], config[2], config[3], config[4], config[5], config[6], config[7],
                        //     nearest_node[0], nearest_node[1], nearest_node[2], nearest_node[3], nearest_node[4], nearest_node[5], nearest_node[6], nearest_node[7]
                        // );
                        // trace back to the start and goal.
                        int current = index[wid];
                        int parent;
                        int t_path_size = 0;
                        int o_path_size = 0;
                        while (t_parents[current] != current) {
                            parent = t_parents[current];
                            cost += device_utils::l2_dist((float *)&t_nodes[current * dim], (float *)&t_nodes[parent * dim], dim);
                            for (int i = 0; i < dim; i++) path[t_tree_id][t_path_size * dim + i] = t_nodes[current * dim + i];
                            // printf("added to path[%d]: %f %f %f %f %f %f %f %f\n", t_tree_id, path[t_tree_id][t_path_size * dim], path[t_tree_id][t_path_size * dim + 1], path[t_tree_id][t_path_size * dim + 2], path[t_tree_id][t_path_size * dim + 3], path[t_tree_id][t_path_size * dim + 4], path[t_tree_id][t_path_size * dim + 5], path[t_tree_id][t_path_size * dim + 6], path[t_tree_id][t_path_size * dim + 7]);
                            //print_config(&t_nodes[current * dim], dim);
                            t_path_size++;
                            current = parent;
                            
                        }
                        //printf("GPU path above");
                        if (t_tree_id == 1) reached_goal_idx = current;
                        current = min_index;
                        // printf("entered here2\n");
                        while(o_parents[current] != current) {
                            parent = o_parents[current];
                            cost += device_utils::l2_dist((float *)&o_nodes[current * dim], (float *)&o_nodes[parent * dim], dim);
                            for (int i = 0; i < dim; i++) path[o_tree_id][o_path_size * dim + i] = o_nodes[current * dim + i];
                            // printf("added to path[%d]: %f %f %f %f %f %f %f %f\n", o_tree_id, path[o_tree_id][o_path_size * dim], path[o_tree_id][o_path_size * dim + 1], path[o_tree_id][o_path_size * dim + 2], path[o_tree_id][o_path_size * dim + 3], path[o_tree_id][o_path_size * dim + 4], path[o_tree_id][o_path_size * dim + 5], path[o_tree_id][o_path_size * dim + 6], path[o_tree_id][o_path_size * dim + 7]);
                            //print_config(&o_nodes[current * dim], dim);
                            o_path_size++;
                            current = parent;
                        }
                        //printf("GPU path above 2");
                        if (t_tree_id == 0) reached_goal_idx = current;
                        path_size[t_tree_id] = t_path_size;
                        path_size[o_tree_id] = o_path_size;
                        solved_iters = iter;
                        
                        // printf("t_tree_id %d, t_path_size: %d, o_path_size: %d\n", t_tree_id, t_path_size, o_path_size);
                        // return;
                    }
                }
                // printf("here8\n");
            }
            else if (d_settings.dynamic_domain && lid == 0) {
                volatile float *radius_ptr = &t_radii[min_index];
                float old_radius, new_radius;
                int expected, desired;
                do {
                    // printf("in d settings end\n");
                    old_radius = *radius_ptr;
                    if (old_radius == FLT_MAX) {
                        new_radius = d_settings.dd_radius;
                    } else {
                        new_radius = fmaxf(old_radius * (1.f - d_settings.dd_alpha), d_settings.dd_min_radius);
                    }
                    expected = __float_as_int(old_radius);
                    desired = __float_as_int(new_radius);
                } while (atomicCAS((int *)radius_ptr, expected, desired) != expected);
            }
            if (solved != 0) return;
        }
    }




    template <typename Robot>
    PlannerResult<Robot> solve(
        typename Robot::Configuration &start,
        std::vector<typename Robot::Configuration> &goals,
        ppln::collision::Environment<float> &h_environment,
        pRRTC_settings &settings
    ) 
    {
        // std::cout << "here" << std::endl;
        hipSetDevice(1);
        // std::cout << "here1" << std::endl;
        auto start_time = std::chrono::steady_clock::now();
        static constexpr auto dim = Robot::dimension;
        // std::size_t iter = 0;
        std::size_t start_index = 0;

        
        PlannerResult<Robot> res;
        // copy data to GPU
        hipMemcpyToSymbol(HIP_SYMBOL(d_settings), &settings, sizeof(settings));

        // std::cout << "here2" << std::endl;
        int num_goals = goals.size();
        float *nodes[2];
        int *parents[2];
        float *radii[2];
        float **d_nodes;
        int **d_parents;
        float **d_radii;
        hipMalloc(&d_nodes, 2 * sizeof(float*));
        hipMalloc(&d_parents, 2 * sizeof(int*));
        hipMalloc(&d_radii, 2 * sizeof(float*));
        const std::size_t config_size = dim * sizeof(float);

        for (int i = 0; i < 2; i++) {
            hipMalloc(&nodes[i], settings.max_samples * config_size);
            hipMalloc(&parents[i], settings.max_samples * sizeof(int));
            hipMalloc(&radii[i], settings.max_samples * sizeof(float));
        }
        hipMemcpy(d_nodes, nodes, 2 * sizeof(float*), hipMemcpyHostToDevice);
        hipMemcpy(d_parents, parents, 2 * sizeof(int*), hipMemcpyHostToDevice);
        hipMemcpy(d_radii, radii, 2 * sizeof(float*), hipMemcpyHostToDevice);

        // set nodes to unitialized
        std::vector<float> nodes_init(settings.max_samples * dim, UNWRITTEN_VAL);
        hipMemcpy((void *)nodes[0], nodes_init.data(), config_size * settings.max_samples, hipMemcpyHostToDevice);
        hipMemcpy((void *)nodes[1], nodes_init.data(), config_size * settings.max_samples, hipMemcpyHostToDevice);
        // add start to tree_a and goals to tree_b
        hipMemcpy((void *)nodes[0], start.data(), config_size, hipMemcpyHostToDevice);
        hipMemcpy((void *)parents[0], &start_index, sizeof(int), hipMemcpyHostToDevice);

        hipMemcpy((void *)nodes[1], goals.data(), config_size * num_goals, hipMemcpyHostToDevice);
        std::vector<int> parents_b_init(num_goals);
        iota(parents_b_init.begin(), parents_b_init.end(), 0); // consecutive integers from 0 ... num_goals - 1
        hipMemcpy((void *)parents[1], parents_b_init.data(), sizeof(int) * num_goals, hipMemcpyHostToDevice);

        // initialize radii
        std::vector<float> radii_init(num_goals, FLT_MAX);
        hipMemcpy((void *)radii[0], radii_init.data(), sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy((void *)radii[1], radii_init.data(), sizeof(float) * num_goals, hipMemcpyHostToDevice);
        // std::cout << "here4" << std::endl;
        // create a hiprandState for each thread -> holds state of RNG for each thread seperately
        // For growing the tree we will create NUM_NEW_CONFIGS threads
        hiprandState *rng_states;
        int num_rng_states = settings.num_new_configs * dim;
        hipMalloc(&rng_states, num_rng_states * sizeof(hiprandState));
        int numBlocks = (num_rng_states + BLOCK_SIZE - 1) / BLOCK_SIZE;
        init_rng<<<numBlocks, BLOCK_SIZE>>>(rng_states, 1, num_rng_states);

        HaltonState<Robot> *halton_states;
        hipMalloc(&halton_states, settings.num_new_configs * sizeof(HaltonState<Robot>));
        int numBlocks1 = (settings.num_new_configs + BLOCK_SIZE - 1) / BLOCK_SIZE;
        init_halton<Robot><<<numBlocks1, BLOCK_SIZE>>>(halton_states, rng_states);

        // free index for next available position in tree_a and tree_b
        int h_free_index[2] = {1, num_goals};
        hipMemcpyToSymbol(HIP_SYMBOL(atomic_free_index), &h_free_index, sizeof(int) * 2);
        hipMemcpyToSymbol(HIP_SYMBOL(nodes_size), &h_free_index, sizeof(int) * 2);
        // std::cout << "here5" << std::endl;
        // allocate for obstacles
        ppln::collision::Environment<float> *env;
        setup_environment_on_device(env, h_environment);
        // std::cout << "here6" << std::endl;
        cudaCheckError(hipGetLastError());
        // Setup pinned memory for signaling
        int *h_solved;
        int current_samples[2];
        int h_solved_iters = -1;
        hipHostMalloc(&h_solved, sizeof(int));  // Pinned memory
        *h_solved = -1;


        auto kernel_start_time = std::chrono::steady_clock::now();
        rrtc<Robot><<<settings.num_new_configs, settings.granularity>>> (
            d_nodes,
            d_parents,
            d_radii,
            halton_states,
            rng_states,
            env
        );
        hipDeviceSynchronize();
        res.kernel_ns = get_elapsed_nanoseconds(kernel_start_time);
        cudaCheckError(hipGetLastError());

        // void* kernelArgs[] = {
        //     (void*)&d_nodes,
        //     (void*)&d_parents,
        //     (void*)&d_radii,
        //     (void*)&halton_states,
        //     (void*)&rng_states,
        //     (void*)&env
        // };
        // auto kernel_start_time = std::chrono::steady_clock::now();
        // hipError_t err = hipLaunchCooperativeKernel(
        //     (void*)rrtc<Robot>,  // Kernel function
        //     settings.num_new_configs, settings.granularity,  // Grid and block dimensions
        //     kernelArgs,  // Kernel arguments
        //     0  // Shared memory per block (set to 0 so the compiler and auto compute)
        // );
        // if (err != hipSuccess) {
        //     std::cerr << "CUDA Kernel launch failed: " << hipGetErrorString(err) << "\n";
        // }
        // hipDeviceSynchronize();
        // cudaCheckError(hipGetLastError());
        // res.kernel_ns = get_elapsed_nanoseconds(kernel_start_time);
        
        
        hipMemcpyFromSymbol(current_samples, HIP_SYMBOL(atomic_free_index), sizeof(int) * 2, 0, hipMemcpyDeviceToHost);
        // hipMemcpyFromSymbol(current_samples, HIP_SYMBOL(nodes_size), sizeof(int) * 2, 0, hipMemcpyDeviceToHost);
        hipMemcpyFromSymbol(h_solved, HIP_SYMBOL(solved), sizeof(int), 0, hipMemcpyDeviceToHost);
        hipMemcpyFromSymbol(&h_solved_iters, HIP_SYMBOL(solved_iters), sizeof(int), 0, hipMemcpyDeviceToHost);

        // currently, iteration count is not copied because each block may have different iteration count
        if (*h_solved!=1) *h_solved=0;
        std::cout << "current_samples: start: " << current_samples[0] << ", goal: " << current_samples[1] << "\n";
        std::cout << "solved iters: " << h_solved_iters << "\n";
        res.start_tree_size = current_samples[0];
        res.goal_tree_size = current_samples[1];
        Robot::print_robot_config(start);
        Robot::print_robot_config(goals[0]);
        if (*h_solved) {
            std::cout << "solved!\n";
            int h_path_size[2];
            float h_paths[2][500];
            float h_cost;
            int h_reached_goal_idx;
            hipMemcpyFromSymbol(h_path_size, HIP_SYMBOL(path_size), sizeof(int) * 2, 0, hipMemcpyDeviceToHost);
            hipMemcpyFromSymbol(h_paths, HIP_SYMBOL(path), sizeof(float) * 2 * 500, 0, hipMemcpyDeviceToHost);
            hipMemcpyFromSymbol(&h_cost, HIP_SYMBOL(cost), sizeof(float), 0, hipMemcpyDeviceToHost);
            hipMemcpyFromSymbol(&h_reached_goal_idx, HIP_SYMBOL(reached_goal_idx), sizeof(int), 0, hipMemcpyDeviceToHost);
            cudaCheckError(hipGetLastError());
            // Robot::print_robot_config(goals[h_reached_goal_idx]);
            res.path.emplace_back(goals[h_reached_goal_idx]);
            typename Robot::Configuration config;
            for (int i = h_path_size[1] - 1; i >= 0; i--) {
                std::copy_n(h_paths[1] + i * dim, dim, config.begin());
                res.path.emplace_back(config);
                // print_cfg_ptr<Robot>(&h_paths[1][i * dim]);
            }
            for (int i = 0; i < h_path_size[0]; i++) {
                std::copy_n(h_paths[0] + i * dim, dim, config.begin());
                res.path.emplace_back(config);
                // print_cfg_ptr<Robot>(&h_paths[0][i * dim]);
            }
            // Robot::print_robot_config(start);
            res.path.emplace_back(start);
            res.cost = h_cost;
            res.path_length = (h_path_size[0] + h_path_size[1]);
            std::cout << "cost: " << res.cost << "\n";
        }
        res.solved = (*h_solved) != 0;
        res.iters = h_solved_iters;
        
        cleanup_environment_on_device(env, h_environment);
        reset_device_variables();
        hipFree((void *)nodes[0]);
        hipFree((void *)nodes[1]);
        hipFree((void *)parents[0]);
        hipFree((void *)parents[1]);
        hipFree((void *)radii[0]);
        hipFree((void *)radii[1]);
        hipFree(rng_states);
        hipFree(halton_states);
        hipFree(d_nodes);
        hipFree(d_parents);
        hipFree(d_radii);
        hipHostFree(h_solved);
        cudaCheckError(hipGetLastError());
        res.wall_ns = get_elapsed_nanoseconds(start_time);
        hipDeviceReset();
        return res;
    }

    template PlannerResult<typename ppln::robots::Sphere> solve<ppln::robots::Sphere>(std::array<float, 3>&, std::vector<std::array<float, 3>>&, ppln::collision::Environment<float>&, pRRTC_settings&);
    template PlannerResult<typename ppln::robots::Panda> solve<ppln::robots::Panda>(std::array<float, 7>&, std::vector<std::array<float, 7>>&, ppln::collision::Environment<float>&, pRRTC_settings&);
    template PlannerResult<typename ppln::robots::Fetch> solve<ppln::robots::Fetch>(std::array<float, 8>&, std::vector<std::array<float, 8>>&, ppln::collision::Environment<float>&, pRRTC_settings&);
}
