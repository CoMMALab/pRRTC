#include "hip/hip_runtime.h"
#include "Planners.hh"
#include "Robots.hh"
#include "utils.cuh"
#include "pRRTC_settings.hh"
#include "src/collision/environment.hh"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <float.h>

#include <vector>
#include <iostream>
#include <cassert>
#include <algorithm>
#include <numeric>

#define FULL_MASK 0xffffffff

/*
Parallelized RRTC:
Each warp grows it's own tree.
*/


namespace nRRTC {
    using namespace ppln;
    __device__ volatile int solved = 0;
    __device__ int solved_free_index[2] = {0, 0};
    __device__ float path[2][500]; // solution path segments for tree_a, and tree_b
    __device__ int path_size[2] = {0, 0};
    __device__ float cost = 0.0;
    __device__ int reached_goal_idx = 0;
    __device__ int solved_iters = 0; // value of iters in the block that solves the problem
    __constant__ pRRTC_settings d_settings;

    // constexpr int MAX_GRANULARITY = 256;
    constexpr int BLOCK_SIZE = 64;
    constexpr int NUM_TREES = 256;
    constexpr int MAX_TREE_SIZE = (10000000 / NUM_TREES);

    // Constants
    __constant__ float primes[16] = {
        3.f, 5.f, 7.f, 11.f, 13.f, 17.f, 19.f, 23.f,
        29.f, 31.f, 37.f, 41.f, 43.f, 47.f, 53.f, 59.f
    };

    template<typename Robot>
    struct HaltonState {
        float b[Robot::dimension];   // bases
        float n[Robot::dimension];   // numerators
        float d[Robot::dimension];   // denominators
    };

    template<typename Robot>
    __device__ void halton_initialize(HaltonState<Robot>& state, size_t skip_iterations, hiprandState& rng_state, int idx) {
        
        float primes[16] = {
            3.f, 5.f, 7.f, 11.f, 13.f, 17.f, 19.f, 23.f,
            29.f, 31.f, 37.f, 41.f, 43.f, 47.f, 53.f, 59.f
        };
        if (idx != 0) shuffle_array(primes, 16, rng_state);
        
        // Initialize bases from primes
        for (size_t i = 0; i < Robot::dimension; i++) {
            state.b[i] = primes[i];
            state.n[i] = 0.0f;
            state.d[i] = 1.0f;
        }
        
        // Skip iterations if requested
        volatile float temp_result[Robot::dimension];
        for (size_t i = 0; i < skip_iterations; i++) {
            halton_next(state, (float *)temp_result);
        }
    }

    template<typename Robot>
    __device__ void halton_next(HaltonState<Robot>& state, float* result) {
        for (size_t i = 0; i < Robot::dimension; i++) {
            float xf = state.d[i] - state.n[i];
            bool x_eq_1 = (xf == 1.0f);
            
            if (x_eq_1) {
                // x == 1 case
                state.d[i] = floorf(state.d[i] * state.b[i]);
                state.n[i] = 1.0f;
            } else {
                // x != 1 case
                float y = floorf(state.d[i] / state.b[i]);
                
                // Continue dividing by b until we find the right digit position
                while (xf <= y) {
                    y = floorf(y / state.b[i]);
                }
                
                state.n[i] = floorf((state.b[i] + 1.0f) * y) - xf;
            }
            
            result[i] = state.n[i] / state.d[i];
        }
    }

    __global__ void init_rng(hiprandState* states, unsigned long seed, int num_rng_states) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= num_rng_states) return;
        hiprand_init(seed + idx, idx, 0, &states[idx]);
    }

    template <typename Robot>
    __global__ void init_halton(HaltonState<Robot>* states, hiprandState* cr_states) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= d_settings.num_new_configs) return;
        int skip = (hiprand_uniform(&cr_states[idx]) * 50000.0f);
        if (idx == 0) skip = 0;
        // if (idx == 1) skip = 100000;
        halton_initialize(states[idx], skip, cr_states[idx], idx);
    }

    __device__ inline void print_config(float *config, int dim) {
        for (int i = 0; i < dim; i++) {
            printf("%f ,", config[i]);
        }
        printf("\n");
    }

    inline void setup_environment_on_device(ppln::collision::Environment<float> *&d_env, 
                                      const ppln::collision::Environment<float> &h_env) {
        // First allocate the environment struct
        hipMalloc(&d_env, sizeof(ppln::collision::Environment<float>));
        
        // Initialize struct to zeros first
        hipMemset(d_env, 0, sizeof(ppln::collision::Environment<float>));

        // Handle each primitive type separately
        if (h_env.num_spheres > 0) {
            // Allocate and copy spheres array
            ppln::collision::Sphere<float> *d_spheres;
            hipMalloc(&d_spheres, sizeof(ppln::collision::Sphere<float>) * h_env.num_spheres);
            hipMemcpy(d_spheres, h_env.spheres, 
                    sizeof(ppln::collision::Sphere<float>) * h_env.num_spheres, 
                    hipMemcpyHostToDevice);
            
            // Update the struct fields directly
            hipMemcpy(&(d_env->spheres), &d_spheres, sizeof(ppln::collision::Sphere<float>*), 
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_spheres), &h_env.num_spheres, sizeof(unsigned int), 
                    hipMemcpyHostToDevice);
        }

        if (h_env.num_capsules > 0) {
            ppln::collision::Capsule<float> *d_capsules;
            hipMalloc(&d_capsules, sizeof(ppln::collision::Capsule<float>) * h_env.num_capsules);
            hipMemcpy(d_capsules, h_env.capsules,
                    sizeof(ppln::collision::Capsule<float>) * h_env.num_capsules,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->capsules), &d_capsules, sizeof(ppln::collision::Capsule<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_capsules), &h_env.num_capsules, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }

        // Repeat for each primitive type...
        if (h_env.num_z_aligned_capsules > 0) {
            ppln::collision::Capsule<float> *d_z_capsules;
            hipMalloc(&d_z_capsules, sizeof(ppln::collision::Capsule<float>) * h_env.num_z_aligned_capsules);
            hipMemcpy(d_z_capsules, h_env.z_aligned_capsules,
                    sizeof(ppln::collision::Capsule<float>) * h_env.num_z_aligned_capsules,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->z_aligned_capsules), &d_z_capsules, sizeof(ppln::collision::Capsule<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_z_aligned_capsules), &h_env.num_z_aligned_capsules, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }

        if (h_env.num_cylinders > 0) {
            ppln::collision::Cylinder<float> *d_cylinders;
            hipMalloc(&d_cylinders, sizeof(ppln::collision::Cylinder<float>) * h_env.num_cylinders);
            hipMemcpy(d_cylinders, h_env.cylinders,
                    sizeof(ppln::collision::Cylinder<float>) * h_env.num_cylinders,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->cylinders), &d_cylinders, sizeof(ppln::collision::Cylinder<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_cylinders), &h_env.num_cylinders, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }

        if (h_env.num_cuboids > 0) {
            ppln::collision::Cuboid<float> *d_cuboids;
            hipMalloc(&d_cuboids, sizeof(ppln::collision::Cuboid<float>) * h_env.num_cuboids);
            hipMemcpy(d_cuboids, h_env.cuboids,
                    sizeof(ppln::collision::Cuboid<float>) * h_env.num_cuboids,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->cuboids), &d_cuboids, sizeof(ppln::collision::Cuboid<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_cuboids), &h_env.num_cuboids, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }

        if (h_env.num_z_aligned_cuboids > 0) {
            ppln::collision::Cuboid<float> *d_z_cuboids;
            hipMalloc(&d_z_cuboids, sizeof(ppln::collision::Cuboid<float>) * h_env.num_z_aligned_cuboids);
            hipMemcpy(d_z_cuboids, h_env.z_aligned_cuboids,
                    sizeof(ppln::collision::Cuboid<float>) * h_env.num_z_aligned_cuboids,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->z_aligned_cuboids), &d_z_cuboids, sizeof(ppln::collision::Cuboid<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_z_aligned_cuboids), &h_env.num_z_aligned_cuboids, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }
    }


    inline void cleanup_environment_on_device(ppln::collision::Environment<float> *d_env, 
                                        const ppln::collision::Environment<float> &h_env) {
        // Get the pointers from device struct before freeing
        ppln::collision::Sphere<float> *d_spheres = nullptr;
        ppln::collision::Capsule<float> *d_capsules = nullptr;
        ppln::collision::Capsule<float> *d_z_capsules = nullptr;
        ppln::collision::Cylinder<float> *d_cylinders = nullptr;
        ppln::collision::Cuboid<float> *d_cuboids = nullptr;
        ppln::collision::Cuboid<float> *d_z_cuboids = nullptr;

        // Copy each pointer from device memory
        if (h_env.num_spheres > 0) {
            hipMemcpy(&d_spheres, &(d_env->spheres), sizeof(ppln::collision::Sphere<float>*), hipMemcpyDeviceToHost);
            hipFree(d_spheres);
        }
        
        if (h_env.num_capsules > 0) {
            hipMemcpy(&d_capsules, &(d_env->capsules), sizeof(ppln::collision::Capsule<float>*), hipMemcpyDeviceToHost);
            hipFree(d_capsules);
        }
        
        if (h_env.num_z_aligned_capsules > 0) {
            hipMemcpy(&d_z_capsules, &(d_env->z_aligned_capsules), sizeof(ppln::collision::Capsule<float>*), hipMemcpyDeviceToHost);
            hipFree(d_z_capsules);
        }
        
        if (h_env.num_cylinders > 0) {
            hipMemcpy(&d_cylinders, &(d_env->cylinders), sizeof(ppln::collision::Cylinder<float>*), hipMemcpyDeviceToHost);
            hipFree(d_cylinders);
        }
        
        if (h_env.num_cuboids > 0) {
            hipMemcpy(&d_cuboids, &(d_env->cuboids), sizeof(ppln::collision::Cuboid<float>*), hipMemcpyDeviceToHost);
            hipFree(d_cuboids);
        }
        
        if (h_env.num_z_aligned_cuboids > 0) {
            hipMemcpy(&d_z_cuboids, &(d_env->z_aligned_cuboids), sizeof(ppln::collision::Cuboid<float>*), hipMemcpyDeviceToHost);
            hipFree(d_z_cuboids);
        }

        // Finally free the environment struct itself
        hipFree(d_env);
    }

    __global__ void reset_device_variables_kernel() {
        solved = 0;
        
        path_size[0] = 0;
        path_size[1] = 0;
        
        for (int tree = 0; tree < 2; tree++) {
            for (int i = 0; i < 500; i++) {
                path[tree][i] = 0.0f;
            }
        }
        
        cost = 0.0f;
        reached_goal_idx = 0;
        }

    void reset_device_variables() {
        reset_device_variables_kernel<<<1, 1>>>();
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(error));
        }
    }
    
    /* each warp handles a separate RRTC tree, 32 threads grow the tree 1 at a time*/
    // 64 blocks, 128 threads per block, 4 warps per block, 256 trees total
    // 
    template <typename Robot>
    __global__ void
    __launch_bounds__(128, 4)
    rrtc(
        float **nodes,
        int **parents,
        // float **radii,
        HaltonState<Robot> *halton_states,
        hiprandState *rng_states,
        ppln::collision::Environment<float> *env,
        int num_goals
    )
    {
        static constexpr auto dim = Robot::dimension;
        // const int tid = blockIdx.x * blockDim.x + threadIdx.x; // global thread id
        const int lid = threadIdx.x % 32; // lane id
        const int wid = threadIdx.x / 32; // warp id
        const int bid = blockIdx.x; // block id
        const int tree_idx = bid * 4 + wid;
        // if (lid == 0) printf("initialized tree_idx: %d\n", tree_idx);
        // if (tree_idx >= NUM_TREES) return;
        __shared__ float config[4][dim]; // new config for each tree
        __shared__ int free_index[4][2]; // free indexes for each tree
        __shared__ float delta[4][dim]; // delta for each tree
        // __shared__ float var_cache[256][10]; // cache for forward kinematics
        int t_tree_id = 0;
        int o_tree_id = 1 - t_tree_id;
        float *t_nodes = nodes[t_tree_id] + (tree_idx * MAX_TREE_SIZE * dim);
        float *o_nodes = nodes[o_tree_id] + (tree_idx * MAX_TREE_SIZE * dim);
        int *t_parents = parents[t_tree_id] + (tree_idx * MAX_TREE_SIZE);
        int *o_parents = parents[o_tree_id] + (tree_idx * MAX_TREE_SIZE);
        unsigned int iter = 0;
        if (threadIdx.x < 4) {
            free_index[threadIdx.x][0] = 1;
            free_index[threadIdx.x][1] = num_goals;
        }
        __syncthreads();
        while (solved == 0) {
            // __syncthreads();
            iter ++;
            if (iter > d_settings.max_iters) {
                if (lid == 0) {
                    printf("DBG: Tree %d exceeded max iterations\n", tree_idx);
                }
                atomicCAS((int *)&solved, 0, -1);
            }
            // __syncthreads();

            if (d_settings.balance == 2) {
                float ratio = abs(free_index[wid][t_tree_id] - free_index[wid][o_tree_id]) / (float) free_index[wid][t_tree_id];
                if (ratio < d_settings.tree_ratio)
                {
                    t_tree_id = 1 - t_tree_id;
                    o_tree_id = 1 - t_tree_id;
                    float *temp_nodes = t_nodes;
                    t_nodes = o_nodes;
                    o_nodes = temp_nodes;
                    int *temp_parents = t_parents;
                    t_parents = o_parents;
                    o_parents = temp_parents;
                }
            }
            else { // swap every iteration
                t_tree_id = 1 - t_tree_id;
                o_tree_id = 1 - t_tree_id;
                float *temp_nodes = t_nodes;
                t_nodes = o_nodes;
                o_nodes = temp_nodes;
                int *temp_parents = t_parents;
                t_parents = o_parents;
                o_parents = temp_parents;
            }
            // __syncthreads();


            /* sample configuration */
            if (lid == 0) {
                // printf("DBG: Tree %d sampling configuration\n", tree_idx);
                halton_next(halton_states[tree_idx], (float *)config[wid]);
                Robot::scale_cfg((float *)config[wid]);
                // printf("DBG: Tree %d sampled config: ", tree_idx);
                // print_config(config[wid], dim);
            }
            // __syncthreads();


            /* find nearest neighbor */
            float min_dist = FLT_MAX;
            int min_index = -1;
            float dist;
            // __syncthreads();
            // if (lid == 0) printf("free_index: %d, %d | tree_idx: %d\n", free_index[wid][0], free_index[wid][1], tree_idx);
            // __syncthreads();
            for (unsigned int i = lid; i < free_index[wid][t_tree_id] && i < MAX_TREE_SIZE; i += 32) {
                // printf("i: %d, tree_idx: %d\n", i, tree_idx);
                // __syncthreads();
                dist = device_utils::sq_l2_dist(&t_nodes[i * dim], config[wid], dim);
                if (dist < min_dist) {
                    min_dist = dist;
                    min_index = i;
                }
            }
            __syncwarp();
            // __syncthreads();
            // if (lid == 0) printf("here3, tree_idx: %d\n", tree_idx);
            // __syncthreads();
            for (unsigned int offset = 16; offset > 0; offset /= 2) {
                float other_dist = __shfl_down_sync(FULL_MASK, min_dist, offset);
                int other_index = __shfl_down_sync(FULL_MASK, min_index, offset);
                if (other_dist < min_dist) {
                    min_dist = other_dist;
                    min_index = other_index;
                }
            }
            // __syncthreads();
            // if (lid == 0) printf("min_dist: %f, tree_idx: %d\n", min_dist, tree_idx);
            // __syncthreads();
            __syncwarp();
            min_dist = __shfl_sync(FULL_MASK, min_dist, 0);
            min_index = __shfl_sync(FULL_MASK, min_index, 0);
            // __syncwarp();
            min_dist = sqrt(min_dist);
            // printf("min_dist: %f\n", min_dist);
            // printf("min_index: %d\n", min_index);
            float scale = min(1.0f, d_settings.range / min_dist);
            float *nearest_node = &t_nodes[min_index * dim];
            // if (lid == 0) {
            //     printf("nearest node: ");
            //     print_config(nearest_node, dim);
            // }
            // __syncthreads();
            if (lid < dim) {
                // printf("setting config and delta for lid: %d, tree_idx: %d\n", lid, tree_idx);
                // __syncthreads();
                config[wid][lid] = (1 - scale) * nearest_node[lid] + scale * config[wid][lid];
                // printf("here4");
                // __syncthreads();
                delta[wid][lid] = (config[wid][lid] - nearest_node[lid]) / d_settings.granularity;
            }
            __syncwarp();
            // if (lid == 0) {
            //     printf("here4.5, tree_idx: %d\n", tree_idx);
            // }
            // __syncthreads();
            /* validate edge to config*/
            float interp_cfg[dim];
            bool config_in_collision = false;
            for (unsigned int i = lid; i < d_settings.granularity; i+=32) {
                // if (lid == 0) {
                //     printf("i: %d\n", i);
                // }
                // printf("lid: %d, wid: %d\n", lid, wid);
                for (unsigned int j = 0; j < dim; j++) {
                    interp_cfg[j] = nearest_node[j] + delta[wid][j] * (i + 1);
                }
                // volatile unsigned int local_cc_result = 0;
                // if (lid == 0) printf("here5, tree_idx: %d\n", tree_idx);
                // __syncthreads();
                config_in_collision = not ppln::collision::fkcc<Robot>(interp_cfg, env, threadIdx.x);
                // __syncthreads();
                // if (lid == 0) printf("here6, tree_idx: %d\n", tree_idx);
                // printf("tree_idx: %d, lid: %d", tree_idx, lid);
                // __syncthreads();
                __syncwarp();
                for (unsigned int offset = 16; offset > 0; offset /= 2) {
                    // if (lid == 0) printf("here7\n");
                    bool other_config_in_collision = __shfl_down_sync(FULL_MASK, config_in_collision, offset);
                    // printf("other_config_in_collision: %d\n", other_config_in_collision);
                    config_in_collision = config_in_collision || other_config_in_collision;
                }
                __syncwarp();
                // if (lid == 0) printf("config_in_collision: %d\n", config_in_collision);
                config_in_collision = __shfl_sync(FULL_MASK, config_in_collision, 0);
                if (config_in_collision) break;
            }
            // if (lid == 0) printf("here8\n");
            /* add new config to this tree */
            if (!config_in_collision) {
                // printf("lid: %d\n", lid);
                unsigned int index = free_index[wid][t_tree_id];
                if (lid == 0) {
                    if (index >= MAX_TREE_SIZE) {
                        atomicCAS((int *)&solved, 0, -1);
                    }
                    else {
                        free_index[wid][t_tree_id] = index + 1;
                        t_parents[index] = min_index;
                    }
                }
                // if (lid == 0) printf("here9\n");
                if (lid < dim) {
                    t_nodes[index * dim + lid] = config[wid][lid];
                }
                __syncwarp();
                // if (lid == 0) {
                //     printf("Added new config: ");
                //     print_config(&t_nodes[index * dim], dim);
                // }
                /* connect (find nearest node in other tree and attempt connection) */
                float min_dist = FLT_MAX;
                int min_index = -1;
                for (unsigned int i = lid; i < free_index[wid][o_tree_id]; i += 32) {
                    dist = device_utils::sq_l2_dist(&o_nodes[i * dim], config[wid], dim);
                    if (dist < min_dist) {
                        min_dist = dist;
                        min_index = i;
                    }
                }
                __syncwarp();
                for (unsigned int offset = 16; offset > 0; offset /= 2) {
                    float other_dist = __shfl_down_sync(FULL_MASK, min_dist, offset);
                    int other_index = __shfl_down_sync(FULL_MASK, min_index, offset);
                    if (other_dist < min_dist) {
                        min_dist = other_dist;
                        min_index = other_index;
                    }
                }
                min_dist = __shfl_sync(FULL_MASK, min_dist, 0);
                min_index = __shfl_sync(FULL_MASK, min_index, 0);
                min_dist = sqrt(min_dist);
                int n_extensions = ceil(min_dist / d_settings.range);
                // if (lid == 0) printf("here11\n");
                if (lid < dim) {
                    delta[wid][lid] = (o_nodes[min_index * dim + lid] - config[wid][lid]) / (float) n_extensions;
                }
                __syncwarp();
                // if (lid == 0) {
                //     printf("n_extensions: %d\n", n_extensions);
                //     printf("delta: ");
                //     print_config(delta[wid], dim);
                // }
                /* extend as far as we can to NN in opposite tree */
                int i_extensions = 0;
                int extension_parent_idx = index;
                while (i_extensions < n_extensions) {
                    // if (lid == 0) printf("i_extensions: %d\n", i_extensions);
                    /* collision check the extension */
                    bool config_in_collision = false;
                    for (unsigned int i = lid; i < d_settings.granularity; i+=32) {
                        // if (lid == 0) {
                        //     printf("i: %d\n", i);
                        // }
                        // printf("lid: %d, wid: %d\n", lid, wid);
                        for (unsigned int j = 0; j < dim; j++) {
                            interp_cfg[j] = config[wid][j] + ((delta[wid][j] / d_settings.granularity) * (i + 1));
                        }
                        // volatile unsigned int local_cc_result = 0;
                        // if (lid == 0) printf("here13\n");
                        config_in_collision = not ppln::collision::fkcc<Robot>(interp_cfg, env, threadIdx.x);
                        // if (lid == 0) printf("here14\n");
                        __syncwarp();
                        for (unsigned int offset = 16; offset > 0; offset /= 2) {
                            // if (lid == 0) printf("here15\n");
                            bool other_config_in_collision = __shfl_down_sync(FULL_MASK, config_in_collision, offset);
                            config_in_collision = config_in_collision || other_config_in_collision;
                        }
                        __syncwarp();
                        // if (lid == 0) printf("config_in_collision (ext): %d\n", config_in_collision);
                        config_in_collision = __shfl_sync(FULL_MASK, config_in_collision, 0);
                        if (config_in_collision) break;
                    }
                    if (config_in_collision) break;
                    /* add extended config to tree */
                    index = free_index[wid][t_tree_id];
                    if (lid == 0) {
                        // printf("extending: %d\n", i_extensions);
                        if (index >= MAX_TREE_SIZE) {
                            atomicCAS((int *)&solved, 0, -1);
                        }
                        else {
                            free_index[wid][t_tree_id] = index + 1;
                            t_parents[index] = extension_parent_idx;
                            extension_parent_idx = index;
                        }
                    }
                    if (lid < dim) {
                        config[wid][lid] = config[wid][lid] + delta[wid][lid];
                        t_nodes[index * dim + lid] = config[wid][lid];
                    }
                    __syncwarp();
                    // if (lid == 0) {
                    //     printf("Added config: ");
                    //     print_config(&t_nodes[index * dim], dim);
                    // }
                    // __syncwarp();
                    i_extensions ++;
                }
                /* check if we reached the goal */
                if (i_extensions == n_extensions) {
                    if (lid == 0 && atomicCAS((int *)&solved, 0, 1) == 0) {
                        // printf("ENTERED SOLUTION CODE\n");
                        // printf("tree_idx: %d\n", tree_idx);
                        // trace back to the start and goal.
                        int current = index;
                        int parent;
                        int t_path_size = 0;
                        int o_path_size = 0;
                        while (t_parents[current] != current) {
                            parent = t_parents[current];
                            cost += device_utils::l2_dist(&t_nodes[current * dim], &t_nodes[parent * dim], dim);
                            for (int i = 0; i < dim; i++) path[t_tree_id][t_path_size * dim + i] = t_nodes[current * dim + i];
                            t_path_size++;
                            current = parent;
                            
                        }
                        if (t_tree_id == 1) reached_goal_idx = current;
                        current = min_index;
                        while (o_parents[current] != current) {
                            parent = o_parents[current];
                            cost += device_utils::l2_dist(&o_nodes[current * dim], &o_nodes[parent * dim], dim);
                            for (int i = 0; i < dim; i++) path[o_tree_id][o_path_size * dim + i] = o_nodes[current * dim + i];
                            o_path_size++;
                            current = parent;
                        }
                        if (t_tree_id == 0) reached_goal_idx = current;
                        // printf("t_path_size: %d, o_path_size: %d\n", t_path_size, o_path_size);
                        path_size[t_tree_id] = t_path_size;
                        path_size[o_tree_id] = o_path_size;
                        solved_iters = iter;
                        solved_free_index[0] = free_index[wid][0];
                        solved_free_index[1] = free_index[wid][1];
                    }
                    __syncthreads();
                }
            }
            if (solved != 0) break;
        }
    }



    template <typename Robot>
    PlannerResult<Robot> solve(
        typename Robot::Configuration &start,
        std::vector<typename Robot::Configuration> &goals,
        ppln::collision::Environment<float> &h_environment,
        pRRTC_settings &settings
    ) 
    {
        // std::cout << "here" << std::endl;
        settings.max_samples = MAX_TREE_SIZE * NUM_TREES;
        hipSetDevice(1);
        // std::cout << "here1" << std::endl;
        auto start_time = std::chrono::steady_clock::now();
        static constexpr auto dim = Robot::dimension;
        std::size_t start_index = 0;

        
        PlannerResult<Robot> res;
        // copy data to GPU
        hipMemcpyToSymbol(HIP_SYMBOL(d_settings), &settings, sizeof(settings));

        // std::cout << "here2" << std::endl;
        int num_goals = goals.size();
        float *nodes[2];
        int *parents[2];
        float *radii[2];
        float **d_nodes;
        int **d_parents;
        float **d_radii;
        hipMalloc(&d_nodes, 2 * sizeof(float*));
        hipMalloc(&d_parents, 2 * sizeof(int*));
        // hipMalloc(&d_radii, 2 * sizeof(float*));
        const std::size_t config_size = dim * sizeof(float);

        for (int i = 0; i < 2; i++) {
            hipMalloc(&nodes[i], settings.max_samples * config_size);
            hipMalloc(&parents[i], settings.max_samples * sizeof(int));
            // hipMalloc(&radii[i], settings.max_samples * sizeof(float));
        }
        hipMemcpy(d_nodes, nodes, 2 * sizeof(float*), hipMemcpyHostToDevice);
        hipMemcpy(d_parents, parents, 2 * sizeof(int*), hipMemcpyHostToDevice);
        // hipMemcpy(d_radii, radii, 2 * sizeof(float*), hipMemcpyHostToDevice);

        // std::cout << "here3" << std::endl;

        // add start and goal configurations to trees
        for (int i = 0; i < NUM_TREES; i++) {
            int global_idx = i * MAX_TREE_SIZE;
            hipMemcpy(nodes[0] + (global_idx * dim), start.data(), config_size, hipMemcpyHostToDevice);
            hipMemcpy(parents[0] + global_idx, &start_index, sizeof(int), hipMemcpyHostToDevice);

            hipMemcpy(nodes[1] + (global_idx * dim), goals.data(), config_size * num_goals, hipMemcpyHostToDevice);
            std::vector<int> parents_b_init(num_goals);
            iota(parents_b_init.begin(), parents_b_init.end(), 0); // consecutive integers from 0 ... num_goals - 1
            hipMemcpy(parents[1] + global_idx, parents_b_init.data(), sizeof(int) * num_goals, hipMemcpyHostToDevice);

            // std::vector<float> radii_init(num_goals, FLT_MAX);
            // hipMemcpy(radii[0] + global_idx, radii_init.data(), sizeof(float), hipMemcpyHostToDevice);
            // hipMemcpy(radii[1] + global_idx, radii_init.data(), sizeof(float) * num_goals, hipMemcpyHostToDevice);
        }
        cudaCheckError(hipGetLastError());
        
        // create a hiprandState for each thread -> holds state of RNG for each thread seperately
        // For growing the tree we will create NUM_NEW_CONFIGS threads
        hiprandState *rng_states;
        int num_rng_states = NUM_TREES * 32;
        hipMalloc(&rng_states, num_rng_states * sizeof(hiprandState));
        int numBlocks = (num_rng_states + BLOCK_SIZE - 1) / BLOCK_SIZE;
        init_rng<<<numBlocks, BLOCK_SIZE>>>(rng_states, 1, num_rng_states);

        HaltonState<Robot> *halton_states;
        hipMalloc(&halton_states, NUM_TREES * sizeof(HaltonState<Robot>));
        int numBlocks1 = (NUM_TREES + BLOCK_SIZE - 1) / BLOCK_SIZE;
        init_halton<Robot><<<numBlocks1, BLOCK_SIZE>>>(halton_states, rng_states);

        
        // std::cout << "here5" << std::endl;
        // allocate for obstacles
        ppln::collision::Environment<float> *env;
        setup_environment_on_device(env, h_environment);
        // std::cout << "here6" << std::endl;
        cudaCheckError(hipGetLastError());
        // Setup pinned memory for signaling
        int *h_solved;
        int current_samples[2];
        int h_solved_iters = -1;
        hipHostMalloc(&h_solved, sizeof(int));  // Pinned memory
        *h_solved = -1;
        // std::cout << "here7" << std::endl;
        auto kernel_start_time = std::chrono::steady_clock::now();
        rrtc<Robot><<<64, 128>>> (
            d_nodes,
            d_parents,
            // d_radii,
            halton_states,
            rng_states,
            env,
            num_goals
        );
        hipDeviceSynchronize();
        res.kernel_ns = get_elapsed_nanoseconds(kernel_start_time);
        cudaCheckError(hipGetLastError());
        
        // std::cout << "here8" << std::endl;
        

        hipMemcpyFromSymbol(h_solved, HIP_SYMBOL(solved), sizeof(int), 0, hipMemcpyDeviceToHost);
        hipMemcpyFromSymbol(&h_solved_iters, HIP_SYMBOL(solved_iters), sizeof(int), 0, hipMemcpyDeviceToHost);
        hipMemcpyFromSymbol(current_samples, HIP_SYMBOL(solved_free_index), sizeof(int) * 2, 0, hipMemcpyDeviceToHost);
        // currently, iteration count is not copied because each block may have different iteration count
        if (*h_solved!=1) *h_solved=0;
        std::cout << "current_samples: start: " << current_samples[0] << ", goal: " << current_samples[1] << "\n";
        std::cout << "solved iters: " << h_solved_iters << "\n";
        // res.start_tree_size = current_samples[0];
        // res.goal_tree_size = current_samples[1];
        Robot::print_robot_config(start);
        Robot::print_robot_config(goals[0]);
        if (*h_solved) {
            std::cout << "solved!\n";
            int h_path_size[2];
            float h_paths[2][500];
            float h_cost;
            int h_reached_goal_idx;
            hipMemcpyFromSymbol(h_path_size, HIP_SYMBOL(path_size), sizeof(int) * 2, 0, hipMemcpyDeviceToHost);
            hipMemcpyFromSymbol(h_paths, HIP_SYMBOL(path), sizeof(float) * 2 * 500, 0, hipMemcpyDeviceToHost);
            hipMemcpyFromSymbol(&h_cost, HIP_SYMBOL(cost), sizeof(float), 0, hipMemcpyDeviceToHost);
            hipMemcpyFromSymbol(&h_reached_goal_idx, HIP_SYMBOL(reached_goal_idx), sizeof(int), 0, hipMemcpyDeviceToHost);
            cudaCheckError(hipGetLastError());
            // Robot::print_robot_config(goals[h_reached_goal_idx]);
            res.path.emplace_back(goals[h_reached_goal_idx]);
            typename Robot::Configuration config;
            for (int i = h_path_size[1] - 1; i >= 0; i--) {
                std::copy_n(h_paths[1] + i * dim, dim, config.begin());
                res.path.emplace_back(config);
                // print_cfg_ptr<Robot>(&h_paths[1][i * dim]);
            }
            for (int i = 0; i < h_path_size[0]; i++) {
                std::copy_n(h_paths[0] + i * dim, dim, config.begin());
                res.path.emplace_back(config);
                // print_cfg_ptr<Robot>(&h_paths[0][i * dim]);
            }
            // Robot::print_robot_config(start);
            res.path.emplace_back(start);
            res.cost = h_cost;
            res.path_length = (h_path_size[0] + h_path_size[1]);
            std::cout << "cost: " << res.cost << "\n";
        }
        res.solved = (*h_solved) != 0;
        res.iters = h_solved_iters;
        
        cleanup_environment_on_device(env, h_environment);
        reset_device_variables();
        hipFree(nodes[0]);
        hipFree(nodes[1]);
        hipFree(parents[0]);
        hipFree(parents[1]);
        // hipFree(radii[0]);
        // hipFree(radii[1]);
        hipFree(rng_states);
        hipFree(halton_states);
        hipFree(d_nodes);
        hipFree(d_parents);
        // hipFree(d_radii);
        hipHostFree(h_solved);
        cudaCheckError(hipGetLastError());
        res.wall_ns = get_elapsed_nanoseconds(start_time);
        hipDeviceReset();
        return res;
    }

    template PlannerResult<typename ppln::robots::Sphere> solve<ppln::robots::Sphere>(std::array<float, 3>&, std::vector<std::array<float, 3>>&, ppln::collision::Environment<float>&, pRRTC_settings&);
    template PlannerResult<typename ppln::robots::Panda> solve<ppln::robots::Panda>(std::array<float, 7>&, std::vector<std::array<float, 7>>&, ppln::collision::Environment<float>&, pRRTC_settings&);
    template PlannerResult<typename ppln::robots::Fetch> solve<ppln::robots::Fetch>(std::array<float, 8>&, std::vector<std::array<float, 8>>&, ppln::collision::Environment<float>&, pRRTC_settings&);
}
