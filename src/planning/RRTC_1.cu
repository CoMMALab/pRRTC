#include "hip/hip_runtime.h"
#include "Planners.hh"
#include "Robots.hh"
#include "utils.cuh"
#include "src/collision/environment.hh"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <vector>
#include <iostream>
#include <cassert>
#include <algorithm>
#include <numeric>

/*
New attempt to rewrite RRTC from scratch.
I realize now that everything can be organized into one kernel with launch parameters <<<NUM_NEW_CONFIGS, GRANULARITY>>>
I am going to try this approach and see how it goes.
*/


namespace pRRTC {
    using namespace ppln;
    __device__ volatile int solved = 0;
    __device__ volatile int atomic_free_index[2]; // separate for tree_a and tree_b
    __device__ float path[2][500]; // solution path segments for tree_a, and tree_b
    __device__ int path_size[2] = {0, 0};
    __device__ float cost = 0.0;
    __device__ int reached_goal_idx = 0;

    constexpr int MAX_SAMPLES = 1000000;
    constexpr int MAX_ITERS = 1000000;
    constexpr int NUM_NEW_CONFIGS = 400;
    constexpr int GRANULARITY = 256;
    constexpr float RRT_RADIUS = 1.0;
    constexpr float TREE_RATIO = 0.5;
    constexpr bool balance = true;

    // threads per block for sample_edges and grow_tree
    constexpr int BLOCK_SIZE = 256;

    // Constants
    __constant__ float primes[16] = {
        3.f, 5.f, 7.f, 11.f, 13.f, 17.f, 19.f, 23.f,
        29.f, 31.f, 37.f, 41.f, 43.f, 47.f, 53.f, 59.f
    };

    template<typename Robot>
    struct HaltonState {
        float b[Robot::dimension];   // bases
        float n[Robot::dimension];   // numerators
        float d[Robot::dimension];   // denominators
    };

    template<typename Robot>
    __device__ void halton_initialize(HaltonState<Robot>& state, size_t skip_iterations) {
        // Initialize bases from primes
        for (size_t i = 0; i < Robot::dimension; i++) {
            state.b[i] = primes[i];
            state.n[i] = 0.0f;
            state.d[i] = 1.0f;
        }
        
        // Skip iterations if requested
        volatile float temp_result[Robot::dimension];
        for (size_t i = 0; i < skip_iterations; i++) {
            halton_next(state, (float *)temp_result);
        }
    }

    template<typename Robot>
    __device__ void halton_next(HaltonState<Robot>& state, float* result) {
        for (size_t i = 0; i < Robot::dimension; i++) {
            float xf = state.d[i] - state.n[i];
            bool x_eq_1 = (xf == 1.0f);
            
            if (x_eq_1) {
                // x == 1 case
                state.d[i] = floorf(state.d[i] * state.b[i]);
                state.n[i] = 1.0f;
            } else {
                // x != 1 case
                float y = floorf(state.d[i] / state.b[i]);
                
                // Continue dividing by b until we find the right digit position
                while (xf <= y) {
                    y = floorf(y / state.b[i]);
                }
                
                state.n[i] = floorf((state.b[i] + 1.0f) * y) - xf;
            }
            
            result[i] = state.n[i] / state.d[i];
        }
    }

    __global__ void init_rng(hiprandState* states, unsigned long seed, int num_rng_states) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= num_rng_states) return;
        hiprand_init(seed + idx, idx, 0, &states[idx]);
    }

    template <typename Robot>
    __global__ void init_halton(HaltonState<Robot>* states, hiprandState* cr_states) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= NUM_NEW_CONFIGS) return;
        int skip = (hiprand_uniform(&cr_states[idx]) * 500000.0f);
        if (idx == 0) skip = 0;
        if (idx == 1) skip = 100000;
        halton_initialize(states[idx], skip);
    }

    __device__ inline void print_config(float *config, int dim) {
        for (int i = 0; i < dim; i++) {
            printf("%f ,", config[i]);
        }
        printf("\n");
    }

    inline void setup_environment_on_device(ppln::collision::Environment<float> *&d_env, 
                                      const ppln::collision::Environment<float> &h_env) {
        // First allocate the environment struct
        hipMalloc(&d_env, sizeof(ppln::collision::Environment<float>));
        
        // Initialize struct to zeros first
        hipMemset(d_env, 0, sizeof(ppln::collision::Environment<float>));

        // Handle each primitive type separately
        if (h_env.num_spheres > 0) {
            // Allocate and copy spheres array
            ppln::collision::Sphere<float> *d_spheres;
            hipMalloc(&d_spheres, sizeof(ppln::collision::Sphere<float>) * h_env.num_spheres);
            hipMemcpy(d_spheres, h_env.spheres, 
                    sizeof(ppln::collision::Sphere<float>) * h_env.num_spheres, 
                    hipMemcpyHostToDevice);
            
            // Update the struct fields directly
            hipMemcpy(&(d_env->spheres), &d_spheres, sizeof(ppln::collision::Sphere<float>*), 
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_spheres), &h_env.num_spheres, sizeof(unsigned int), 
                    hipMemcpyHostToDevice);
        }

        if (h_env.num_capsules > 0) {
            ppln::collision::Capsule<float> *d_capsules;
            hipMalloc(&d_capsules, sizeof(ppln::collision::Capsule<float>) * h_env.num_capsules);
            hipMemcpy(d_capsules, h_env.capsules,
                    sizeof(ppln::collision::Capsule<float>) * h_env.num_capsules,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->capsules), &d_capsules, sizeof(ppln::collision::Capsule<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_capsules), &h_env.num_capsules, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }

        // Repeat for each primitive type...
        if (h_env.num_z_aligned_capsules > 0) {
            ppln::collision::Capsule<float> *d_z_capsules;
            hipMalloc(&d_z_capsules, sizeof(ppln::collision::Capsule<float>) * h_env.num_z_aligned_capsules);
            hipMemcpy(d_z_capsules, h_env.z_aligned_capsules,
                    sizeof(ppln::collision::Capsule<float>) * h_env.num_z_aligned_capsules,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->z_aligned_capsules), &d_z_capsules, sizeof(ppln::collision::Capsule<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_z_aligned_capsules), &h_env.num_z_aligned_capsules, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }

        if (h_env.num_cylinders > 0) {
            ppln::collision::Cylinder<float> *d_cylinders;
            hipMalloc(&d_cylinders, sizeof(ppln::collision::Cylinder<float>) * h_env.num_cylinders);
            hipMemcpy(d_cylinders, h_env.cylinders,
                    sizeof(ppln::collision::Cylinder<float>) * h_env.num_cylinders,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->cylinders), &d_cylinders, sizeof(ppln::collision::Cylinder<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_cylinders), &h_env.num_cylinders, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }

        if (h_env.num_cuboids > 0) {
            ppln::collision::Cuboid<float> *d_cuboids;
            hipMalloc(&d_cuboids, sizeof(ppln::collision::Cuboid<float>) * h_env.num_cuboids);
            hipMemcpy(d_cuboids, h_env.cuboids,
                    sizeof(ppln::collision::Cuboid<float>) * h_env.num_cuboids,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->cuboids), &d_cuboids, sizeof(ppln::collision::Cuboid<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_cuboids), &h_env.num_cuboids, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }

        if (h_env.num_z_aligned_cuboids > 0) {
            ppln::collision::Cuboid<float> *d_z_cuboids;
            hipMalloc(&d_z_cuboids, sizeof(ppln::collision::Cuboid<float>) * h_env.num_z_aligned_cuboids);
            hipMemcpy(d_z_cuboids, h_env.z_aligned_cuboids,
                    sizeof(ppln::collision::Cuboid<float>) * h_env.num_z_aligned_cuboids,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->z_aligned_cuboids), &d_z_cuboids, sizeof(ppln::collision::Cuboid<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_z_aligned_cuboids), &h_env.num_z_aligned_cuboids, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }
    }


    inline void cleanup_environment_on_device(ppln::collision::Environment<float> *d_env, 
                                        const ppln::collision::Environment<float> &h_env) {
        // Get the pointers from device struct before freeing
        ppln::collision::Sphere<float> *d_spheres = nullptr;
        ppln::collision::Capsule<float> *d_capsules = nullptr;
        ppln::collision::Capsule<float> *d_z_capsules = nullptr;
        ppln::collision::Cylinder<float> *d_cylinders = nullptr;
        ppln::collision::Cuboid<float> *d_cuboids = nullptr;
        ppln::collision::Cuboid<float> *d_z_cuboids = nullptr;

        // Copy each pointer from device memory
        if (h_env.num_spheres > 0) {
            hipMemcpy(&d_spheres, &(d_env->spheres), sizeof(ppln::collision::Sphere<float>*), hipMemcpyDeviceToHost);
            hipFree(d_spheres);
        }
        
        if (h_env.num_capsules > 0) {
            hipMemcpy(&d_capsules, &(d_env->capsules), sizeof(ppln::collision::Capsule<float>*), hipMemcpyDeviceToHost);
            hipFree(d_capsules);
        }
        
        if (h_env.num_z_aligned_capsules > 0) {
            hipMemcpy(&d_z_capsules, &(d_env->z_aligned_capsules), sizeof(ppln::collision::Capsule<float>*), hipMemcpyDeviceToHost);
            hipFree(d_z_capsules);
        }
        
        if (h_env.num_cylinders > 0) {
            hipMemcpy(&d_cylinders, &(d_env->cylinders), sizeof(ppln::collision::Cylinder<float>*), hipMemcpyDeviceToHost);
            hipFree(d_cylinders);
        }
        
        if (h_env.num_cuboids > 0) {
            hipMemcpy(&d_cuboids, &(d_env->cuboids), sizeof(ppln::collision::Cuboid<float>*), hipMemcpyDeviceToHost);
            hipFree(d_cuboids);
        }
        
        if (h_env.num_z_aligned_cuboids > 0) {
            hipMemcpy(&d_z_cuboids, &(d_env->z_aligned_cuboids), sizeof(ppln::collision::Cuboid<float>*), hipMemcpyDeviceToHost);
            hipFree(d_z_cuboids);
        }

        // Finally free the environment struct itself
        hipFree(d_env);
    }

    __global__ void reset_device_variables_kernel() {
        solved = 0;
        
        atomic_free_index[0] = 0;
        atomic_free_index[1] = 0;
        
        path_size[0] = 0;
        path_size[1] = 0;
        
        for (int tree = 0; tree < 2; tree++) {
            for (int i = 0; i < 500; i++) {
                path[tree][i] = 0.0f;
            }
        }
        
        cost = 0.0f;
        reached_goal_idx = 0;
        }

    void reset_device_variables() {
        reset_device_variables_kernel<<<1, 1>>>();
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(error));
        }
    }
    
    template <typename Robot>
    __global__ void rrtc(
        float **nodes,
        int **parents,
        HaltonState<Robot> *halton_states,
        hiprandState *rng_states,
        ppln::collision::Environment<float> *env
    )
    {
        
        // printf("rrtc\n");
        static constexpr auto dim = Robot::dimension;
        const int tid = threadIdx.x;
        const int bid = blockIdx.x; // 0 ... NUM_NEW_CONFIGS
        __shared__ int t_tree_id; // this tree
        __shared__ int o_tree_id; // the other tree
        __shared__ volatile float config[dim];
        __shared__ volatile float sdata[GRANULARITY];
        __shared__ volatile unsigned int sindex[GRANULARITY];
        __shared__ volatile unsigned int local_cc_result;
        __shared__ float *t_nodes;
        __shared__ float *o_nodes;
        __shared__ int *t_parents;
        __shared__ int *o_parents;
        __shared__ float scale;
        __shared__ volatile float *nearest_node;
        __shared__ volatile float delta[dim];
        __shared__ volatile float var_cache[GRANULARITY][10];
        __shared__ volatile int index;
        __shared__ volatile float vec[dim];
        __shared__ unsigned int n_extensions;

        // printf("here1\n");
        /* sample_edges */
        // if (tid < dim) {
        //     config[tid] = hiprand_uniform(&rng_states[bid * dim + tid]);
        // }
        // __syncthreads();

        int iter=0;

        while (true){

            
            if (tid == 0) {
                iter++;
                if (iter>MAX_ITERS) atomicCAS((int *)&solved, 0, -1);
                // t_tree_id = 0;

                if (!balance || iter==1){
                    t_tree_id = (bid < (NUM_NEW_CONFIGS / 2))? 0 : 1;
                    o_tree_id = 1 - t_tree_id;
                }
                else if (balance){
                    float ratio = atomic_free_index[0]/(float)(atomic_free_index[0]+atomic_free_index[1]);
                    
                    float balance_factor = 1-ratio;
                    t_tree_id = (bid < (NUM_NEW_CONFIGS * balance_factor))? 0 : 1;
                    o_tree_id = 1 - t_tree_id;
                    
                   /*
                    if (ratio<TREE_RATIO) t_tree_id=0;
                    else t_tree_id=1;
                    o_tree_id = 1 - t_tree_id;
                    */
                }


                // if (bid == 0) printf("t_tree_id: %d\n", t_tree_id);
                t_nodes = nodes[t_tree_id];
                o_nodes = nodes[o_tree_id];
                t_parents = parents[t_tree_id];
                o_parents = parents[o_tree_id];
                // printf("here2\n");
                
                halton_next(halton_states[bid], (float *)config);
                Robot::scale_cfg((float *)config);
                //printf("iter %d\n", iter);
                //printf("config x %f\n", config[0]);
                local_cc_result = 0;
            }
            __syncthreads();

            

            // if (tid == 0 && bid == 1) {
            //     printf("sample: ");
            //     print_config(config, dim);
            // }
            // __syncthreads();

            // divide up the work of finding nearest neighbor among the threads
            float local_min_dist = INFINITY;
            unsigned int local_near_idx = 0;
            float dist;
            for (unsigned int i = 0; i < atomic_free_index[t_tree_id]; i += blockDim.x) {
                dist = device_utils::sq_l2_dist(&t_nodes[i * dim], (float *) config, dim);
                if (dist < local_min_dist) {
                    local_min_dist = dist;
                    local_near_idx = i;
                }
            }
            sdata[tid] = local_min_dist;
            sindex[tid] = local_near_idx;
            __syncthreads();

            for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
                
                float sdata_tid_s = sdata[tid + s];
                float sdata_tid = sdata[tid];
                
                __syncthreads();
                if (tid < s && (tid+s)<atomic_free_index[t_tree_id]){
                    if (sdata_tid_s < sdata_tid) {
                        sdata[tid] = sdata[tid + s];
                        sindex[tid] = sindex[tid + s];
                    }
                }
                    
                
                __syncthreads();
            }



            // by this point NN dist = sdata[0], NN index = sindex[0]
            // if (tid == 0 && bid == 0) {
            //     printf("NN dist, idx: %f, %d\n", sqrt(sdata[0]), sindex[0]);
            //     print_config(&t_nodes[sindex[0] * dim], dim);
            //     print_config(&nodes[1][0], dim);
            // }
            // __syncthreads();
            // now calculate the extension
            
            if (tid == 0) {
                scale = min(1.0f, RRT_RADIUS / sqrt(sdata[0]));
                nearest_node = &t_nodes[sindex[0] * dim];
            }
            __syncthreads();

            if (tid < dim) {
                config[tid] = nearest_node[tid] + ((config[tid] - nearest_node[tid]) * scale);
                delta[tid] = (config[tid] - nearest_node[tid]) / (float) GRANULARITY;
            }
            __syncthreads();

            //if (tid==0) printf("sample %f %f %f %f %f %f %f neighbor %f %f %f %f %f %f %f\n", config[0], config[1], config[2], config[3], config[4], config[5], config[6], nearest_node[0], nearest_node[1], nearest_node[2], nearest_node[3], nearest_node[4], nearest_node[5], nearest_node[6]);
            
            /* validate_edges */
            float interp_cfg[dim];
            for (int i = 0; i < dim; i++) {
                interp_cfg[i] = nearest_node[i] + ((tid + 1) * delta[i]);

            }
            
            
            bool config_in_collision = not ppln::collision::fkcc<Robot>(interp_cfg, env, var_cache, tid);
            // if (tid == 200 && bid == 1) {
            //     printf("device num spheres, capsules, cuboids: %d, %d, %d\n", env->num_spheres, env->num_capsules, env->num_cuboids);
            //     printf("iterp_cfg: ");
            //     print_config(interp_cfg, dim);
            //     printf("config_in_collision: %d\n", config_in_collision);
            // }
            // __syncthreads();
            atomicOr((unsigned int *)&local_cc_result, config_in_collision ? 1u : 0u);
            __syncthreads();
            // printf("here3\n");
            if (local_cc_result == 0) {
                // if (tid == 2 && bid == 0) printf("entered local_cc_result\n");
                // printf("entered local_cc_result %d %d\n", tid, bid);
                /* grow tree */
                
                if (tid == 0) {
                    index = atomicAdd((int *)&atomic_free_index[t_tree_id], 1);
                    if (index >= MAX_SAMPLES) solved=-1;
                    t_parents[index] = sindex[0];
                }
                __syncthreads();

                if (tid < dim) {
                    t_nodes[index * dim + tid] = config[tid];
                }
                __syncthreads();

                // if (tid == 0 && bid == 1) {
                //     printf("added to tree: ");
                //     print_config(config, dim);
                // }
                // __syncthreads();
                // printf("here4\n");
                /* connect */
                // find nearest neighbor in opposite tree
                local_min_dist = INFINITY;
                local_near_idx = 0;
                for (unsigned int i = 0; i < atomic_free_index[o_tree_id]; i += blockDim.x) {
                    dist = device_utils::sq_l2_dist(&o_nodes[i * dim], (float *)config, dim);
                    if (dist < local_min_dist) {
                        local_min_dist = dist;
                        local_near_idx = i;
                    }
                }
                sdata[tid] = local_min_dist;
                sindex[tid] = local_near_idx;
                __syncthreads();
                // printf("here5\n");
                for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
                    if (tid < s) {
                        if (sdata[tid + s] < sdata[tid]) {
                            sdata[tid] = sdata[tid + s];
                            sindex[tid] = sindex[tid + s];
                        }
                    }
                    __syncthreads();
                }
                // if (tid == 0 && bid == 1) {
                // printf("NN in opposite tree dist, idx: %f, %d\n", sqrt(sdata[0]), sindex[0]);
                // printf("NN o tree: ");
                // print_config(&o_nodes[sindex[0] * dim], dim);
                // }
                __syncthreads();
                
                if (tid == 0) {
                    sdata[0] = sqrt(sdata[0]);
                    // scale = min(1.0f, RRT_RADIUS / sdata[0]);
                    nearest_node = &o_nodes[sindex[0] * dim];
                    n_extensions = ceil(sdata[0] / RRT_RADIUS);
                    local_cc_result = 0;
                }
                __syncthreads();

                if (tid < dim) {
                    // vec[tid] = (nearest_node[tid] - config[tid]) * scale;
                    vec[tid] = (nearest_node[tid] - config[tid]) / (float) n_extensions;
                }
                __syncthreads();


                // if (tid == 0 && bid == 1) {
                //     printf("vec:");
                //     print_config(vec, dim);
                // }
                // __syncthreads();

                // validate the edge to the nearest neighbor in opposite tree, go as far as we can
                int i_extensions = 0;
                int extension_parent_idx = index;
                // printf("here6\n");
                while (i_extensions < n_extensions) {
                    /* each thread checking an interpolated config along the extension vector*/
                    for (int i = 0; i < dim; i++) {
                        interp_cfg[i] = config[i] + ((tid + 1) * (vec[i] / GRANULARITY));
                    }
                    bool config_in_collision = not ppln::collision::fkcc<Robot>(interp_cfg, env, var_cache, tid);
                    atomicOr((unsigned int *)&local_cc_result, config_in_collision ? 1u : 0u);
                    __syncthreads();
                    if (local_cc_result != 0) break;
                    /* add extension to tree */
                    if (tid == 0) {
                        index = atomicAdd((int *)&atomic_free_index[t_tree_id], 1);
                        if (index >= MAX_SAMPLES) solved=-1;
                        t_parents[index] = extension_parent_idx;
                        extension_parent_idx = index;
                        local_cc_result = 0;
                    }
                    __syncthreads();
                    if (tid < dim) {
                        config[tid] = config[tid] + vec[tid];
                        t_nodes[index * dim + tid] = config[tid];
                    }
                    __syncthreads();
                    // if (tid == 0 && bid == 1) {
                    //     printf("added to tree: ");
                    //     print_config(config, dim);
                    // }
                    i_extensions++;
                    __syncthreads();
                }
                // if (tid == 0) {
                //     printf("n_extensions: %d\n", n_extensions);
                //     printf("i_extensions: %d\n", i_extensions);
                // }
                // printf("here7\n");
                if (i_extensions == n_extensions) { // connected
                    if (tid == 0 && atomicCAS((int *)&solved, 0, 1) == 0) {
                        //printf("entered here %d %d\n", tid, bid);
                        //printf("n_extensions: %d\n", n_extensions);
                        // trace back to the start and goal.
                        // int current = atomic_free_index[t_tree_id] - 1;
                        int current = index;
                        int parent;
                        int t_path_size = 0;
                        int o_path_size = 0;
                        while (t_parents[current] != current) {
                            // printf("entered here1\n");
                            // printf("path config: ");
                            // print_config(&t_nodes[current*dim], dim);
                            parent = t_parents[current];
                            cost += device_utils::l2_dist(&t_nodes[current * dim], &t_nodes[parent * dim], dim);
                            for (int i = 0; i < dim; i++) path[t_tree_id][t_path_size * dim + i] = t_nodes[current * dim + i];
                            
                            print_config(&t_nodes[current * dim], dim);
                            t_path_size++;
                            current = parent;
                            
                        }
                        printf("GPU path above");
                        if (t_tree_id == 1) reached_goal_idx = current;
                        current = sindex[0];
                        // printf("entered here2\n");
                        while(o_parents[current] != current) {
                            parent = o_parents[current];
                            cost += device_utils::l2_dist(&o_nodes[current * dim], &o_nodes[parent * dim], dim);
                            for (int i = 0; i < dim; i++) path[o_tree_id][o_path_size * dim + i] = o_nodes[current * dim + i];
                            print_config(&o_nodes[current * dim], dim);
                            o_path_size++;
                            current = parent;
                        }
                        printf("GPU path above 2");
                        if (t_tree_id == 0) reached_goal_idx = current;
                        path_size[t_tree_id] = t_path_size;
                        path_size[o_tree_id] = o_path_size;
                        //printf("path_size: {%d, %d}; cost: %f\n", path_size[0], path_size[1], cost);
                        // printf("entered here3\n");
                        return;
                    }
                    __syncthreads();
                }
                // printf("here8\n");
            }
            __syncthreads();
            if (solved!=0) return;
        }
        
        
    }




    template <typename Robot>
    PlannerResult<Robot> solve(typename Robot::Configuration &start, std::vector<typename Robot::Configuration> &goals, ppln::collision::Environment<float> &h_environment) {
        static constexpr auto dim = Robot::dimension;
        std::size_t iter = 0;
        std::size_t start_index = 0;

        
        PlannerResult<Robot> res;

        // copy data to GPU
        float *start_config;
        float *goal_configs;
        int num_goals = goals.size();
        float *nodes[2];
        int *parents[2];
        float **d_nodes;
        int **d_parents;
        hipMalloc(&d_nodes, 2 * sizeof(float*));
        hipMalloc(&d_parents, 2 * sizeof(int*));
        const std::size_t config_size = dim * sizeof(float);
        hipMalloc(&start_config, config_size);
        hipMalloc(&goal_configs, config_size * num_goals);
        for (int i = 0; i < 2; i++) {
            hipMalloc(&nodes[i], MAX_SAMPLES * config_size);
            hipMalloc(&parents[i], MAX_SAMPLES * sizeof(int));
        }
        hipMemcpy(d_nodes, nodes, 2 * sizeof(float*), hipMemcpyHostToDevice);
        hipMemcpy(d_parents, parents, 2 * sizeof(int*), hipMemcpyHostToDevice);

        hipMemcpy(start_config, start.data(), config_size, hipMemcpyHostToDevice);
        hipMemcpy(goal_configs, goals.data(), config_size, hipMemcpyHostToDevice);

        // add start to tree_a and goals to tree_b
        hipMemcpy(nodes[0], start.data(), config_size, hipMemcpyHostToDevice);
        hipMemcpy(parents[0], &start_index, sizeof(int), hipMemcpyHostToDevice);

        hipMemcpy(nodes[1], goals.data(), config_size * num_goals, hipMemcpyHostToDevice);
        std::vector<int> parents_b_init(num_goals);
        iota(parents_b_init.begin(), parents_b_init.end(), 0); // consecutive integers from 0 ... num_goals - 1
        hipMemcpy(parents[1], parents_b_init.data(), sizeof(int) * num_goals, hipMemcpyHostToDevice);

        // create a hiprandState for each thread -> holds state of RNG for each thread seperately
        // For growing the tree we will create NUM_NEW_CONFIGS threads
        hiprandState *rng_states;
        int num_rng_states = NUM_NEW_CONFIGS * dim;
        hipMalloc(&rng_states, num_rng_states * sizeof(hiprandState));
        int numBlocks = (num_rng_states + BLOCK_SIZE - 1) / BLOCK_SIZE;
        init_rng<<<numBlocks, BLOCK_SIZE>>>(rng_states, 1, num_rng_states);

        HaltonState<Robot> *halton_states;
        hipMalloc(&halton_states, NUM_NEW_CONFIGS * sizeof(HaltonState<Robot>));
        int numBlocks1 = (NUM_NEW_CONFIGS + BLOCK_SIZE - 1) / BLOCK_SIZE;
        init_halton<Robot><<<numBlocks1, BLOCK_SIZE>>>(halton_states, rng_states);

        // free index for next available position in tree_a and tree_b
        int h_free_index[2] = {1, num_goals};
        hipMemcpyToSymbol(HIP_SYMBOL(atomic_free_index), &h_free_index, sizeof(int) * 2);

        // allocate for obstacles
        ppln::collision::Environment<float> *env;
        setup_environment_on_device(env, h_environment);

        cudaCheckError(hipGetLastError());
        // Setup pinned memory for signaling
        int *h_solved;
        int current_samples[2];
        hipHostMalloc(&h_solved, sizeof(int));  // Pinned memory
        *h_solved = 0;
        
        auto start_time = std::chrono::steady_clock::now();
        
        // std::cout << "iter: " << iter << "\n";
        rrtc<Robot><<<NUM_NEW_CONFIGS, GRANULARITY>>> (
            d_nodes,
            d_parents,
            halton_states,
            rng_states,
            env
        );
        hipDeviceSynchronize();
        // cudaCheckError(hipGetLastError());

        res.nanoseconds = get_elapsed_nanoseconds(start_time);

        // int current_samples[2];
        hipMemcpyFromSymbol(current_samples, HIP_SYMBOL(atomic_free_index), sizeof(int) * 2, 0, hipMemcpyDeviceToHost);
        

        hipMemcpyFromSymbol(h_solved, HIP_SYMBOL(solved), sizeof(int), 0, hipMemcpyDeviceToHost);

        // currently, iteration count is not copied because each block may have different iteration count

        if (*h_solved!=1) *h_solved=0;
        
        std::cout << "current_samples: start: " << current_samples[0] << ", goal: " << current_samples[1] << "\n";
        // printf("current_samples: %d, %d\n", current_samples[0], current_samples[1]);
        std::cout << "iters: " << iter << "\n";
        std::cout << "exited loop\n";
        Robot::print_robot_config(start);
        Robot::print_robot_config(goals[0]);
        if (*h_solved) {
            std::cout << "solved!\n";
            int h_path_size[2];
            float h_paths[2][500];
            float h_cost;
            int h_reached_goal_idx;
            hipMemcpyFromSymbol(h_path_size, HIP_SYMBOL(path_size), sizeof(int) * 2, 0, hipMemcpyDeviceToHost);
            hipMemcpyFromSymbol(h_paths, HIP_SYMBOL(path), sizeof(float) * 2 * 500, 0, hipMemcpyDeviceToHost);
            hipMemcpyFromSymbol(&h_cost, HIP_SYMBOL(cost), sizeof(float), 0, hipMemcpyDeviceToHost);
            hipMemcpyFromSymbol(&h_reached_goal_idx, HIP_SYMBOL(reached_goal_idx), sizeof(int), 0, hipMemcpyDeviceToHost);
            Robot::print_robot_config(goals[h_reached_goal_idx]);
            for (int i = h_path_size[1] - 1; i >= 0; i--) print_cfg<Robot>(&h_paths[1][i * dim]);
            for (int i = 0; i < h_path_size[0]; i++) print_cfg<Robot>(&h_paths[0][i * dim]);
            Robot::print_robot_config(start);
            res.cost = h_cost;
            std::cout << "cost: " << res.cost << "\n";
        }
        // cudaCheckError(hipGetLastError());
        res.solved = (*h_solved) != 0;
        res.iters = iter;
        
        cleanup_environment_on_device(env, h_environment);
        reset_device_variables();
        // cudaCheckError(hipGetLastError());
        hipFree(start_config);
        hipFree(goal_configs);
        // cudaCheckError(hipGetLastError());
        hipFree(nodes[0]);
        hipFree(nodes[1]);
        // cudaCheckError(hipGetLastError());
        hipFree(parents[0]);
        hipFree(parents[1]);
        // cudaCheckError(hipGetLastError());
        hipFree(rng_states);
        hipFree(halton_states);
        hipFree(d_nodes);
        hipFree(d_parents);
        // hipFree(h_solved);
        cudaCheckError(hipGetLastError());
        return res;
    }

    template PlannerResult<typename ppln::robots::Sphere> solve<ppln::robots::Sphere>(std::array<float, 3>&, std::vector<std::array<float, 3>>&, ppln::collision::Environment<float>&);
    template PlannerResult<typename ppln::robots::Panda> solve<ppln::robots::Panda>(std::array<float, 7>&, std::vector<std::array<float, 7>>&, ppln::collision::Environment<float>&);
    template PlannerResult<typename ppln::robots::Fetch> solve<ppln::robots::Fetch>(std::array<float, 8>&, std::vector<std::array<float, 8>>&, ppln::collision::Environment<float>&);
}


