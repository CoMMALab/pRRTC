#include "hip/hip_runtime.h"
#include "RRT_interleaved.hh"
#include "Robots.hh"
#include "utils.cuh"
#include "collision/environment.hh"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <vector>
#include <iostream>
#include <cassert>
#include <algorithm>

/*
Parallelized RRT with parallelized collision checking.
Interleaved strategy: sample states in parallel, then check edges in parallel, then repeat.
sample states in parallel, check edges in parallel, grow tree in parallel, check if the new configs can reach the goal in parallel
*/

namespace RRT_new {
    // Constants
    __constant__ float primes[16] = {
        3.f, 5.f, 7.f, 11.f, 13.f, 17.f, 19.f, 23.f,
        29.f, 31.f, 37.f, 41.f, 43.f, 47.f, 53.f, 59.f
    };

    template<typename Robot>
    struct HaltonState {
        float b[Robot::dimension];   // bases
        float n[Robot::dimension];   // numerators
        float d[Robot::dimension];   // denominators
    };

    template<typename Robot>
    __device__ void halton_initialize(HaltonState<Robot>& state, size_t skip_iterations) {
        // Initialize bases from primes
        for (size_t i = 0; i < Robot::dimension; i++) {
            state.b[i] = primes[i];
            state.n[i] = 0.0f;
            state.d[i] = 1.0f;
        }
        
        // Skip iterations if requested
        float temp_result[Robot::dimension];
        for (size_t i = 0; i < skip_iterations; i++) {
            halton_next(state, temp_result);
        }
    }

    template<typename Robot>
    __device__ void halton_next(HaltonState<Robot>& state, float* result) {
        for (size_t i = 0; i < Robot::dimension; i++) {
            float xf = state.d[i] - state.n[i];
            bool x_eq_1 = (xf == 1.0f);
            
            if (x_eq_1) {
                // x == 1 case
                state.d[i] = floorf(state.d[i] * state.b[i]);
                state.n[i] = 1.0f;
            } else {
                // x != 1 case
                float y = floorf(state.d[i] / state.b[i]);
                
                // Continue dividing by b until we find the right digit position
                while (xf <= y) {
                    y = floorf(y / state.b[i]);
                }
                
                state.n[i] = floorf((state.b[i] + 1.0f) * y) - xf;
            }
            
            result[i] = state.n[i] / state.d[i];
        }
    }

    __global__ void init_rng(hiprandState* states, unsigned long seed) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        hiprand_init(seed, idx, 0, &states[idx]);
    }

    template <typename Robot>
    __global__ void init_halton(HaltonState<Robot>* states) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        halton_initialize(states[idx], 0);
    }

    __device__ int atomic_free_index;
    __device__ int reached_goal = 0;
    __device__ int reached_goal_idx = -1;
    __device__ int goal_parent_idx = -1;

    constexpr int MAX_SAMPLES = 1000000;
    constexpr int MAX_ITERS = 1000000;
    constexpr int NUM_NEW_CONFIGS = 256;
    constexpr int GRANULARITY = 256;
    constexpr float RRT_RADIUS = 2.0;

    // threads per block for sample_edges and grow_tree
    constexpr int BLOCK_SIZE = 256;

    using namespace ppln;

    __device__ inline void print_config(float *config, int dim) {
        for (int i = 0; i < dim; i++) {
            printf("%f ", config[i]);
        }
        printf("\n");
    }

    inline void reset_device_variables() {
        int zero = 0;
        bool false_val = false;
        
        hipMemcpyToSymbol(HIP_SYMBOL(atomic_free_index), &zero, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(reached_goal), &zero, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(reached_goal_idx), &zero, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(goal_parent_idx), &zero, sizeof(int));
    }

    inline void setup_environment_on_device(ppln::collision::Environment<float> *&d_env, 
                                      const ppln::collision::Environment<float> &h_env) {
        // First allocate the environment struct
        hipMalloc(&d_env, sizeof(ppln::collision::Environment<float>));
        
        // Initialize struct to zeros first
        hipMemset(d_env, 0, sizeof(ppln::collision::Environment<float>));

        // Handle each primitive type separately
        if (h_env.num_spheres > 0) {
            // Allocate and copy spheres array
            ppln::collision::Sphere<float> *d_spheres;
            hipMalloc(&d_spheres, sizeof(ppln::collision::Sphere<float>) * h_env.num_spheres);
            hipMemcpy(d_spheres, h_env.spheres, 
                    sizeof(ppln::collision::Sphere<float>) * h_env.num_spheres, 
                    hipMemcpyHostToDevice);
            
            // Update the struct fields directly
            hipMemcpy(&(d_env->spheres), &d_spheres, sizeof(ppln::collision::Sphere<float>*), 
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_spheres), &h_env.num_spheres, sizeof(unsigned int), 
                    hipMemcpyHostToDevice);
        }

        if (h_env.num_capsules > 0) {
            ppln::collision::Capsule<float> *d_capsules;
            hipMalloc(&d_capsules, sizeof(ppln::collision::Capsule<float>) * h_env.num_capsules);
            hipMemcpy(d_capsules, h_env.capsules,
                    sizeof(ppln::collision::Capsule<float>) * h_env.num_capsules,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->capsules), &d_capsules, sizeof(ppln::collision::Capsule<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_capsules), &h_env.num_capsules, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }

        // Repeat for each primitive type...
        if (h_env.num_z_aligned_capsules > 0) {
            ppln::collision::Capsule<float> *d_z_capsules;
            hipMalloc(&d_z_capsules, sizeof(ppln::collision::Capsule<float>) * h_env.num_z_aligned_capsules);
            hipMemcpy(d_z_capsules, h_env.z_aligned_capsules,
                    sizeof(ppln::collision::Capsule<float>) * h_env.num_z_aligned_capsules,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->z_aligned_capsules), &d_z_capsules, sizeof(ppln::collision::Capsule<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_z_aligned_capsules), &h_env.num_z_aligned_capsules, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }

        if (h_env.num_cylinders > 0) {
            ppln::collision::Cylinder<float> *d_cylinders;
            hipMalloc(&d_cylinders, sizeof(ppln::collision::Cylinder<float>) * h_env.num_cylinders);
            hipMemcpy(d_cylinders, h_env.cylinders,
                    sizeof(ppln::collision::Cylinder<float>) * h_env.num_cylinders,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->cylinders), &d_cylinders, sizeof(ppln::collision::Cylinder<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_cylinders), &h_env.num_cylinders, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }

        if (h_env.num_cuboids > 0) {
            ppln::collision::Cuboid<float> *d_cuboids;
            hipMalloc(&d_cuboids, sizeof(ppln::collision::Cuboid<float>) * h_env.num_cuboids);
            hipMemcpy(d_cuboids, h_env.cuboids,
                    sizeof(ppln::collision::Cuboid<float>) * h_env.num_cuboids,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->cuboids), &d_cuboids, sizeof(ppln::collision::Cuboid<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_cuboids), &h_env.num_cuboids, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }

        if (h_env.num_z_aligned_cuboids > 0) {
            ppln::collision::Cuboid<float> *d_z_cuboids;
            hipMalloc(&d_z_cuboids, sizeof(ppln::collision::Cuboid<float>) * h_env.num_z_aligned_cuboids);
            hipMemcpy(d_z_cuboids, h_env.z_aligned_cuboids,
                    sizeof(ppln::collision::Cuboid<float>) * h_env.num_z_aligned_cuboids,
                    hipMemcpyHostToDevice);
            
            hipMemcpy(&(d_env->z_aligned_cuboids), &d_z_cuboids, sizeof(ppln::collision::Cuboid<float>*),
                    hipMemcpyHostToDevice);
            hipMemcpy(&(d_env->num_z_aligned_cuboids), &h_env.num_z_aligned_cuboids, sizeof(unsigned int),
                    hipMemcpyHostToDevice);
        }
    }


    inline void cleanup_environment_on_device(ppln::collision::Environment<float> *d_env, 
                                        const ppln::collision::Environment<float> &h_env) {
        // Get the pointers from device struct before freeing
        ppln::collision::Sphere<float> *d_spheres = nullptr;
        ppln::collision::Capsule<float> *d_capsules = nullptr;
        ppln::collision::Capsule<float> *d_z_capsules = nullptr;
        ppln::collision::Cylinder<float> *d_cylinders = nullptr;
        ppln::collision::Cuboid<float> *d_cuboids = nullptr;
        ppln::collision::Cuboid<float> *d_z_cuboids = nullptr;

        // Copy each pointer from device memory
        if (h_env.num_spheres > 0) {
            hipMemcpy(&d_spheres, &(d_env->spheres), sizeof(ppln::collision::Sphere<float>*), hipMemcpyDeviceToHost);
            hipFree(d_spheres);
        }
        
        if (h_env.num_capsules > 0) {
            hipMemcpy(&d_capsules, &(d_env->capsules), sizeof(ppln::collision::Capsule<float>*), hipMemcpyDeviceToHost);
            hipFree(d_capsules);
        }
        
        if (h_env.num_z_aligned_capsules > 0) {
            hipMemcpy(&d_z_capsules, &(d_env->z_aligned_capsules), sizeof(ppln::collision::Capsule<float>*), hipMemcpyDeviceToHost);
            hipFree(d_z_capsules);
        }
        
        if (h_env.num_cylinders > 0) {
            hipMemcpy(&d_cylinders, &(d_env->cylinders), sizeof(ppln::collision::Cylinder<float>*), hipMemcpyDeviceToHost);
            hipFree(d_cylinders);
        }
        
        if (h_env.num_cuboids > 0) {
            hipMemcpy(&d_cuboids, &(d_env->cuboids), sizeof(ppln::collision::Cuboid<float>*), hipMemcpyDeviceToHost);
            hipFree(d_cuboids);
        }
        
        if (h_env.num_z_aligned_cuboids > 0) {
            hipMemcpy(&d_z_cuboids, &(d_env->z_aligned_cuboids), sizeof(ppln::collision::Cuboid<float>*), hipMemcpyDeviceToHost);
            hipFree(d_z_cuboids);
        }

        // Finally free the environment struct itself
        hipFree(d_env);
    }

    // granularity = number of interpolated points to check along each edge
    // total number of threads we need is edges * granularity
    // Each block is of size granularity and it checks one edge. Each thread in the block checks a consecutive interpolated point along the edge.
    template <typename Robot>
    __global__ void validate_edges(float *new_configs, int *new_config_parents, unsigned int *cc_result, int *num_colliding_edges, ppln::collision::Environment<float> *env, float *nodes) {
        static constexpr auto dim = Robot::dimension;
        int tid_in_block = threadIdx.x;
        int bid = blockIdx.x;
        // total_threads = num_samples * granularity;
        if (bid >= NUM_NEW_CONFIGS) return;
        if (tid_in_block >= GRANULARITY) return;
        // if (bid == 0 and tid_in_block == 0) {
        //     printf("device num spheres, capsules, cuboids: %d, %d, %d\n", env->num_spheres, env->num_capsules, env->num_cuboids);
        // }
        __shared__ float delta[dim];
        __shared__ float shared_edge_start[dim];
        if (tid_in_block == 0) {
            float *edge_start = &nodes[new_config_parents[bid] * dim];
            float *edge_end = &new_configs[bid * dim];
            for (int i = 0; i < dim; i++) {
                shared_edge_start[i] = edge_start[i];
                delta[i] = (edge_end[i] - edge_start[i]) / (float) GRANULARITY;;
            }
            // if (bid == 0) {
            //     printf("edge end: ");
            //     print_config(edge_end, dim);
            // }
        }
        __syncthreads();
        // if (bid == 0 and tid_in_block == 0) {
        //     printf("edge start: ");
        //     print_config(shared_edge_start, dim);
        //     printf("projected edge end: ");
        //     for (int i = 0; i < dim; i++) {
        //         printf("%f ", shared_edge_start[i] + (GRANULARITY * delta[i]));
        //     }
        //     printf("\n");
        // }
        // calculate the configuration this thread will be checking
        float config[dim];
        for (int i = 0; i < dim; i++) {
            config[i] = shared_edge_start[i] + ((tid_in_block + 1) * delta[i]);
        }
       
        // if (bid == 0 and tid_in_block == 0) {
        //     printf("config being validated: ");
        //     print_config(config, dim);
        // }
        // __syncthreads();
        // check for collision
        bool config_in_collision = not ppln::collision::fkcc<Robot>(config, env);
        // if (bid == 0 and tid_in_block == 0) {
        //     printf("first point in edge colliding?: %d\n", config_in_collision);
        // }
        // __syncthreads();
        // if (bid == 0 and tid_in_block == 255) {
        //     printf("last config in edge: (");
        //     print_config(config, dim);
        //     printf(")last point in edge colliding?: %d\n", config_in_collision);
        // }
        // __syncthreads();
        // if (bid == 0 and (not config_in_collision)) {
        //     printf("no collision: thread %d\n", tid_in_block);
        // }
        // if (bid == 0 and config_in_collision) {
        //     printf("collision: thread %d\n", tid_in_block);
        // }
        atomicOr(&cc_result[bid], config_in_collision ? 1u : 0u);
    }


    // initialize cuda random
    // __global__ void init_rng(hiprandState* states, unsigned long seed) {
    //     int idx = blockIdx.x * blockDim.x + threadIdx.x;
    //     hiprand_init(seed, idx, 0, &states[idx]);
    // }

    // each thread is responsible for finding a new edge to check
    // sample a new state -> connect it to nearest neighbor in our tree
    template <typename Robot>
    __global__ void sample_edges(float *new_configs, int *new_config_parents, float *nodes, float *goal_configs, int num_goals, hiprandState *rng_states, HaltonState<Robot> *halton_states) {
        // printf("here!");
        static constexpr auto dim = Robot::dimension;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        if (tid >= NUM_NEW_CONFIGS) return;
        hiprandState local_rng_state = rng_states[tid];

        float *new_config = &new_configs[tid * dim];
        float config[dim];
        
        for (int i = 0; i < dim; i++) {
            config[i] = hiprand_uniform(&local_rng_state);
        }
        // halton_next(halton_states[tid], config);
        // for (int i = 0; i < dim; i++) {
        //     config[i] = set_configs[set_cfg_idx * dim + i];
        // }
        // set_cfg_idx++;
    
        // if (tid == 0) {printf("config before scaling: "); print_config(config, dim);}
        
        ppln::device_utils::scale_configuration<Robot>(config);
        // if (tid == 0) {printf("config after scaling: "); print_config(config, dim);}
        // __syncthreads();
        rng_states[tid] = local_rng_state;
        
        // Track both nearest and second nearest
        float min_dist = 1000000000.0;
        int nearest_idx = -1;

        float dist;
    
        for (int i = 0; i < atomic_free_index; i++) {
            dist = device_utils::l2_dist(&nodes[i * dim], config, dim);
            // printf("dist: %f\n", dist);
            if (dist < min_dist) {
                min_dist = dist;
                nearest_idx = i;
            }
        }

        // if (tid == 0) {
        //     printf("free_index: %d\n", atomic_free_index);
        //     printf("dist to goal: %f\n", min_dist);
        //     printf("neares idx to goal: %d\n", nearest_idx);
        // }

        // keep it within the rrt range
        float scale = min(1.0f, RRT_RADIUS / min_dist);
        float *nearest_node = &nodes[nearest_idx * dim];
        float vec[dim];
        for (int i = 0; i < dim; i++) {
            vec[i] = (config[i] - nearest_node[i]) * scale;
        }

        for (int i = 0; i < dim; i++) {
            new_config[i] = nearest_node[i] + vec[i];
        }
        // if (tid == 0) {
        //     printf("nearest (id: %d): ", nearest_idx);
        //     print_config(nearest_node, dim);
        //     printf("nearest + vec: ");
        //     print_config(new_config, dim);
        //     printf("config: ");
        //     print_config(config, dim);
        // }
        // __syncthreads();
        min_dist *= scale;
        

        // set the parent of the new config
        new_config_parents[tid] = nearest_idx;
    }

    // grow the RRT tree after we figure out what edges have no collisions
    // each thread is responsible for adding one edge to the tree
    template <typename Robot>
    __global__ void grow_tree(float *new_configs, int *new_config_parents, unsigned int *cc_result, float *nodes, int *parents, int *num_colliding_edges, float *goal_configs, int num_goals, int *new_config_idxs) {
        static constexpr auto dim = Robot::dimension;
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        if (tid >= NUM_NEW_CONFIGS) return;
        if (cc_result[tid] != 0) return;  // this edge had a collision, don't add it

        // Atomically get the next free index
        int my_index = atomicAdd(&atomic_free_index, 1);
        if (my_index >= MAX_SAMPLES) return;

        new_config_idxs[tid] = my_index;
        // Copy the configuration to the nodes array
        for (int i = 0; i < dim; i++) {
            nodes[my_index * dim + i] = new_configs[tid * dim + i];
        }
        
        // Set the parent
        parents[my_index] = new_config_parents[tid];
    }

    // Each thread will check one edge from a new_config to a goal
    template <typename Robot>
    __global__ void check_goal(float *new_configs, float *goal_configs, int num_goals, ppln::collision::Environment<float> *env, float *nodes, int *parents, unsigned int *cc_result, int *new_config_idxs) {
        static constexpr auto dim = Robot::dimension;
        // Calculate 3D thread indices
        // Each block handles one goal
        int goal_idx = blockIdx.z;
        int config_idx = blockIdx.y;  // which config we're checking
        int point_idx = threadIdx.x;  // which interpolated point along path

        if (goal_idx >= num_goals || config_idx >= NUM_NEW_CONFIGS || point_idx >= GRANULARITY) return;
        if (cc_result[config_idx] != 0) return;  // edge had collision during tree growth

        __shared__ float delta[dim];
        __shared__ float shared_start[dim];
    
        // First thread in block computes deltas
        if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
            float *new_config = &new_configs[config_idx * dim];
            float *goal = &goal_configs[goal_idx * dim];
            for (int i = 0; i < dim; i++) {
                shared_start[i] = new_config[i];
                delta[i] = (goal[i] - new_config[i]) / (float) GRANULARITY;
            }
        }
        __syncthreads();

        // Calculate and check this thread's point
        float config[dim];
        for (int i = 0; i < dim; i++) {
            config[i] = shared_start[i] + ((point_idx + 1) * delta[i]);
        }
        
        bool point_valid = ppln::collision::fkcc<Robot>(config, env);
        
        // Use shared memory to track validity within the block
        __shared__ bool edge_valid;
        if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
            edge_valid = true;
        }
        __syncthreads();
        
        // If any point is invalid, mark the edge as invalid
        atomicAnd((int*)&edge_valid, point_valid);
        __syncthreads();

        // Only one thread per edge updates the global state
        if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0 && edge_valid) {
            atomicCAS(&reached_goal, 0, 1);
            atomicCAS(&reached_goal_idx, -1, goal_idx);
            atomicCAS(&goal_parent_idx, -1, new_config_idxs[config_idx]);
        }
    }



    template <typename Robot>
    PlannerResult<Robot> solve(typename Robot::Configuration &start, std::vector<typename Robot::Configuration> &goals, ppln::collision::Environment<float> &h_environment) {
        // printCUDADeviceInfo();
        // printf("num spheres, capsules, cuboids: %d, %d, %d\n", h_environment.num_spheres, h_environment.num_capsules, h_environment.num_cuboids);
        static constexpr auto dim = Robot::dimension;
        for (int i = 0; i < dim; i++) {
            printf("%f ", start[i]);
        }
        printf("\n");
        for (int i = 0; i < dim; i++) {
            printf("%f ", goals[0][i]);
        }
        printf("\n");
        std::size_t iter = 0;
        std::size_t start_index = 0;
        std::size_t free_index = start_index + 1;

        auto start_time = std::chrono::steady_clock::now();
        PlannerResult<Robot> res;

        // copy stuff to GPU
        // GPU needs: start, goal, tree, parents, nodes

        float *start_config;
        float *goal_configs;
        int num_goals = goals.size();
        float *nodes;
        int *parents;
        const std::size_t config_size = dim * sizeof(float);
        hipMalloc(&start_config, config_size);
        cudaCheckError(hipGetLastError());
        hipMalloc(&goal_configs, config_size * num_goals);
        cudaCheckError(hipGetLastError());
        hipMalloc(&nodes, MAX_SAMPLES * config_size);
        cudaCheckError(hipGetLastError());
        hipMalloc(&parents, MAX_SAMPLES * sizeof(int));
        cudaCheckError(hipGetLastError());
        hipMemcpy(start_config, start.data(), config_size, hipMemcpyHostToDevice);
        cudaCheckError(hipGetLastError());
        hipMemcpy(goal_configs, goals.data(), config_size, hipMemcpyHostToDevice);
        cudaCheckError(hipGetLastError());
        // add the start config to the tree, and set the start to be it's own parent.
        hipMemcpy(nodes, start.data(), config_size, hipMemcpyHostToDevice);
        cudaCheckError(hipGetLastError());
        hipMemcpy(parents, &start_index, sizeof(int), hipMemcpyHostToDevice);
        cudaCheckError(hipGetLastError());

        // create a hiprandState for each thread -> holds state of RNG for each thread seperately
        // For growing the tree we will create NUM_NEW_CONFIGS threads
        hiprandState *rng_states;
        hipMalloc(&rng_states, NUM_NEW_CONFIGS * sizeof(hiprandState));
        // constexpr int blockSize = 256;
        int numBlocks = (NUM_NEW_CONFIGS + BLOCK_SIZE - 1) / BLOCK_SIZE;
        init_rng<<<numBlocks, BLOCK_SIZE>>>(rng_states, 1);

        HaltonState<Robot> *halton_states;
        hipMalloc(&halton_states, NUM_NEW_CONFIGS * sizeof(HaltonState<Robot>));
        init_halton<Robot><<<numBlocks, BLOCK_SIZE>>>(halton_states);

        // create arrays on the gpu to hold the newly sampled configs, and their parents, and dist to parent
        float *new_configs;
        hipMalloc(&new_configs, NUM_NEW_CONFIGS * config_size);
        int *new_config_parents;
        hipMalloc(&new_config_parents, NUM_NEW_CONFIGS * sizeof(int));
        cudaCheckError(hipGetLastError());
        int *new_config_idxs;
        hipMalloc(&new_config_idxs, NUM_NEW_CONFIGS * sizeof(int));

        // create an array to hold the result of collision check for each new edge
        unsigned int *cc_result;
        hipMalloc(&cc_result, NUM_NEW_CONFIGS * sizeof(unsigned int));
        hipMemset(cc_result, 0, NUM_NEW_CONFIGS * sizeof(unsigned int));
        int *num_colliding_edges;
        hipMalloc(&num_colliding_edges, sizeof(int));
        cudaCheckError(hipGetLastError());

        // free index for next available position in the nodes array
        hipMemcpyToSymbol(HIP_SYMBOL(atomic_free_index), &free_index, sizeof(int));
        cudaCheckError(hipGetLastError());

        // allocate for obstacles
        ppln::collision::Environment<float> *env;
        // hipMalloc(&env, sizeof(env));
        // hipMemcpy(env, &h_environment, sizeof(env), hipMemcpyHostToDevice);
        setup_environment_on_device(env, h_environment);
        cudaCheckError(hipGetLastError());
        int done = 0;

        // calculate launch configuration for check_goals
        dim3 threadsPerBlock1(GRANULARITY);  // 512 threads total
        dim3 numBlocks1(
            1, NUM_NEW_CONFIGS, num_goals
        );


        // main RRT loop
        while (iter++ < MAX_ITERS && free_index < MAX_SAMPLES) {
            std::cout << "iter: " << iter << std::endl;
            
            // sample configurations and get edges to be checked
            // auto kernel_start_time = std::chrono::steady_clock::now();
            sample_edges<Robot><<<numBlocks, BLOCK_SIZE>>>(new_configs, new_config_parents, nodes, goal_configs, num_goals, rng_states, halton_states);
            cudaCheckError(hipGetLastError());
            hipDeviceSynchronize();
            // std::cout << "sample edges (ns): " << get_elapsed_nanoseconds(kernel_start_time) << "\n";
            
            // collision check all the edges
            // kernel_start_time = std::chrono::steady_clock::now();
            hipMemset(cc_result, 0, NUM_NEW_CONFIGS * sizeof(unsigned int));
            validate_edges<Robot><<<NUM_NEW_CONFIGS, GRANULARITY>>>(new_configs, new_config_parents, cc_result, num_colliding_edges, env, nodes);
            cudaCheckError(hipGetLastError());
            hipDeviceSynchronize();
            // std::cout << "validate edges (ns): " << get_elapsed_nanoseconds(kernel_start_time) << "\n";
            
            // add all the new edges to the tree
            // kernel_start_time = std::chrono::steady_clock::now();
            grow_tree<Robot><<<numBlocks, BLOCK_SIZE>>>(new_configs, new_config_parents, cc_result, nodes, parents, num_colliding_edges, goal_configs, num_goals, new_config_idxs);
            cudaCheckError(hipGetLastError());
            hipDeviceSynchronize();
            // std::cout << "grow tree (ns): " << get_elapsed_nanoseconds(kernel_start_time) << "\n";

            // check whether each new configuration added to the tree can reach the goal
            // kernel_start_time = std::chrono::steady_clock::now();
            check_goal<Robot><<<numBlocks1, threadsPerBlock1>>>(new_configs, goal_configs, num_goals, env, nodes, parents, cc_result, new_config_idxs);
            cudaCheckError(hipGetLastError());
            hipDeviceSynchronize();
            // std::cout << "check goal (ns): " << get_elapsed_nanoseconds(kernel_start_time) << "\n";

            // update free index
            hipMemcpyFromSymbol(&free_index, HIP_SYMBOL(atomic_free_index), sizeof(int), 0, hipMemcpyDeviceToHost);
            cudaCheckError(hipGetLastError());
            hipMemcpyFromSymbol(&done, HIP_SYMBOL(reached_goal), sizeof(int), 0, hipMemcpyDeviceToHost);
            cudaCheckError(hipGetLastError());
            if (done) break;
        }
        res.tree_size = free_index;
        res.iters = iter;
        res.attempted_tree_size = NUM_NEW_CONFIGS * iter;
        // retrieve data from gpu
        std::vector<int> h_parents(MAX_SAMPLES);
        std::vector<float> h_nodes(MAX_SAMPLES * dim);
        hipMemcpy(h_parents.data(), parents, MAX_SAMPLES * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_nodes.data(), nodes, MAX_SAMPLES * config_size, hipMemcpyDeviceToHost);
        
        if (done) {
            printf("done!\n");
            // get the index of the goal we found in the goals array
            int h_goal_idx;
            hipMemcpyFromSymbol(&h_goal_idx, HIP_SYMBOL(reached_goal_idx), sizeof(int), 0, hipMemcpyDeviceToHost);
            std::cout << "Found Goal: " << h_goal_idx << std::endl;
            res.solved = true;
            // get parent at position 0 in new_config_parents, because that will be the parent of the goal
            int parent_idx = -1;
            hipMemcpyFromSymbol(&parent_idx, HIP_SYMBOL(goal_parent_idx), sizeof(int), 0, hipMemcpyDeviceToHost);
            assert(parent_idx != -1);
            typename Robot::Configuration cfg;
            typename Robot::Configuration cfg_parent;
            std::copy_n(h_nodes.begin() + (parent_idx * dim), dim, cfg.begin());
            res.cost += l2dist<Robot>(goals[h_goal_idx], cfg);
            Robot::print_robot_config(cfg);
            while (parent_idx != h_parents[parent_idx]) {
                // std::cout << parent_idx << std::endl;
                std::copy_n(h_nodes.begin() + parent_idx * dim, dim, cfg.begin());
                std::copy_n(h_nodes.begin() + h_parents[parent_idx] * dim, dim, cfg_parent.begin());
                Robot::print_robot_config(cfg);
                res.cost += l2dist<Robot>(cfg, cfg_parent);
                res.path.emplace_back(parent_idx);
                parent_idx = h_parents[parent_idx];
            }
            res.path.emplace_back(parent_idx);
            std::reverse(res.path.begin(), res.path.end());
        }
        res.nanoseconds = get_elapsed_nanoseconds(start_time);
        // printf("almost done\n");
        reset_device_variables();
        cudaCheckError(hipGetLastError());
        cleanup_environment_on_device(env, h_environment);
        cudaCheckError(hipGetLastError());
        hipFree(start_config);
        hipFree(goal_configs);
        hipFree(nodes);
        hipFree(parents);
        hipFree(rng_states);
        hipFree(new_configs);
        hipFree(new_config_parents);
        hipFree(cc_result);
        hipFree(num_colliding_edges);
        // hipFree(env);
        cudaCheckError(hipGetLastError());
        return res;
    }

    template PlannerResult<typename ppln::robots::Sphere> solve<ppln::robots::Sphere>(std::array<float, 3>&, std::vector<std::array<float, 3>>&, ppln::collision::Environment<float>&);
    template PlannerResult<typename ppln::robots::Panda> solve<ppln::robots::Panda>(std::array<float, 7>&, std::vector<std::array<float, 7>>&, ppln::collision::Environment<float>&);
}



